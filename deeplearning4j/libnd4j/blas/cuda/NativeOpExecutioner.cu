#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

#include "../NativeOpExecutioner.h"
#include <hip/hip_runtime.h>
#include <op_boilerplate.h>
#include <helpers/DebugHelper.h>
#include <DataTypeUtils.h>
#include <exceptions/datatype_exception.h>
#include <exceptions/cuda_exception.h>
#include <helpers/CudaLaunchHelper.h>
#include <helpers/ShapeBuilders.h>
#include <PointersManager.h>

#include <array/ConstantDataBuffer.h>
#include <array/ShapeDescriptor.h>
#include <helpers/ConstantShapeHelper.h>

#include <loops/transform_float.h>
#include <loops/transform_bool.h>
#include <loops/transform_any.h>
#include <loops/transform_same.h>
#include <loops/transform_strict.h>
#include <loops/reduce_float.h>
#include <loops/reduce_same.h>
#include <loops/reduce_bool.h>
#include <loops/reduce_long.h>
#include <loops/indexreduce.h>
#include <loops/pairwise_transform.h>
#include <loops/pairwise_bool.h>
#include <loops/pairwise_int.h>
#include <loops/broadcasting_bool.h>
#include <loops/broadcasting_int.h>
#include <loops/broadcasting.h>
#include <loops/reduce_float.h>
#include <loops/reduce3.h>
#include <loops/summarystatsreduce.h>
#include <loops/transform_same.h>
#include <loops/random.h>
#include <loops/special_kernels.h>
#include <loops/scalar.h>
#include <loops/scalar_bool.h>
#include <loops/scalar_int.h>

using namespace nd4j;

/**
* This is utility kernel, that updates given special buffer with proper values in device memory
*/
extern "C" __global__ void prepareShapeBuffer(int *dimension, int *maxDimension, Nd4jLong *specialPointer, int rows, nd4j::DataType dataType) {
    Nd4jLong tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid > 0)
        return;

    dimension[0] = 0;
    maxDimension[0] = 1;

    specialPointer[0] = 2;
    specialPointer[1] = rows;
    specialPointer[2] = 1;
    specialPointer[3] = 1;
    specialPointer[4] = 1;
    specialPointer[5] = 0;
    specialPointer[6] = 1;
    specialPointer[7] = 99;

    ArrayOptions::setDataType(specialPointer, dataType);

    //printf("special[0]: [%lld]\n", (long long) specialPointer[0]);
    //shape::printShapeInfoLinear("prepareShapeBuffer", specialPointer);
}


////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execPairwiseTransform(nd4j::LaunchContext  *lc,
                                    int opNum,
                                    void *hX, Nd4jLong *hXShapeInfo,
                                    void *dX, Nd4jLong *dXShapeInfo,
                                    void *hY, Nd4jLong *hYShapeInfo,
                                    void *dY, Nd4jLong *dYShapeInfo,
                                    void *hZ, Nd4jLong *hZShapeInfo,
                                    void *dZ, Nd4jLong *dZShapeInfo,
                                    void *extraParams) {

    auto stream = lc->getCudaStream();

    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto yType = nd4j::ArrayOptions::dataType(hYShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

    if (xType != zType && yType != zType)
        throw std::runtime_error("NativeOpExecutioner::execPairwiseTransform requires Z operand to have either X or Y type");
    if (lc == nullptr)
        throw std::runtime_error("NativeOpExecutioner::execPairwiseTransform: launch context cannot be nullptr !");
    if (stream == nullptr)
        throw std::runtime_error("NativeOpExecutioner::execPairwiseTransform: CUDA stream cannot be nullptr !");

    dim3 launchDims(256, 1024, 8192);

#ifdef __ND4J_EXPERIMENTAL__
    BUILD_PAIRWISE_SELECTOR(xType, yType, zType, functions::pairwise_transforms::PairWiseTransform, ::executeCudaShaped(launchDims, stream, opNum, dX, dXShapeInfo, dY, dYShapeInfo, dZ, dZShapeInfo, extraParams), LIBND4J_TYPES, LIBND4J_TYPES)
#else
    BUILD_SINGLE_SELECTOR_THRICE(xType, functions::pairwise_transforms::PairWiseTransform, ::executeCudaShaped(launchDims, stream, opNum, dX, dXShapeInfo, dY, dYShapeInfo, dZ, dZShapeInfo, extraParams), LIBND4J_TYPES)
#endif

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execPairwiseTransform failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execPairwiseBoolTransform( nd4j::LaunchContext  *lc,
                                                    int opNum,
                                                    void *hX, Nd4jLong *hXShapeInfo,
                                                    void *dX, Nd4jLong *dXShapeInfo,
                                                    void *hY, Nd4jLong *hYShapeInfo,
                                                    void *dY, Nd4jLong *dYShapeInfo,
                                                    void *hZ, Nd4jLong *hZShapeInfo,
                                                    void *dZ, Nd4jLong *dZShapeInfo,
                                                    void *extraParams) {

	auto stream = lc->getCudaStream();

    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto yType = nd4j::ArrayOptions::dataType(hYShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

    if (!DataTypeUtils::isB(zType))
		throw nd4j::datatype_exception::build("NativeOpExecutioner::execPairwiseBoolTransform wrong Z operand data type", nd4j::DataType::BOOL, zType);

    if (yType != xType)
        throw nd4j::datatype_exception::build("NativeOpExecutioner::execPairwiseBoolTransform both operands must have same data type", xType, yType);

    dim3 launchDims(256, 1024, 16384);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::pairwise_transforms::PairWiseBoolTransform, ::executeCudaShaped(launchDims, stream, opNum, dX, dXShapeInfo, dY, dYShapeInfo, dZ, dZShapeInfo, extraParams), LIBND4J_TYPES, BOOL_TYPES)

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execPairwiseBoolTransform failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execPairwiseIntTransform( nd4j::LaunchContext  *lc,
                                                     int opNum,
                                                     void *hX, Nd4jLong *hXShapeInfo,
                                                     void *dX, Nd4jLong *dXShapeInfo,
                                                     void *hY, Nd4jLong *hYShapeInfo,
                                                     void *dY, Nd4jLong *dYShapeInfo,
                                                     void *hZ, Nd4jLong *hZShapeInfo,
                                                     void *dZ, Nd4jLong *dZShapeInfo,
                                                     void *extraParams) {

    auto stream = lc->getCudaStream();

    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto yType = nd4j::ArrayOptions::dataType(hYShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

    if (!DataTypeUtils::isZ(zType))
        throw nd4j::datatype_exception::build("NativeOpExecutioner::execPairwiseIntTransform wrong Z operand data type", nd4j::DataType::BOOL, zType);

    if (yType != xType || zType != xType)
        throw nd4j::datatype_exception::build("NativeOpExecutioner::execPairwiseIntTransform both operands must have same data type", xType, yType);

    dim3 launchDims(256, 1024, 16384);

    BUILD_SINGLE_SELECTOR(xType, functions::pairwise_transforms::PairWiseIntTransform, ::executeCudaShaped(launchDims, stream, opNum, dX, dXShapeInfo, dY, dYShapeInfo, dZ, dZShapeInfo, extraParams), INTEGER_TYPES)

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execPairwiseIntTransform failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execSummaryStatsScalar(nd4j::LaunchContext  *lc,
                                    int opNum,
                                    void *hX, Nd4jLong *hXShapeInfo,
                                    void *dX, Nd4jLong *dXShapeInfo,
                                    void *extraParams,
                                    void *hZ, Nd4jLong *hZShapeInfo,
                                    void *dZ, Nd4jLong *dZShapeInfo,
                                    bool biasCorrected) {

	auto stream = lc->getCudaStream();
    auto reductionPointer = lc->getReductionPointer();

    dim3 launchDims = dim3(256, 256, 32768);

	auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
	auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::summarystats::SummaryStatsReduce, ::execSummaryStatsReduceScalar(launchDims, stream, opNum, dX, dXShapeInfo, hXShapeInfo, extraParams, dZ, dZShapeInfo, hZShapeInfo, nullptr, nullptr, biasCorrected, reductionPointer), LIBND4J_TYPES, FLOAT_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execSummaryStatsScalar failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execBroadcastBool(nd4j::LaunchContext  *lc,
                            int opNum,
                            void *hX, Nd4jLong *hXShapeInfo,
                            void *dX, Nd4jLong *dXShapeInfo,
                            void *hY, Nd4jLong *hYShapeInfo,
                            void *dY, Nd4jLong *dYShapeInfo,
                            void *hZ, Nd4jLong *hZShapeInfo,
                            void *dZ, Nd4jLong *dZShapeInfo,
                            int *dimension, int dimensionLength,
                            Nd4jLong *tadOnlyShapeInfo, Nd4jLong *tadOffsets,
                            Nd4jLong *tadOnlyShapeInfoZ,Nd4jLong *tadOffsetsZ) {

	auto stream = lc->getCudaStream();

	auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
	auto yType = nd4j::ArrayOptions::dataType(hYShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

	if (!DataTypeUtils::isB(zType))
        throw std::runtime_error("NativeOpExecutioner::execBroadcastBool requires Z operand to have BOOL type");

    if (yType != xType)
        throw std::runtime_error("NativeOpExecutioner::execBroadcastBool requires both X & Y operands to have same type");

	if (nd4j::Environment::getInstance()->isDebugAndVerbose())
		printf("F3B opNum:[%i]\n", opNum);

	dim3 launchDims(256, 256, 1024);

	BUILD_DOUBLE_SELECTOR(xType, zType, functions::broadcast::BroadcastBool, ::execBroadcast(launchDims, stream, opNum, dX, dXShapeInfo, dY, dYShapeInfo, dZ, dZShapeInfo, dimension, dimensionLength, tadOnlyShapeInfo, tadOffsets, tadOnlyShapeInfoZ, tadOffsetsZ), LIBND4J_TYPES, BOOL_TYPES)

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execBroadcastBool failed", res);
}

void NativeOpExecutioner::execInverseBroadcastBool(nd4j::LaunchContext  *lc,
                                                   int opNum,
                                                   void *hX, Nd4jLong *hXShapeInfo,
                                                   void *dX, Nd4jLong *dXShapeInfo,
                                                   void *hY, Nd4jLong *hYShapeInfo,
                                                   void *dY, Nd4jLong *dYShapeInfo,
                                                   void *hZ, Nd4jLong *hZShapeInfo,
                                                   void *dZ, Nd4jLong *dZShapeInfo,
                                                   int *dimension, int dimensionLength,
                                                   Nd4jLong *tadOnlyShapeInfo, Nd4jLong *tadOffsets,
                                                   Nd4jLong *tadOnlyShapeInfoZ,Nd4jLong *tadOffsetsZ) {
    auto stream = lc->getCudaStream();

    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto yType = nd4j::ArrayOptions::dataType(hYShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

    if (!DataTypeUtils::isB(zType))
        throw std::runtime_error("NativeOpExecutioner::execBroadcastBool requires Z operand to have BOOL type");

    if (yType != xType)
        throw std::runtime_error("NativeOpExecutioner::execBroadcastBool requires both X & Y operands to have same type");

    if (nd4j::Environment::getInstance()->isDebugAndVerbose())
        printf("F3BI opNum:[%i]\n", opNum);

    dim3 launchDims(256, 256, 1024);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::broadcast::BroadcastBool, ::execInverseBroadcast(launchDims, stream, opNum, dX, dXShapeInfo, dY, dYShapeInfo, dZ, dZShapeInfo, dimension, dimensionLength, tadOnlyShapeInfo, tadOffsets, tadOnlyShapeInfoZ, tadOffsetsZ), LIBND4J_TYPES, BOOL_TYPES)

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execInverseBroadcastBool failed", res);
}


////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execBroadcastInt(nd4j::LaunchContext  *lc,
                                            int opNum,
                                            void *hX, Nd4jLong *hXShapeInfo,
                                            void *dX, Nd4jLong *dXShapeInfo,
                                            void *hY, Nd4jLong *hYShapeInfo,
                                            void *dY, Nd4jLong *dYShapeInfo,
                                            void *hZ, Nd4jLong *hZShapeInfo,
                                            void *dZ, Nd4jLong *dZShapeInfo,
                                            int *dimension, int dimensionLength,
                                            Nd4jLong *tadOnlyShapeInfo, Nd4jLong *tadOffsets,
                                            Nd4jLong *tadOnlyShapeInfoZ,Nd4jLong *tadOffsetsZ) {

    auto stream = lc->getCudaStream();

    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto yType = nd4j::ArrayOptions::dataType(hYShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

    if (!DataTypeUtils::isZ(zType))
        throw std::runtime_error("NativeOpExecutioner::execBroadcastInt requires Z operand to have INT type");

    if (yType != xType || zType != xType)
        throw std::runtime_error("NativeOpExecutioner::execBroadcastInt requires both X & Y operands to have same type");

    if (nd4j::Environment::getInstance()->isDebugAndVerbose())
        printf("F3B opNum:[%i]\n", opNum);

    dim3 launchDims(256, 256, 1024);

    BUILD_SINGLE_SELECTOR(xType, functions::broadcast::BroadcastInt, ::execBroadcast(launchDims, stream, opNum, dX, dXShapeInfo, dY, dYShapeInfo, dZ, dZShapeInfo, dimension, dimensionLength, tadOnlyShapeInfo, tadOffsets, tadOnlyShapeInfoZ, tadOffsetsZ), INTEGER_TYPES)

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execBroadcastBool failed", res);
}

void NativeOpExecutioner::execInverseBroadcastInt(nd4j::LaunchContext  *lc,
                                                   int opNum,
                                                   void *hX, Nd4jLong *hXShapeInfo,
                                                   void *dX, Nd4jLong *dXShapeInfo,
                                                   void *hY, Nd4jLong *hYShapeInfo,
                                                   void *dY, Nd4jLong *dYShapeInfo,
                                                   void *hZ, Nd4jLong *hZShapeInfo,
                                                   void *dZ, Nd4jLong *dZShapeInfo,
                                                   int *dimension, int dimensionLength,
                                                   Nd4jLong *tadOnlyShapeInfo, Nd4jLong *tadOffsets,
                                                   Nd4jLong *tadOnlyShapeInfoZ,Nd4jLong *tadOffsetsZ) {
    auto stream = lc->getCudaStream();

    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto yType = nd4j::ArrayOptions::dataType(hYShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

    if (!DataTypeUtils::isZ(zType))
        throw std::runtime_error("NativeOpExecutioner::execBroadcastInt requires Z operand to have INT type");

    if (yType != xType || zType != xType)
        throw std::runtime_error("NativeOpExecutioner::execBroadcastInt requires both X & Y operands to have same type");

    if (nd4j::Environment::getInstance()->isDebugAndVerbose())
        printf("F3BI opNum:[%i]\n", opNum);

    dim3 launchDims(256, 256, 1024);

    BUILD_SINGLE_SELECTOR(xType, functions::broadcast::BroadcastInt, ::execInverseBroadcast(launchDims, stream, opNum, dX, dXShapeInfo, dY, dYShapeInfo, dZ, dZShapeInfo, dimension, dimensionLength, tadOnlyShapeInfo, tadOffsets, tadOnlyShapeInfoZ, tadOffsetsZ), INTEGER_TYPES)

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execInverseBroadcastInt failed", res);
}

////////////////////////////////////////////////////////////////////////
/**
 *
 * @param opNum
 * @param dX
 * @param dXShapeInfo
 * @param dY
 * @param dYShapeInfo
 * @param dZ
 * @param dZShapeInfo
 * @param dimension
 * @param dimensionLength
 */
void NativeOpExecutioner::execBroadcast(nd4j::LaunchContext  *lc,
		                              int opNum,
		                              void *hX, Nd4jLong *hXShapeInfo,
		                              void *dX, Nd4jLong *dXShapeInfo,
		                              void *hY, Nd4jLong *hYShapeInfo,
		                              void *dY, Nd4jLong *dYShapeInfo,
		                              void *hZ, Nd4jLong *hZShapeInfo,
		                              void *dZ, Nd4jLong *dZShapeInfo,
		                              int *dimension, int dimensionLength,
		                              Nd4jLong *tadOnlyShapeInfo, Nd4jLong *tadOffsets,
		                              Nd4jLong *tadOnlyShapeInfoZ,Nd4jLong *tadOffsetsZ) {

	auto stream = lc->getCudaStream();

	auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
	auto yType = nd4j::ArrayOptions::dataType(hYShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

	if (nd4j::Environment::getInstance()->isDebugAndVerbose())
		printf("F3 opNum:[%i]\n", opNum);

	dim3 launchDims(256, 256, 1024);

#ifdef __ND4J_EXPERIMENTAL__
	BUILD_PAIRWISE_SELECTOR(xType, yType, zType, functions::broadcast::Broadcast, ::execBroadcast(launchDims, stream, opNum, dX, dXShapeInfo, dY, dYShapeInfo, dZ, dZShapeInfo, dimension, dimensionLength, tadOnlyShapeInfo, tadOffsets, tadOnlyShapeInfoZ, tadOffsetsZ), LIBND4J_TYPES, LIBND4J_TYPES);
#else
    BUILD_SINGLE_SELECTOR_THRICE(xType, functions::broadcast::Broadcast, ::execBroadcast(launchDims, stream, opNum, dX, dXShapeInfo, dY, dYShapeInfo, dZ, dZShapeInfo, dimension, dimensionLength, tadOnlyShapeInfo, tadOffsets, tadOnlyShapeInfoZ, tadOffsetsZ), LIBND4J_TYPES);
#endif

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execBroadcast failed", res);
}

void NativeOpExecutioner::execInverseBroadcast(nd4j::LaunchContext  *lc,
                                               int opNum,
                                               void *hX, Nd4jLong *hXShapeInfo,
                                               void *dX, Nd4jLong *dXShapeInfo,
                                               void *hY, Nd4jLong *hYShapeInfo,
                                               void *dY, Nd4jLong *dYShapeInfo,
                                               void *hZ, Nd4jLong *hZShapeInfo,
                                               void *dZ, Nd4jLong *dZShapeInfo,
                                               int *dimension, int dimensionLength,
                                               Nd4jLong *tadOnlyShapeInfo, Nd4jLong *tadOffsets,
                                               Nd4jLong *tadOnlyShapeInfoZ,Nd4jLong *tadOffsetsZ) {

    auto stream = lc->getCudaStream();

    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto yType = nd4j::ArrayOptions::dataType(hYShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

    if (nd4j::Environment::getInstance()->isDebugAndVerbose())
        printf("F3I opNum:[%i]\n", opNum);

    dim3 launchDims(256, 256, 1024);

#ifdef __ND4J_EXPERIMENTAL__
    BUILD_PAIRWISE_SELECTOR(xType, yType, zType, functions::broadcast::Broadcast, ::execInverseBroadcast(launchDims, stream, opNum, dX, dXShapeInfo, dY, dYShapeInfo, dZ, dZShapeInfo, dimension, dimensionLength, tadOnlyShapeInfo, tadOffsets, tadOnlyShapeInfoZ, tadOffsetsZ), LIBND4J_TYPES, LIBND4J_TYPES);
#else
    BUILD_SINGLE_SELECTOR_THRICE(xType, functions::broadcast::Broadcast, ::execInverseBroadcast(launchDims, stream, opNum, dX, dXShapeInfo, dY, dYShapeInfo, dZ, dZShapeInfo, dimension, dimensionLength, tadOnlyShapeInfo, tadOffsets, tadOnlyShapeInfoZ, tadOffsetsZ), LIBND4J_TYPES);
#endif

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execInverseBroadcast failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execReduceSame(nd4j::LaunchContext  *lc,
                            int opNum,
                            void *hX, Nd4jLong *hXShapeInfo,
                            void *dX, Nd4jLong *dXShapeInfo,
                            void *extraParams,
                            void *hZ, Nd4jLong *hZShapeInfo,
                            void *dZ, Nd4jLong *dZShapeInfo,
                            int *dimension, int dimensionLength,
                            Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets) {

	auto stream = lc->getCudaStream();
	auto reductionPointer = lc->getReductionPointer();

    if (nd4j::Environment::getInstance()->isDebugAndVerbose())
        printf("SF7 opNum:[%i]\n", opNum);

    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);
    auto xRank = shape::rank(hXShapeInfo);

    if (zType != xType)
        throw datatype_exception::build("NativeOpExecutioner::execReduceSame requires both X & Z operands to have same type", xType, zType);

    auto numBlocks = shape::length(hZShapeInfo);
    dim3 launchDims(numBlocks, 256, 8192);

    BUILD_SINGLE_SELECTOR(xType, functions::reduce::ReduceSameFunction, ::execReduceXD(launchDims, stream, opNum, xRank, dX, dXShapeInfo, hXShapeInfo, extraParams, dZ, dZShapeInfo, hZShapeInfo, dimension, dimensionLength, reductionPointer, tadShapeInfo, tadOffsets), LIBND4J_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execReduceSame failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execReduceLong(nd4j::LaunchContext  *lc,
                            int opNum,
                            void *hX, Nd4jLong *hXShapeInfo,
                            void *dX, Nd4jLong *dXShapeInfo,
                            void *extraParams,
                            void *hZ, Nd4jLong *hZShapeInfo,
                            void *dZ, Nd4jLong *dZShapeInfo,
                            int *dimension,int dimensionLength,
                            Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets) {

	auto stream = lc->getCudaStream();
	auto reductionPointer = lc->getReductionPointer();

    if (nd4j::Environment::getInstance()->isDebugAndVerbose())
        printf("LF7 opNum:[%i]\n", opNum);

    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

    if (zType != nd4j::DataType::INT64)
        throw datatype_exception::build("NativeOpExecutioner::execReduceLong wrong Z data type", nd4j::DataType::INT64, zType);

    auto xRank = shape::rank(hXShapeInfo);
    auto numBlocks = shape::length(hZShapeInfo);
    dim3 launchDims(numBlocks, 256, 32768);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::reduce::ReduceLongFunction, ::execReduceXD(launchDims, stream, opNum, xRank, dX, dXShapeInfo, hXShapeInfo, extraParams, dZ, dZShapeInfo, hZShapeInfo, dimension, dimensionLength, reductionPointer, tadShapeInfo, tadOffsets), LIBND4J_TYPES, LONG_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execReduceLong failed", res);

}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execReduceBool(nd4j::LaunchContext  *lc,
                            int opNum,
                            void *hX, Nd4jLong *hXShapeInfo,
                            void *dX, Nd4jLong *dXShapeInfo,
                            void *extraParams,
                            void *hZ, Nd4jLong *hZShapeInfo,
                            void *dZ, Nd4jLong *dZShapeInfo,
                            int *dimension, int dimensionLength,
                            Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets) {

	auto stream = lc->getCudaStream();
	auto reductionPointer = lc->getReductionPointer();

    if (nd4j::Environment::getInstance()->isDebugAndVerbose())
        printf("BF7 opNum:[%i]\n", opNum);

    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

    if (zType != nd4j::DataType::BOOL)
        throw std::runtime_error("NativeOpExecutioner::execReduceBool requires Z operand to have BOOL type");

    auto xRank = shape::rank(hXShapeInfo);
    auto numBlocks = shape::length(hZShapeInfo);
    dim3 launchDims(numBlocks, 256, 32768);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::reduce::ReduceBoolFunction, ::execReduceXD(launchDims, stream, opNum, xRank, dX, dXShapeInfo, hXShapeInfo, extraParams, dZ, dZShapeInfo, hZShapeInfo, dimension, dimensionLength, reductionPointer, tadShapeInfo, tadOffsets), LIBND4J_TYPES, BOOL_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execReduceBool failed", res);
}

////////////////////////////////////////////////////////////////////////
/**
 *
 * @param opNum
 * @param dX
 * @param dXShapeInfo
 * @param extraParams
 * @param dZ
 * @param dZShapeInfo
 * @param dimension
 * @param dimensionLength
 */
void NativeOpExecutioner::execIndexReduce(nd4j::LaunchContext  *lc,
                                int opNum,
                                void *hX, Nd4jLong *hXShapeInfo,
                                void *dX, Nd4jLong *dXShapeInfo,
                                void *extraParams,
                                void *hZ, Nd4jLong *hZShapeInfo,
                                void *dZ, Nd4jLong *dZShapeInfo,
                                int *dimension, int dimensionLength,
                                Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets) {

	auto stream = lc->getCudaStream();
	auto reductionPointer = lc->getReductionPointer();
	auto allocationPointer = lc->getAllocationPointer();

	if (nd4j::Environment::getInstance()->isDebugAndVerbose())
		printf("F2 opNum:[%i]\n", opNum);

	auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);
	auto numBlocks = shape::length(hZShapeInfo);
    dim3 launchDims(numBlocks, 256, 32768);

    if (zType != nd4j::DataType::INT64 && zType != nd4j::DataType::INT32)
        throw datatype_exception::build("NativeOpExecutioner::execIndexReduce requires Z operand to have INT32/INT64 type", zType);

	auto dz = reinterpret_cast<Nd4jLong*>(dZ);

	BUILD_DOUBLE_SELECTOR(xType, zType, functions::indexreduce::IndexReduce,  ::executeIndexReduce(launchDims, stream, opNum, dX, dXShapeInfo, shape::rank(hXShapeInfo), extraParams, dz, dZShapeInfo, shape::rank(hZShapeInfo), dimension, dimensionLength, 1, allocationPointer, reductionPointer, tadShapeInfo, tadOffsets), LIBND4J_TYPES, INDEXING_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execIndexReduce failed", res);
}

////////////////////////////////////////////////////////////////////////
/**
 *
 * @param opNum
 * @param dX
 * @param dXShapeInfo
 * @param extraParams
 * @param dZ
 * @param dZShapeInfo
 */
void  NativeOpExecutioner::execReduceFloat(nd4j::LaunchContext  *lc,
										int opNum,
										void *hX, Nd4jLong *hXShapeInfo,
        								void *dX, Nd4jLong *dXShapeInfo,
        								void *extraParams,
        								void *hZ, Nd4jLong *hZShapeInfo,
										void *dZ, Nd4jLong *dZShapeInfo,
										int *dimension,int dimensionLength,
										Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets) {

	auto stream = lc->getCudaStream();
	auto reductionPointer = lc->getReductionPointer();

	if (nd4j::Environment::getInstance()->isDebugAndVerbose())
		printf("F8 opNum:[%i]\n", opNum);

	auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

    auto xRank = shape::rank(hXShapeInfo);
    auto numBlocks = shape::length(hZShapeInfo);
    dim3 launchDims(numBlocks, 256, 32768);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::reduce::ReduceFloatFunction, ::execReduceXD(launchDims, stream, opNum, xRank, dX, dXShapeInfo, hXShapeInfo, extraParams, dZ, dZShapeInfo, hZShapeInfo, dimension, dimensionLength, reductionPointer, tadShapeInfo, tadOffsets), LIBND4J_TYPES, FLOAT_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execReduceFloat failed", res);
}


/**
 *
 * @param opNum
 * @param dX
 * @param dXShapeInfo
 * @param extraParams
 */
////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execIndexReduceScalar(nd4j::LaunchContext  *lc,
											int opNum,
											void *hX, Nd4jLong *hXShapeInfo,
        									void *dX, Nd4jLong *dXShapeInfo,
        									void *extraParams,
        									void *hZ, Nd4jLong *hZShapeInfo,
											void *dZ, Nd4jLong *dZShapeInfo){

	if (nd4j::Environment::getInstance()->isDebug())
		printf("F1 opNum:[%i]\n", opNum);

	auto stream = lc->getCudaStream();
	auto reductionPointer = lc->getReductionPointer();
	auto allocationPointer = lc->getAllocationPointer();

    auto xLength = shape::length(hXShapeInfo);
    auto blockWidth = 256;
    auto numBlocks = CudaLaunchHelper::getReductionBlocks(xLength, blockWidth);
    dim3 launchDims(numBlocks, blockWidth, 32768);

	if (nd4j::Environment::getInstance()->isDebugAndVerbose() && launchDims.x == 1)
		printf("AF1 opNum:[%i]\n", opNum);

	auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

    // FIXME: we want Z to be one of integer types
	//if (!DataTypeUtils::isZ(zType))
	//    throw nd4j::datatype_exception("NativeOpExecutioner::execIndexReduceScalar requires Z operand to have one of integer types")
	if (zType != nd4j::DataType::INT64 && zType != nd4j::DataType::INT32)
        throw nd4j::datatype_exception::build("NativeOpExecutioner::execIndexReduceScalar requires Z operand to have INT32/INT64 data type", zType);

    auto dz = reinterpret_cast<Nd4jLong*>(dZ);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::indexreduce::IndexReduce, ::executeIndexReduceScalar(launchDims, stream,
                                                                                                opNum,
                                                                                                dX, dXShapeInfo, shape::rank(hXShapeInfo),
                                                                                                extraParams,
                                                                                                dz, dZShapeInfo, 0,
                                                                                                nullptr, 0,
                                                                                                1,
                                                                                                allocationPointer, reductionPointer,
                                                                                                nullptr, nullptr), LIBND4J_TYPES, INDEXING_TYPES);
    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execIndexReduceScalar failed", res);
}


////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execReduceFloatScalar(nd4j::LaunchContext  *lc,
                                                int opNum,
                                                void *hX, Nd4jLong *hXShapeInfo,
                                                void *dX, Nd4jLong *dXShapeInfo,
                                                void *extraParams,
                                                void *hZ, Nd4jLong *hZShapeInfo,
                                                void *dZ, Nd4jLong *dZShapeInfo) {

    auto stream = lc->getCudaStream();
    auto reductionPointer = lc->getReductionPointer();

    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

    auto xLength = shape::length(hXShapeInfo);
    auto blockWidth = 256;
    auto numBlocks = CudaLaunchHelper::getReductionBlocks(xLength, blockWidth);
    dim3 launchDims(numBlocks, blockWidth, 32768);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::reduce::ReduceFloatFunction, ::execReduceScalar(launchDims, stream, opNum, dX,dXShapeInfo, hXShapeInfo, extraParams, dZ,dZShapeInfo, hZShapeInfo, nullptr, 0, reductionPointer, nullptr), LIBND4J_TYPES, FLOAT_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execReduceFloatScalar failed", res);
}


////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execReduceBoolScalar(nd4j::LaunchContext  *lc,
                                        int opNum,
                                        void *hX, Nd4jLong *hXShapeInfo,
                                        void *dX, Nd4jLong *dXShapeInfo,
                                        void *extraParams,
                                        void *hZ, Nd4jLong *hZShapeInfo,
                                        void *dZ, Nd4jLong *dZShapeInfo) {

    auto stream = lc->getCudaStream();
    auto reductionPointer = lc->getReductionPointer();

    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

    if (zType != nd4j::DataType::BOOL)
        throw std::runtime_error("NativeOpExecutioner::execReduceBoolScalar requires Z operand to have BOOL type");

    auto xLength = shape::length(hXShapeInfo);
    auto blockWidth = 256;
    auto numBlocks = CudaLaunchHelper::getReductionBlocks(xLength, blockWidth);
    dim3 launchDims(numBlocks, blockWidth, 32768);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::reduce::ReduceBoolFunction, ::execReduceScalar(launchDims, stream, opNum, dX, dXShapeInfo, hXShapeInfo, extraParams, dZ, dZShapeInfo, hZShapeInfo, nullptr, 0, reductionPointer, nullptr), LIBND4J_TYPES, BOOL_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execReduceBoolScalar failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execReduceSameScalar(nd4j::LaunchContext  *lc,
                                        int opNum,
                                        void *hX, Nd4jLong *hXShapeInfo,
                                        void *dX, Nd4jLong *dXShapeInfo,
                                        void *extraParams,
                                        void *hZ, Nd4jLong *hZShapeInfo,
                                        void *dZ, Nd4jLong *dZShapeInfo) {

    auto stream = lc->getCudaStream();
    auto reductionPointer = lc->getReductionPointer();

    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

    if (zType != xType)
        throw datatype_exception::build("NativeOpExecutioner::execReduceSameScalar requires both X & Z operands to have same type", xType, zType);

    auto xLength = shape::length(hXShapeInfo);
    auto blockWidth = 256;
    auto numBlocks = CudaLaunchHelper::getReductionBlocks(xLength, blockWidth);
    dim3 launchDims(numBlocks, blockWidth, 32768);

    BUILD_SINGLE_SELECTOR(xType, functions::reduce::ReduceSameFunction, ::execReduceScalar(launchDims, stream, opNum, dX, dXShapeInfo, hXShapeInfo, extraParams, dZ, dZShapeInfo, hZShapeInfo, nullptr, 0, reductionPointer, nullptr), LIBND4J_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execReduceSameScalar failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execReduceLongScalar(nd4j::LaunchContext  *lc,
                                    int opNum,
                                    void *hX, Nd4jLong *hXShapeInfo,
                                    void *dX, Nd4jLong *dXShapeInfo,
                                    void *extraParams,
                                    void *hZ, Nd4jLong *hZShapeInfo,
                                    void *dZ, Nd4jLong *dZShapeInfo) {

    auto stream = lc->getCudaStream();
    auto reductionPointer = lc->getReductionPointer();

    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

    if (zType != nd4j::DataType::INT64)
        throw datatype_exception::build("NativeOpExecutioner::execReduceLongScalar wrong Z data type", nd4j::DataType::INT64, zType);

    auto xLength = shape::length(hXShapeInfo);
    auto blockWidth = 256;
    auto numBlocks = CudaLaunchHelper::getReductionBlocks(xLength, blockWidth);
    dim3 launchDims(numBlocks, blockWidth, 32768);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::reduce::ReduceLongFunction, ::execReduceScalar(launchDims, stream, opNum, dX, dXShapeInfo, hXShapeInfo, extraParams, dZ, dZShapeInfo, hZShapeInfo, nullptr, 0, reductionPointer, nullptr), LIBND4J_TYPES, LONG_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execReduceLongScalar failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execTransformSame(nd4j::LaunchContext  *lc,
									int opNum,
                                   	void *hX, Nd4jLong *hXShapeInfo,
                                   	void *dX, Nd4jLong *dXShapeInfo,
                                   	void *hZ, Nd4jLong *hZShapeInfo,
                                   	void *dZ, Nd4jLong *dZShapeInfo,
                                   	void *extraParams,
                                   	Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets) {

    auto stream = lc->getCudaStream();
    dim3 launchDims(512, 512, 16384);

    auto xRank = shape::rank(hXShapeInfo);
	auto zRank = shape::rank(hZShapeInfo);
	auto xType = ArrayOptions::dataType(hXShapeInfo);
    auto zType = ArrayOptions::dataType(hZShapeInfo);

    if (xType != zType)
        throw std::runtime_error("NativeOpExecutioner::execTransformSame requires X & Z to have same type");

    BUILD_SINGLE_SELECTOR(xType, functions::transform::TransformSame, ::executeTransformShaped(launchDims, stream, opNum, dX, dXShapeInfo, xRank, extraParams, dZ, dZShapeInfo, zRank, nullptr, nullptr, nullptr, nullptr), LIBND4J_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execTransformSame failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execTransformBool(nd4j::LaunchContext  *lc,
                                int opNum,
                                void *hX, Nd4jLong *hXShapeInfo,
                                void *dX, Nd4jLong *dXShapeInfo,
                                void *hZ, Nd4jLong *hZShapeInfo,
                                void *dZ, Nd4jLong *dZShapeInfo,
                                void *extraParams,
                                Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets) {

	auto stream = lc->getCudaStream();
	dim3 launchDims(512, 512, 16384);

	auto xRank = shape::rank(hXShapeInfo);
	auto zRank = shape::rank(hZShapeInfo);
	auto xType = ArrayOptions::dataType(hXShapeInfo);
    auto zType = ArrayOptions::dataType(hZShapeInfo);

    if (!DataTypeUtils::isB(zType))
        throw std::runtime_error("NativeOpExecutioner::execTransformBool requires Z to have same boolean type");

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::transform::TransformBool, ::executeTransformShaped(launchDims, stream, opNum, dX, dXShapeInfo, xRank, extraParams, dZ, dZShapeInfo, zRank, nullptr, nullptr, nullptr, nullptr), LIBND4J_TYPES, BOOL_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execTransformBool failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execTransformAny(nd4j::LaunchContext  *lc,
                                		int opNum,
                                		void *hX, Nd4jLong *hXShapeInfo,
                                		void *dX, Nd4jLong *dXShapeInfo,
                                		void *hZ, Nd4jLong *hZShapeInfo,
                                		void *dZ, Nd4jLong *dZShapeInfo,
                                		void *extraParams,
                                		Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets, bool allowParallelism) {

	auto stream = lc->getCudaStream();

	auto xRank = shape::rank(hXShapeInfo);
	auto zRank = shape::rank(hZShapeInfo);
	auto xType = ArrayOptions::dataType(hXShapeInfo);
	auto zType = ArrayOptions::dataType(hZShapeInfo);

	dim3 launchDims(512, 512, 2048);

	BUILD_DOUBLE_SELECTOR(xType, zType, functions::transform::TransformAny, ::executeTransformShaped(launchDims, stream, opNum, dX, dXShapeInfo, xRank, extraParams, dZ, dZShapeInfo, zRank, nullptr, nullptr, nullptr, nullptr), LIBND4J_TYPES, LIBND4J_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execTransformAny failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execTransformStrict(nd4j::LaunchContext  *lc,
                                    int opNum,
                                    void *hX, Nd4jLong *hXShapeInfo,
                                    void *dX, Nd4jLong *dXShapeInfo,
                                    void *hZ, Nd4jLong *hZShapeInfo,
                                    void *dZ, Nd4jLong *dZShapeInfo,
                                    void *extraParams,
                                    Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets) {

    auto stream = lc->getCudaStream();
    dim3 launchDims(512, 512, 16384);

    auto xRank = shape::rank(hXShapeInfo);
    auto zRank = shape::rank(hZShapeInfo);
    auto xType = ArrayOptions::dataType(hXShapeInfo);
    auto zType = ArrayOptions::dataType(hZShapeInfo);

    if (xType != zType || !DataTypeUtils::isR(xType))
        throw datatype_exception::build("NativeOpExecutioner::execTransformStrict requires X & Z to have same floating point type", xType, zType);

    BUILD_SINGLE_SELECTOR(xType, functions::transform::TransformStrict, ::executeTransformShaped(launchDims, stream, opNum, dX, dXShapeInfo, xRank, extraParams, dZ, dZShapeInfo, zRank, nullptr, nullptr, nullptr, nullptr), FLOAT_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execTransformStrict failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execTransformFloat(nd4j::LaunchContext  *lc,
                                int opNum,
                                void *hX, Nd4jLong *hXShapeInfo,
                                void *dX, Nd4jLong *dXShapeInfo,
                                void *hZ, Nd4jLong *hZShapeInfo,
                                void *dZ, Nd4jLong *dZShapeInfo,
                                void *extraParams,
                                Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets) {

    auto stream = lc->getCudaStream();
    auto reductionPointer = lc->getReductionPointer();

    auto xRank = shape::rank(hXShapeInfo);
    auto zRank = shape::rank(hZShapeInfo);
    auto xType = ArrayOptions::dataType(hXShapeInfo);
    auto zType = ArrayOptions::dataType(hZShapeInfo);

    if (!DataTypeUtils::isR(zType))
        throw datatype_exception::build("NativeOpExecutioner::execTransformFloat requires Z to have floating point type", zType);

    dim3 launchDims(512, 512, 2048);
    BUILD_DOUBLE_SELECTOR(xType, zType, functions::transform::TransformFloat, ::executeTransformShaped(launchDims, stream, opNum, dX, dXShapeInfo, xRank, extraParams, dZ, dZShapeInfo, zRank, nullptr, nullptr, nullptr, nullptr), LIBND4J_TYPES, FLOAT_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execTransformFloat failed", res);
}


////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execSummaryStats(nd4j::LaunchContext  *lc,
                                int opNum,
                                void *hX, Nd4jLong *hXShapeInfo,
                                void *dX, Nd4jLong *dXShapeInfo,
                                void *extraParams,
                                void *hZ, Nd4jLong *hZShapeInfo,
                                void *dZ, Nd4jLong *dZShapeInfo,
                                bool biasCorrected) {

    auto stream = lc->getCudaStream();
    auto reductionPointer = lc->getReductionPointer();

    dim3 launchDims = dim3(256, 256, 32768);

	auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
	auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

    if (!DataTypeUtils::isR(zType))
        throw nd4j::datatype_exception::build("NativeOpExecutioner::execSummaryStats requires Z operand to have floating point data type", zType);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::summarystats::SummaryStatsReduce, ::execSummaryStatsReduce(launchDims, stream, opNum, dX, dXShapeInfo, hXShapeInfo, extraParams, dZ, dZShapeInfo, hZShapeInfo, nullptr, nullptr, biasCorrected, reductionPointer), LIBND4J_TYPES, FLOAT_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execSummaryStats A failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execSummaryStats(nd4j::LaunchContext  *lc,
                                			int opNum,
                                			void *hX, Nd4jLong *hXShapeInfo,
                                			void *dX, Nd4jLong *dXShapeInfo,
                                			void *extraParams,
                                			void *hZ, Nd4jLong *hZShapeInfo,
                                			void *dZ, Nd4jLong *dZShapeInfo,
                                			int *dimension, int dimensionLength,
                                            Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets,
                                			bool biasCorrected) {
	auto stream = lc->getCudaStream();
	auto reductionPointer = lc->getReductionPointer();

    dim3 launchDims = dim3(256, 256, 32768);

	auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
	auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

    if (!DataTypeUtils::isR(zType))
        throw nd4j::datatype_exception::build("NativeOpExecutioner::execSummaryStats requires Z operand to have floating point data type", zType);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::summarystats::SummaryStatsReduce, ::execSummaryStatsReduce(launchDims, stream, opNum, dX, dXShapeInfo, hXShapeInfo, extraParams, dZ, dZShapeInfo, hZShapeInfo, dimension, dimensionLength, tadShapeInfo, tadOffsets, biasCorrected, reductionPointer), LIBND4J_TYPES, FLOAT_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execSummaryStats B failed", res);
}


////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execReduce3(nd4j::LaunchContext  *lc,
                            int opNum,
                            void *hX, Nd4jLong *hXShapeInfo,
                            void *dX, Nd4jLong *dXShapeInfo,
                            void *extraParams,
                            void *hY, Nd4jLong *hYShapeInfo,
                            void *dY, Nd4jLong *dYShapeInfo,
                            void *hZ, Nd4jLong *hZShapeInfo,
                            void *dZ, Nd4jLong *dZShapeInfo) {

	auto stream = lc->getCudaStream();
    auto reductionPointer = lc->getReductionPointer();
	auto allocationPointer = lc->getAllocationPointer();

    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto yType = nd4j::ArrayOptions::dataType(hYShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

    auto blockWidth = 256;
    auto numBlocks = CudaLaunchHelper::getReductionBlocks(shape::length(hXShapeInfo), blockWidth);
    dim3 launchDims(numBlocks, blockWidth, 32768);

    if (xType != yType)
        throw nd4j::datatype_exception::build("NativeOpExecutioner::execReduce3 requires Y operand to have X type", xType, yType);

    if (!DataTypeUtils::isR(zType))
        throw nd4j::datatype_exception::build("NativeOpExecutioner::execReduce3 requires Z operand to have floating point data type", zType);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::reduce3::Reduce3, ::execScalar(launchDims, stream, opNum, dX, dXShapeInfo, dY, dYShapeInfo, extraParams, dZ, dZShapeInfo, allocationPointer, reductionPointer, nullptr), LIBND4J_TYPES, FLOAT_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execReduce3 failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execReduce3(nd4j::LaunchContext  *lc,
                            int opNum,
                            void *hX, Nd4jLong *hXShapeInfo,
                            void *dX, Nd4jLong *dXShapeInfo,
                            void *extraParams,
                            void *hY, Nd4jLong *hYShapeInfo,
                            void *dY, Nd4jLong *dYShapeInfo,
                            void *hZ, Nd4jLong *hZShapeInfo,
                            void *dZ, Nd4jLong *dZShapeInfo,
                            int *dimension, int dimensionLength,
                            Nd4jLong* tadOnlyShapeInfo, Nd4jLong* tadOffsets,
                            Nd4jLong* yTadOnlyShapeInfo, Nd4jLong* yTadOffsets) {

    if(shape::isScalar(hZShapeInfo)) {
        NativeOpExecutioner::execReduce3(lc, opNum, hX, hXShapeInfo, dX, dXShapeInfo, extraParams, hY, hYShapeInfo, dY, dYShapeInfo, hZ, hZShapeInfo, dZ, dZShapeInfo);
        return;
    }

    auto stream = lc->getCudaStream();
    auto allocationPointer = lc->getAllocationPointer();

    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto yType = nd4j::ArrayOptions::dataType(hYShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

     if (xType != yType)
        throw nd4j::datatype_exception::build("NativeOpExecutioner::execReduce3 requires Y operand to have X type", xType, yType);

    if (!DataTypeUtils::isR(zType))
        throw nd4j::datatype_exception::build("NativeOpExecutioner::execReduce3 requires Z operand to have floating point data type", zType);


    auto numBlocks = shape::length(hZShapeInfo);
    dim3 launchDims(numBlocks, 256, 32768);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::reduce3::Reduce3, ::exec(launchDims, stream, opNum,
                                                                    dX, dXShapeInfo,
                                                                    dY, dYShapeInfo,
                                                                    extraParams,
                                                                    dZ, dZShapeInfo,
                                                                    dimension, dimensionLength,
                                                                    1,
                                                                    allocationPointer,
                                                                    tadOnlyShapeInfo, tadOffsets,
                                                                    yTadOnlyShapeInfo, yTadOffsets), LIBND4J_TYPES, FLOAT_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execReduce3 B failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execReduce3Scalar(nd4j::LaunchContext  *lc,
								  int opNum,
                                  void *hX, Nd4jLong *hXShapeInfo,
                                  void *dX, Nd4jLong *dXShapeInfo,
                                  void *extraParams,
                                  void *hY, Nd4jLong *hYShapeInfo,
                                  void *dY, Nd4jLong *dYShapeInfo,
                                  void *hZ, Nd4jLong *hZShapeInfo,
                                  void *dZ, Nd4jLong *dZShapeInfo) {


	auto stream 		   = lc->getCudaStream();
	auto allocationPointer = lc->getAllocationPointer();
	auto reductionPointer  = lc->getReductionPointer();

    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto yType = nd4j::ArrayOptions::dataType(hYShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

    auto xLength = shape::length(hXShapeInfo);
    auto blockWidth = 256;
    auto numBlocks = CudaLaunchHelper::getReductionBlocks(xLength, blockWidth);
    dim3 launchDims(numBlocks, blockWidth, 32768);

    if (xType != yType)
        throw nd4j::datatype_exception::build("NativeOpExecutioner::execReduce3Scalar requires Y operand to have X type", xType, yType);

    if (!DataTypeUtils::isR(zType))
        throw nd4j::datatype_exception::build("NativeOpExecutioner::execReduce3Scalar requires Z operand to have floating point data type", zType);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::reduce3::Reduce3, ::execScalar(launchDims, stream, opNum, dX, dXShapeInfo, dY, dYShapeInfo, extraParams, dZ, dZShapeInfo, allocationPointer, reductionPointer, nullptr), LIBND4J_TYPES, FLOAT_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execReduce3Scalar failed", res);
}


////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execScalarBool(nd4j::LaunchContext  *lc,
										int opNum,
										void *hX, Nd4jLong *hXShapeInfo,
										void *dX, Nd4jLong *dXShapeInfo,
										void *hZ, Nd4jLong *hZShapeInfo,
										void *dZ, Nd4jLong *dZShapeInfo,
										void *hScalar, Nd4jLong *hScalarShapeInfo,
										void *dScalar, Nd4jLong *dScalarShapeInfo,
										void *extraParams, bool allowParallelism) {

	auto stream = lc->getCudaStream();

	dim3 launchDims = dim3(256, 512, 8192);

	auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
	auto yType = nd4j::ArrayOptions::dataType(hScalarShapeInfo);
	auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

	if (xType != yType )
		throw std::runtime_error("NativeOpExecutioner::execScalarBool requires X & Y to have same type");

	if (!DataTypeUtils::isB(zType) )
		throw std::runtime_error("NativeOpExecutioner::execScalarBool requires Z operand to have BOOL type");

	BUILD_DOUBLE_SELECTOR(xType, zType, functions::scalar::ScalarBoolTransform, ::executeCudaShaped(launchDims, stream, opNum, dX, dXShapeInfo, dZ, dZShapeInfo, dScalar, extraParams), LIBND4J_TYPES, BOOL_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execScalarBool failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execScalarBool(nd4j::LaunchContext  *lc,
						   				int opNum,
						   				void *hX, Nd4jLong *hXShapeInfo,
						   				void *dX, Nd4jLong *dXShapeInfo,
                                        void *extraParams,
						   				void *hZ, Nd4jLong *hZShapeInfo,
						   				void *dZ, Nd4jLong *dZShapeInfo,
						   				void *hScalars, Nd4jLong *hScalarShapeInfo,
						   				void *dScalars, Nd4jLong *dScalarShapeInfo,
						   				int *dimension, int dimensionLength,
                           				Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets,
                           				Nd4jLong *tadShapeInfoZ, Nd4jLong *tadOffsetsZ) {

	auto stream = lc->getCudaStream();

	dim3 launchDims(256, 512, 8192);

	auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
	auto yType = nd4j::ArrayOptions::dataType(hScalarShapeInfo);
	auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

	if (xType != yType )
		throw std::runtime_error("NativeOpExecutioner::execScalarBool requires X & Y to have same type");

	if (!DataTypeUtils::isB(zType) )
		throw std::runtime_error("NativeOpExecutioner::execScalarBool requires Z operand to have BOOL type");

	BUILD_DOUBLE_SELECTOR(xType, zType, functions::scalar::ScalarBoolTransform, ::executeCudaAlongDimension(launchDims, stream, opNum, dX, dXShapeInfo, dZ, dZShapeInfo, dScalars, extraParams, dimension, dimensionLength, tadShapeInfo, tadOffsets, tadShapeInfoZ, tadOffsetsZ), LIBND4J_TYPES, BOOL_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execScalarBool B failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execScalarInt(nd4j::LaunchContext  *lc,
                                         int opNum,
                                         void *hX, Nd4jLong *hXShapeInfo,
                                         void *dX, Nd4jLong *dXShapeInfo,
                                         void *hZ, Nd4jLong *hZShapeInfo,
                                         void *dZ, Nd4jLong *dZShapeInfo,
                                         void *hScalar, Nd4jLong *hScalarShapeInfo,
                                         void *dScalar, Nd4jLong *dScalarShapeInfo,
                                         void *extraParams, bool allowParallelism) {

    auto stream = lc->getCudaStream();

    dim3 launchDims = dim3(256, 512, 8192);

    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto yType = nd4j::ArrayOptions::dataType(hScalarShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

    if (xType != yType || zType != xType)
        throw std::runtime_error("NativeOpExecutioner::execScalarInt requires X & Y to have same type");

    if (!DataTypeUtils::isZ(zType) )
        throw std::runtime_error("NativeOpExecutioner::execScalarInt requires Z operand to have INT type");

    BUILD_SINGLE_SELECTOR(xType, functions::scalar::ScalarIntTransform, ::executeCudaShaped(launchDims, stream, opNum, dX, dXShapeInfo, dZ, dZShapeInfo, dScalar, extraParams), INTEGER_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execScalarInt failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execScalarInt(nd4j::LaunchContext  *lc,
                                         int opNum,
                                         void *hX, Nd4jLong *hXShapeInfo,
                                         void *dX, Nd4jLong *dXShapeInfo,
                                         void *extraParams,
                                         void *hZ, Nd4jLong *hZShapeInfo,
                                         void *dZ, Nd4jLong *dZShapeInfo,
                                         void *hScalars, Nd4jLong *hScalarShapeInfo,
                                         void *dScalars, Nd4jLong *dScalarShapeInfo,
                                         int *dimension, int dimensionLength,
                                         Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets,
                                         Nd4jLong *tadShapeInfoZ, Nd4jLong *tadOffsetsZ) {

    auto stream = lc->getCudaStream();

    dim3 launchDims(256, 512, 8192);

    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto yType = nd4j::ArrayOptions::dataType(hScalarShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

    if (xType != yType || zType != xType)
        throw std::runtime_error("NativeOpExecutioner::execScalarInt requires X & Y to have same type");

    if (!DataTypeUtils::isZ(zType) )
        throw std::runtime_error("NativeOpExecutioner::execScalarInt requires Z operand to have INT type");

    BUILD_SINGLE_SELECTOR(xType, functions::scalar::ScalarIntTransform, ::executeCudaAlongDimension(launchDims, stream, opNum, dX, dXShapeInfo, dZ, dZShapeInfo, dScalars, extraParams, dimension, dimensionLength, tadShapeInfo, tadOffsets, tadShapeInfoZ, tadOffsetsZ), INTEGER_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execScalarInt B failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execScalar(nd4j::LaunchContext  *lc,
									int opNum,
									void *hX, Nd4jLong *hXShapeInfo,
									void *dX, Nd4jLong *dXShapeInfo,
									void *hZ, Nd4jLong *hZShapeInfo,
									void *dZ, Nd4jLong *dZShapeInfo,
									void *hScalar, Nd4jLong *hScalarShapeInfo,
									void *dScalar, Nd4jLong *dScalarShapeInfo,
									void *extraParams, bool allowParallelism) {

	auto stream = lc->getCudaStream();

	dim3 launchDims(256, 512, 8192);

	auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
	auto yType = nd4j::ArrayOptions::dataType(hScalarShapeInfo);
	auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);


#ifdef __ND4J_EXPERIMENTAL__
	BUILD_PAIRWISE_SELECTOR(xType, yType, zType, functions::scalar::ScalarTransform, ::executeCudaShaped(launchDims, stream, opNum, dX, dXShapeInfo, hXShapeInfo, dZ, dZShapeInfo, hZShapeInfo, dScalar, extraParams), LIBND4J_TYPES, LIBND4J_TYPES);
#else
	BUILD_SINGLE_SELECTOR_THRICE(xType, functions::scalar::ScalarTransform, ::executeCudaShaped(launchDims, stream, opNum, dX, dXShapeInfo, hXShapeInfo, dZ, dZShapeInfo, hZShapeInfo, dScalar, extraParams), LIBND4J_TYPES);
#endif

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execScalar failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execScalar(nd4j::LaunchContext  *lc,
					 				int opNum,
					 				void *hX, Nd4jLong *hXShapeInfo,
                     				void *dX, Nd4jLong *dXShapeInfo,
                                    void *extraParams,
                     				void *hZ, Nd4jLong *hZShapeInfo,
                     				void *dZ, Nd4jLong *dZShapeInfo,
                     				void *hScalars, Nd4jLong *hScalarShapeInfo,
                     				void *dScalars, Nd4jLong *dScalarShapeInfo,
					 				int *dimension, int dimensionLength,
                     				Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets,
                     				Nd4jLong *tadShapeInfoZ, Nd4jLong *tadOffsetsZ) {

    auto stream = lc->getCudaStream();

    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto yType = nd4j::ArrayOptions::dataType(hScalarShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

	dim3 launchDims(256, 256, 16384);

#ifdef __ND4J_EXPERIMENTAL__
    BUILD_PAIRWISE_SELECTOR(xType, yType, zType, functions::scalar::ScalarTransform, ::executeCudaAlongDimension(launchDims, stream, opNum, dX, dXShapeInfo, dZ, dZShapeInfo, dScalars, extraParams, dimension, dimensionLength, tadShapeInfo, tadOffsets, tadShapeInfoZ, tadOffsetsZ), LIBND4J_TYPES, LIBND4J_TYPES);
#else
	BUILD_SINGLE_SELECTOR_THRICE(xType, functions::scalar::ScalarTransform, ::executeCudaAlongDimension(launchDims, stream, opNum, dX, dXShapeInfo, dZ, dZShapeInfo, dScalars, extraParams, dimension, dimensionLength, tadShapeInfo, tadOffsets, tadShapeInfoZ, tadOffsetsZ), LIBND4J_TYPES);
#endif

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execScalar B failed", res);
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execRandom(nd4j::LaunchContext  *lc,
						  int opNum,
                          Nd4jPointer stateHost,
                          void *hZ, Nd4jLong *hZShapeInfo,
                          void *dZ, Nd4jLong *dZShapeInfo,
                          void *extraArguments) {

    auto stream = lc->getCudaStream();
    auto sizeOf = sizeof(nd4j::graph::RandomGenerator);
    Nd4jPointer stateDevice;

    hipError_t res = hipMalloc(reinterpret_cast<void **>(&stateDevice), sizeOf);
    checkCudaErrors(hipStreamSynchronize(*stream));
    checkCudaErrors(hipMemcpyAsync(stateDevice, stateHost, sizeOf, hipMemcpyHostToDevice, *stream));

    dim3 launchDims = dim3(512, 512, 32768);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

    auto rng = reinterpret_cast<nd4j::graph::RandomGenerator*>(stateHost);

    // functions::random::RandomFunction<float>::executeCudaSingle(launchDims, extraPointers, opNum, stateHost, dZ, dZShapeInfo, extraArguments),
    BUILD_SINGLE_SELECTOR(zType, functions::random::RandomFunction, ::executeCudaSingle(launchDims, stream, opNum, stateDevice, dZ, dZShapeInfo, extraArguments), FLOAT_TYPES);

    res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execRandom X failed", res);

    hipFree(stateDevice);

    rng->rewindH(shape::length(hZShapeInfo));
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execRandom(nd4j::LaunchContext  *lc,
							int opNum,
							Nd4jPointer stateHost,
						   	void *hX, Nd4jLong *hXShapeInfo,
						   	void *dX, Nd4jLong *dXShapeInfo,
						   	void *hZ, Nd4jLong *hZShapeInfo,
						   	void *dZ, Nd4jLong *dZShapeInfo,
						   	void *extraArguments) {

    auto stream = lc->getCudaStream();

    auto sizeOf = sizeof(nd4j::graph::RandomGenerator);
    Nd4jPointer stateDevice;

    hipError_t res = hipMalloc(reinterpret_cast<void **>(&stateDevice), sizeOf);
    checkCudaErrors(hipStreamSynchronize(*stream));
    checkCudaErrors(hipMemcpyAsync(stateDevice, stateHost, sizeOf, hipMemcpyHostToDevice, *stream));

    auto rng = reinterpret_cast<nd4j::graph::RandomGenerator*>(stateHost);

    dim3 launchDims = dim3(512, 512, 32768);
    auto xType = nd4j::ArrayOptions::dataType(hZShapeInfo);
    // functions::random::RandomFunction<float>::executeCudaDouble(launchDims, extraPointers, opNum, stateHost, dX, dXShapeInfo, dZ, dZShapeInfo, extraArguments);
    BUILD_SINGLE_SELECTOR(xType, functions::random::RandomFunction, ::executeCudaDouble(launchDims, stream, opNum, stateDevice, dX, dXShapeInfo, dZ, dZShapeInfo, extraArguments), FLOAT_TYPES);

    res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execRandom XY failed", res);

    hipFree(stateDevice);

    rng->rewindH(shape::length(hZShapeInfo));
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execRandom(nd4j::LaunchContext  *lc,
							int opNum,
							Nd4jPointer stateHost,
							void *hX, Nd4jLong *hXShapeInfo,
							void *dX, Nd4jLong *dXShapeInfo,
							void *hY, Nd4jLong *hYShapeInfo,
							void *dY, Nd4jLong *dYShapeInfo,
							void *hZ, Nd4jLong *hZShapeInfo,
							void *dZ, Nd4jLong *dZShapeInfo,
							void *extraArguments) {

    auto stream = lc->getCudaStream();
    auto sizeOf = sizeof(nd4j::graph::RandomGenerator);
    Nd4jPointer stateDevice;

    hipError_t res = hipMalloc(reinterpret_cast<void **>(&stateDevice), sizeOf);
    checkCudaErrors(hipStreamSynchronize(*stream));
    checkCudaErrors(hipMemcpyAsync(stateDevice, stateHost, sizeOf, hipMemcpyHostToDevice, *stream));

    auto rng = reinterpret_cast<nd4j::graph::RandomGenerator*>(stateHost);

    dim3 launchDims = dim3(512, 512, 32768);
    auto xType = nd4j::ArrayOptions::dataType(hZShapeInfo);
    // functions::random::RandomFunction<float>::executeCudaTriple(launchDims, extraPointers, opNum, stateHost, dX, dXShapeInfo, dY, dYShapeInfo, dZ, dZShapeInfo, extraArguments);
    BUILD_SINGLE_SELECTOR(xType, functions::random::RandomFunction, ::executeCudaTriple(launchDims, stream, opNum, stateDevice, dX, dXShapeInfo, dY, dYShapeInfo, dZ, dZShapeInfo, extraArguments), FLOAT_TYPES);

    res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execRandom XYZ failed", res);

    hipFree(stateDevice);

    rng->rewindH(shape::length(hZShapeInfo));
}

////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execReduce3All(nd4j::LaunchContext  *lc,
									int opNum,
									void *hX, Nd4jLong *hXShapeInfo,
                            		void *dX, Nd4jLong *dXShapeInfo,
                            		void *extraParamsVals,
									void *hY, Nd4jLong *hYShapeInfo,
                            		void *dY, Nd4jLong *dYShapeInfo,
                            		void *hZ, Nd4jLong *hZShapeInfo,
                            		void *dZ, Nd4jLong *dZShapeInfo,
									int *dimension, int dimensionLength,
									Nd4jLong *xTadShapeInfo, Nd4jLong *xOffsets,
									Nd4jLong *yTadShapeInfo, Nd4jLong *yOffsets) {

    auto stream = lc->getCudaStream();
    auto allocationPointer = lc->getAllocationPointer();
	auto reductionPointer  = lc->getReductionPointer();

    if (nd4j::Environment::getInstance()->isDebugAndVerbose())
        printf("D119 opNum:[%i]\n", opNum);

    dim3 launchDims(shape::length(hZShapeInfo), 256, 32768);

    if (nd4j::Environment::getInstance()->isVerbose() && launchDims.x == 1)
        printf("AD119 opNum:[%i]\n", opNum);

    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto yType = nd4j::ArrayOptions::dataType(hYShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

    if (yType != xType)
        throw nd4j::datatype_exception::build("NativeOpExecutioner::execReduce3All both operands must have same data type", xType, yType);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::reduce3::Reduce3, ::execAll(launchDims, stream, opNum, dX, dXShapeInfo, dY, dYShapeInfo, extraParamsVals, dZ, dZShapeInfo, dimension, dimensionLength, 1, allocationPointer, xTadShapeInfo, xOffsets, yTadShapeInfo, yOffsets), LIBND4J_TYPES, FLOAT_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execReduce3All failed", res);
}


////////////////////////////////////////////////////////////////////////
void NativeOpExecutioner::execReduce3TAD(nd4j::LaunchContext  *lc,
                                            int opNum,
                                            void *hX, Nd4jLong *hXShapeInfo,
                                            void *dX, Nd4jLong *dXShapeInfo,
                                            void *extraParams,
                                            void *hY, Nd4jLong *hYShapeInfo,
                                            void *dY, Nd4jLong *dYShapeInfo,
                                            void *hZ, Nd4jLong *hZShapeInfo,
                                            void *dZ, Nd4jLong *dZShapeInfo,
                                            int *dimension, int dimensionLength,
                                            Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets,
                                            Nd4jLong *yTadShapeInfo, Nd4jLong *yTadOffsets) {

    if(shape::isScalar(hZShapeInfo)) {
        NativeOpExecutioner::execReduce3(lc, opNum, hX, hXShapeInfo, dX, dXShapeInfo, extraParams, hY, hYShapeInfo, dY, dYShapeInfo, hZ, hZShapeInfo, dZ, dZShapeInfo);
        return;
    }

    auto stream = lc->getCudaStream();
    auto allocationPointer = lc->getAllocationPointer();

    auto xType = nd4j::ArrayOptions::dataType(hXShapeInfo);
    auto yType = nd4j::ArrayOptions::dataType(hYShapeInfo);
    auto zType = nd4j::ArrayOptions::dataType(hZShapeInfo);

     if (xType != yType)
        throw nd4j::datatype_exception::build("NativeOpExecutioner::execReduce3TAD requires Y operand to have X type", xType, yType);

    if (!DataTypeUtils::isR(zType))
        throw nd4j::datatype_exception::build("NativeOpExecutioner::execReduce3TAD requires Z operand to have floating point data type", zType);

    auto numBlocks = shape::length(hZShapeInfo);
    dim3 launchDims(numBlocks, 256, 32768);

    BUILD_DOUBLE_SELECTOR(xType, zType, functions::reduce3::Reduce3, ::exec(launchDims, stream, opNum, dX, dXShapeInfo, dY, dYShapeInfo, extraParams, dZ, dZShapeInfo, dimension, dimensionLength, 1, allocationPointer, tadShapeInfo, tadOffsets, yTadShapeInfo, yTadOffsets), LIBND4J_TYPES, FLOAT_TYPES);

    // TODO: remove after the release
    auto res = hipStreamSynchronize(*stream);
    if (res != 0)
        throw cuda_exception::build("execReduce3TAD failed", res);
}

