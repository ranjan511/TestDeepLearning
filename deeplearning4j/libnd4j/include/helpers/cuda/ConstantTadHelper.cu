/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author raver119@gmail.com
//

#include "../ConstantTadHelper.h"
#include <TAD.h>
#include <ConstantHelper.h>
#include <AffinityManager.h>
#include <exceptions/cuda_exception.h>
#include <execution/LaunchContext.h>
#include <ShapeUtils.h>

namespace nd4j {
    ConstantTadHelper::ConstantTadHelper() {
        auto numDevices = AffinityManager::numberOfDevices();

        for (int e = 0; e < numDevices; e++) {
            std::map<TadDescriptor, TadPack> pack;
            _cache.emplace_back(pack);
        }
    }

    ConstantTadHelper* ConstantTadHelper::getInstance() {
        if (!_INSTANCE)
            _INSTANCE = new ConstantTadHelper();

        return _INSTANCE;
    }

    TadPack ConstantTadHelper::tadForDimensions(const Nd4jLong *originalShape, int dimension, const bool keepUnitiesInShape) {
        return tadForDimensions(originalShape, &dimension, 1, keepUnitiesInShape);
    }

    TadPack ConstantTadHelper::tadForDimensions(const Nd4jLong *originalShape, const std::vector<int> &dimensions, const bool keepUnitiesInShape) {
        return tadForDimensions(originalShape, const_cast<int *>(dimensions.data()), dimensions.size(), keepUnitiesInShape);
    }

    TadPack ConstantTadHelper::tadForDimensions(const Nd4jLong *originalShape, int* dimensions, int dimLength, const bool keepUnitiesInShape) {
        TadDescriptor tadDescriptor(originalShape, dimensions, dimLength, keepUnitiesInShape);
        return tadForDimensions(tadDescriptor);
    }

    TadPack ConstantTadHelper::tadForDimensions(ShapeDescriptor &descriptor, std::vector<int> &dimensions, const bool keepUnitiesInShape) {
        TadDescriptor tadDescriptor(descriptor, dimensions, keepUnitiesInShape);
        return tadForDimensions(tadDescriptor);
    }

    TadPack ConstantTadHelper::tadForDimensions(TadDescriptor &descriptor) {
        const int deviceId = AffinityManager::currentDeviceId();

        _mutex.lock();

        if (_cache[deviceId].count(descriptor) == 0) {
            const auto shapeInfo = descriptor.originalShape().toShapeInfo();
            const int rank = shape::rank(shapeInfo);
            const std::vector<int> dimsToExclude = ShapeUtils::evalDimsToExclude(rank, descriptor.axis());
            const Nd4jLong numOfSubArrs = ShapeUtils::getNumOfSubArrs(shapeInfo, dimsToExclude);
            const int subArrRank = (rank == dimsToExclude.size() || descriptor.areUnitiesinShape()) ? rank : rank - dimsToExclude.size();

            auto sPtr = new Nd4jLong[shape::shapeInfoLength(subArrRank)];
            auto oPtr = new Nd4jLong[numOfSubArrs];

            if (numOfSubArrs > 0)
                shape::calcSubArrShapeAndOffsets(shapeInfo, numOfSubArrs, dimsToExclude.size(), dimsToExclude.data(), sPtr, oPtr, descriptor.areUnitiesinShape());

            Nd4jPointer soPtr;
            auto res = hipMalloc(reinterpret_cast<void**>(&soPtr),  numOfSubArrs * sizeof(Nd4jLong));
            if (res != 0)
                throw cuda_exception::build("Memory allocation for tadOffsets failed", res);

            res = hipMemcpy(soPtr, oPtr, numOfSubArrs * sizeof(Nd4jLong), hipMemcpyHostToDevice);
            if (res != 0)
                throw cuda_exception::build("tadOffsets copy failed", res);

            auto ssPtr = ConstantHelper::getInstance()->replicatePointer(sPtr, shape::shapeInfoByteLength(subArrRank));

            ConstantDataBuffer shapesBuffer(sPtr, ssPtr, shape::shapeInfoLength(subArrRank) * sizeof(Nd4jLong), DataType::INT64);
            ConstantDataBuffer offsetsBuffer(oPtr, soPtr, numOfSubArrs * sizeof(Nd4jLong), DataType::INT64);

            TadPack t(shapesBuffer, offsetsBuffer, numOfSubArrs);
            _cache[deviceId][descriptor] = t;

            TadPack r = _cache[deviceId][descriptor];
            _mutex.unlock();

            delete[] shapeInfo;

            return r;
        } else {
            TadPack r = _cache[deviceId][descriptor];
            _mutex.unlock();

            return r;
        }
    }

    nd4j::ConstantTadHelper* nd4j::ConstantTadHelper::_INSTANCE = 0;
}