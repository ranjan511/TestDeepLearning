/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author raver119@gmail.com
//

#include <exceptions/cuda_exception.h>
#include <ConstantHelper.h>
#include <DataTypeUtils.h>
#include <execution/LaunchContext.h>
#include <specials.h>
#include <logger.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <execution/AffinityManager.h>

#define CONSTANT_LIMIT 49152

__constant__ char deviceConstantMemory[CONSTANT_LIMIT];

namespace nd4j {
    static void* getConstantSpace() {
        Nd4jPointer dConstAddr;
        auto dZ = hipGetSymbolAddress(reinterpret_cast<void **>(&dConstAddr), deviceConstantMemory);

        if (dZ != 0)
            throw cuda_exception::build("hipGetSymbolAddress(...) failed", dZ);

        return dConstAddr;
    }

    int ConstantHelper::getCurrentDevice() {
        return AffinityManager::currentDeviceId();
    }

    int ConstantHelper::getNumberOfDevices() {
        return AffinityManager::numberOfDevices();
    }


    ConstantHelper::ConstantHelper() {
        auto initialDevice = getCurrentDevice();

        auto numDevices = getNumberOfDevices();
        _devicePointers.resize(numDevices);
        _deviceOffsets.resize(numDevices);
        _cache.resize(numDevices);
        _counters.resize(numDevices);

        // filling all pointers
        for (int e = 0; e < numDevices; e++) {
            auto res = hipSetDevice(e);
            if (res != 0)
                throw cuda_exception::build("hipSetDevice failed", res);
             auto constant = getConstantSpace();

            std::map<ConstantDescriptor, ConstantHolder*> devCache;

            _devicePointers[e] = constant;
            _deviceOffsets[e] = 0;
            _cache[e] = devCache;
            _counters[e] = 0L;
        }

        //
        auto res = hipSetDevice(initialDevice);
        if (res != 0)
            throw cuda_exception::build("Final hipSetDevice failed", res);
    }

    ConstantHelper* ConstantHelper::getInstance() {
        if (!_INSTANCE)
            _INSTANCE = new nd4j::ConstantHelper();

        return _INSTANCE;
    }

    void* ConstantHelper::replicatePointer(void *src, size_t numBytes, memory::Workspace *workspace) {
        _mutex.lock();

        auto deviceId = getCurrentDevice();
        Nd4jPointer constantPtr = nullptr;
        Nd4jLong constantOffset = 0L;
        if (_devicePointers[deviceId] == 0) {
            auto constant = getConstantSpace();

            // filling default ptr, which will be 0 probably
            _devicePointers[deviceId] = constant;
            _deviceOffsets[deviceId] = 0;
            constantPtr = constant;
        } else {
            constantPtr = _devicePointers[deviceId];
            constantOffset = _deviceOffsets[deviceId];
        }

        if (constantOffset + numBytes >= CONSTANT_LIMIT) {
            int8_t *ptr = nullptr;
            ALLOCATE_SPECIAL(ptr, workspace, numBytes, int8_t);
            auto res = hipMemcpy(ptr, src, numBytes, hipMemcpyHostToDevice);
            if (res != 0)
                throw cuda_exception::build("hipMemcpy failed", res);

            _mutex.unlock();
            return ptr;
        } else {
            auto originalBytes = numBytes;
            auto rem = numBytes % 8;
            if (rem != 0)
                numBytes += 8 - rem;

            _deviceOffsets[deviceId] += numBytes;

            auto res = hipMemcpyToSymbol(HIP_SYMBOL(deviceConstantMemory), const_cast<const void *>(src), originalBytes, constantOffset, hipMemcpyHostToDevice);
            if (res != 0)
                throw cuda_exception::build("hipMemcpyToSymbol failed", res);

            _mutex.unlock();
            return reinterpret_cast<int8_t *>(constantPtr) + constantOffset;
        }
    }

    ConstantDataBuffer* ConstantHelper::constantBuffer(const ConstantDescriptor &descriptor, nd4j::DataType dataType) {
        const auto deviceId = getCurrentDevice();

        // all cache modifications are synchronous
        _mutexHolder.lock();

        if (_cache[deviceId].count(descriptor) == 0) {
            _cache[deviceId][descriptor] = new ConstantHolder();
        }
        auto holder = _cache[deviceId][descriptor];

        // release cache lock
        _mutexHolder.unlock();

        ConstantDataBuffer* result;

        // access to this holder instance is synchronous
        holder->mutex()->lock();

        if (holder->hasBuffer(dataType)) {
             result = holder->getConstantDataBuffer(dataType);
        } else {
            auto numBytes = descriptor.length() * DataTypeUtils::sizeOf(dataType);
            auto cbuff = new int8_t[numBytes];
            _counters[deviceId] += numBytes;

            // create buffer with this dtype
            if (descriptor.isFloat()) {
                BUILD_DOUBLE_SELECTOR(nd4j::DataType::DOUBLE, dataType, nd4j::SpecialTypeConverter::convertGeneric, (nullptr, const_cast<double *>(descriptor.floatValues().data()), descriptor.length(), cbuff), (nd4j::DataType::DOUBLE, double), LIBND4J_TYPES);
            } else if (descriptor.isInteger()) {
                BUILD_DOUBLE_SELECTOR(nd4j::DataType::INT64, dataType, nd4j::SpecialTypeConverter::convertGeneric, (nullptr, const_cast<Nd4jLong *>(descriptor.integerValues().data()), descriptor.length(), cbuff), (nd4j::DataType::INT64, Nd4jLong), LIBND4J_TYPES);
            }

            auto dbuff = replicatePointer(cbuff, descriptor.length() * DataTypeUtils::sizeOf(dataType));

            ConstantDataBuffer dataBuffer(cbuff, dbuff, descriptor.length(), DataTypeUtils::sizeOf(dataType));

            holder->addBuffer(dataBuffer, dataType);
            result = holder->getConstantDataBuffer(dataType);
        }
        // release holder lock
        holder->mutex()->unlock();

        return result;
    }

    Nd4jLong ConstantHelper::getCachedAmount(int deviceId) {
        int numDevices = getNumberOfDevices();
        if (deviceId > numDevices || deviceId < 0)
            return 0L;
        else
            return _counters[deviceId];
    }

    nd4j::ConstantHelper* nd4j::ConstantHelper::_INSTANCE = 0;
}