/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
//


#include <hipblas.h>
#include <hipsolver.h>
#include "../cublasHelper.h"
#include <exceptions/cuda_exception.h>
#include <helpers/logger.h>
#include <execution/AffinityManager.h>

namespace nd4j {
    std::mutex CublasHelper::_mutex;

    static void* handle_() {
        auto _handle = new hipblasHandle_t();
        auto status = hipblasCreate(_handle); // initialize CUBLAS context
        if (status != HIPBLAS_STATUS_SUCCESS)
            throw cuda_exception::build("cuBLAS handle creation failed !", status);

        return reinterpret_cast<void *>(_handle);
    }

    static void* solver_() {
        auto cusolverH = new hipsolverHandle_t();
        auto status = hipsolverDnCreate(cusolverH);
        if (status != HIPSOLVER_STATUS_SUCCESS)
            throw cuda_exception::build("cuSolver handle creation failed !", status);

        return cusolverH;
    }

    static void destroyHandle_(void* handle) {
        auto ch = reinterpret_cast<hipblasHandle_t *>(handle);
        auto status = hipblasDestroy(*ch);
        if (status != HIPBLAS_STATUS_SUCCESS)
            throw cuda_exception::build("cuBLAS handle destruction failed !", status);

        delete ch;
    }

    CublasHelper::CublasHelper() {
        //nd4j_printf("Initializing cuBLAS\n","");
        auto numDevices = AffinityManager::numberOfDevices();
        auto currentDevice = AffinityManager::currentDeviceId();
        _cache.resize(numDevices);
        _solvers.resize(numDevices);
        for (int e = 0; e < numDevices; e++) {
            AffinityManager::setCurrentNativeDevice(e);

            _cache[e] = handle_();
            _solvers[e] = solver_();
        }

        // don't forget to restore back original device
        AffinityManager::setCurrentNativeDevice(currentDevice);
    }

    CublasHelper::~CublasHelper() {
        nd4j_printf("Releasing cuBLAS\n","");
        auto numDevices = AffinityManager::numberOfDevices();

        for (int e = 0; e < numDevices; e++)
            destroyHandle_(_cache[e]);
    }

    CublasHelper* CublasHelper::getInstance() {
        _mutex.lock();
        if (!_INSTANCE)
            _INSTANCE = new nd4j::CublasHelper();
        _mutex.unlock();

        return _INSTANCE;
    }

    void* CublasHelper::handle() {
        auto deviceId = AffinityManager::currentDeviceId();
        return handle(deviceId);
    }

    void* CublasHelper::solver() {
        auto deviceId = AffinityManager::currentDeviceId();
        if (deviceId < 0 || deviceId > _solvers.size())
            throw cuda_exception::build("requested deviceId doesn't look valid", deviceId);

        return _solvers[deviceId];
    }

    void* CublasHelper::handle(int deviceId) {
        if (deviceId < 0 || deviceId > _cache.size())
            throw cuda_exception::build("requested deviceId doesn't look valid", deviceId);

        return _cache[deviceId];
    }


    nd4j::CublasHelper* nd4j::CublasHelper::_INSTANCE = 0;
}