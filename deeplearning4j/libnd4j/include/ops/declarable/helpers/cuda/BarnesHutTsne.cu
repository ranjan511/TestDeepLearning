#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author George A. Shulinok <sgazeos@gmail.com>, created on 4/18/2019
//

#include <ops/declarable/helpers/BarnesHutTsne.h>

namespace nd4j {
namespace ops {
namespace helpers {

    static __global__ void countRowsKernel(int* pRowCounts, int const* pRows, int const* pCols, Nd4jLong N) {
        auto start = blockIdx.x * blockDim.x;
        auto step = blockDim.x * gridDim.x;
        for (int n = threadIdx.x + start; n < N; n += step) {
            int begin = pRows[n];//->e<int>(n);
            int end = pRows[n + 1];//rowP->e<int>(n + 1);
            for (int i = begin; i < end; i++) {
                bool present = false;
                for (int m = pRows[pCols[i]]; m < pRows[pCols[i] + 1]; m++)
                    if (pCols[m] == n) {
                        present = true;
                        break;
                    }

                atomicAdd(&pRowCounts[n], 1);

                if (!present)
                    atomicAdd(&pRowCounts[pCols[i]], 1);
            }
        }
    }
    Nd4jLong barnes_row_count(const NDArray* rowP, const NDArray* colP, Nd4jLong N, NDArray& rowCounts) {

        int* pRowCounts = reinterpret_cast<int*>(rowCounts.specialBuffer());
        int const* pRows = reinterpret_cast<int const*>(rowP->getSpecialBuffer());
        int const* pCols = reinterpret_cast<int const*>(colP->getSpecialBuffer());
        auto stream = rowCounts.getContext()->getCudaStream();
        countRowsKernel<<<1, 1, 128, *stream>>>(pRowCounts, pRows, pCols, N);
        NDArray numElementsArr = rowCounts.sumNumber(); //reduceAlongDimension(reduce::Sum, {});
        //rowCounts.printBuffer("Row counts");
        auto numElements = numElementsArr.e<Nd4jLong>(0);
        return numElements;
    }

    static __global__ void fillUpsymRow(int const* pRowCounts, int* symRowP, int N) {

        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = blockDim.x * gridDim.x;

        for (int n = start; n < N + 1; n += step) {
            symRowP[n] = 0;
            for (int i = 0; i < n; i++)
                atomicAdd(&symRowP[n], pRowCounts[i]);
        }

    }

    template <typename T>
    static __global__ void symmetrizeKernel(int const* pRows, int const* pCols, T const* pVals, int* symRowP, int* symColP, int* offset, T* pOutput, int N) {
        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = blockDim.x * gridDim.x;

        for (int n = start; n < N; n += step) {
            int begin = pRows[n];
            int bound = pRows[n + 1];

            for (int i = begin; i < bound; i++) {
                bool present = false;
                int colPI = pCols[i];
                int start = pRows[colPI];
                int end = pRows[colPI + 1];

                //PRAGMA_OMP_PARALLEL_FOR_ARGS(schedule(guided) firstprivate(offset))
                for (int m = start; m < end; m++) {
                    if (pCols[m] == n) {
                        present = true;
                        if (n <= colPI) {
                            symColP[symRowP[n] + offset[n]]        = colPI;
                            symColP[symRowP[colPI] + offset[colPI]] = n;
                            pOutput[symRowP[n] + offset[n]] = pVals[i] + pVals[m];
                            pOutput[symRowP[colPI] + offset[colPI]] = pVals[i] + pVals[m];
                        }
                    }
                }

                // If (colP[i], n) is not present, there is no addition involved
                if (!present) {
                    //int colPI = pCols[i];
                    //if (n <= colPI) {
                    symColP[symRowP[n] + offset[n]] = colPI;
                    symColP[symRowP[pCols[i]] + offset[colPI]] = n;
                    pOutput[symRowP[n] + offset[n]] = pVals[i];
                    pOutput[symRowP[colPI] + offset[colPI]] = pVals[i];
                    //}

                }
                // Update offsets
                if (!present || (present && n <= colPI)) {
                    atomicAdd(&offset[n], 1);

                    if (colPI != n)
                        atomicAdd(&offset[colPI], 1);
                }
            }
        }

    }

    template <typename T>
    static void barnes_symmetrize_(const NDArray* rowP, const NDArray* colP, const NDArray* valP, Nd4jLong N, NDArray* outputRows, NDArray* outputCols, NDArray* outputVals, NDArray* rowCounts) {
        int const* pRows = reinterpret_cast<int const*>(rowP->getSpecialBuffer());
        int* symRowP = reinterpret_cast<int*>(outputRows->specialBuffer());
        int* pRowCounts = reinterpret_cast<int*>(rowCounts->specialBuffer());
        auto stream = outputCols->getContext()->getCudaStream();

        fillUpsymRow<<<1, N, 128, *stream>>>(pRowCounts, symRowP, N);
        outputRows->syncToHost();
//        outputRows->printBuffer("output rows");
        int* symColP = reinterpret_cast<int*>(outputCols->specialBuffer());
//        outputRows->printBuffer("SymRows are");
        int const* pCols = reinterpret_cast<int const*>(colP->getSpecialBuffer());
        T const* pVals = reinterpret_cast<T const*>(valP->getSpecialBuffer());
        T* pOutput = reinterpret_cast<T*>(outputVals->specialBuffer());
        //std::vector<int> rowCountsV = rowCounts->getBufferAsVector<int>();
        auto offsetArr = NDArrayFactory::create<int>('c', {N});
        int* offset = reinterpret_cast<int*>(offsetArr.specialBuffer());
        symmetrizeKernel<T><<<1, 1, 1024, *stream>>>(pRows, pCols, pVals, symRowP, symColP, offset, pOutput, N);
//PRAGMA_OMP_PARALLEL_FOR_SIMD_ARGS(schedule(guided) shared(offset))
    }
    void barnes_symmetrize(const NDArray* rowP, const NDArray* colP, const NDArray* valP, Nd4jLong N, NDArray* outputRows, NDArray* outputCols, NDArray* outputVals, NDArray* rowCounts) {
        BUILD_SINGLE_SELECTOR(valP->dataType(), barnes_symmetrize_, (rowP, colP, valP, N, outputRows, outputCols, outputVals, rowCounts), NUMERIC_TYPES);

        *outputVals /= 2.0;
    }
    BUILD_SINGLE_TEMPLATE(template void barnes_symmetrize_, (const NDArray* rowP, const NDArray* colP, const NDArray* valP, Nd4jLong N, NDArray* outputRows, NDArray* outputCols, NDArray* outputVals, NDArray* rowCounts), NUMERIC_TYPES);
    template <typename T>
    static __global__ void edgeForcesKernel(int const* pRows, int const* pCols, T const* dataP, T const* vals, T* outputP, int N, int colCount, int rowSize) {
//        std::vector<T> buffer(colCount);

        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = blockDim.x * gridDim.x;

        for (int n = start; n < N; n += step) {
            int start = pRows[n];
            int end = pRows[n + 1];
            int shift = n * colCount;
            for (int i = start; i < end; i++) {
                T const* thisSlice = dataP + pCols[i] * colCount;
                T res = 1;

                for (int k = 0; k < colCount; k++) {
                    auto valTemp = dataP[shift + k] - thisSlice[k];//thisSlice[k];
                    res += valTemp * valTemp; // (dataP[shift + k] * dataP[shift + k] - 2 * dataP[shift + k] * thisSlice[k] + thisSlice[k] * thisSlice[k])
                }
                res = vals[i] / res;
                for (int k = 0; k < colCount; k++)
                    math::atomics::nd4j_atomicAdd(&outputP[shift + k], T((dataP[shift + k] - thisSlice[k]) * res));
            }
            //atomicAdd(&shift, colCount);
        }

    }
    template <typename T>
    static void barnes_edge_forces_(const NDArray* rowP, NDArray const* colP, NDArray const* valP, int N, NDArray const* data, NDArray* output) {
        NDArray::prepareSpecialUse({output}, {data, rowP, colP, valP, valP});
        T const* dataP = reinterpret_cast<T const*>(data->getSpecialBuffer());
        T const* vals  = reinterpret_cast<T const*>(valP->getSpecialBuffer());
        T* outputP = reinterpret_cast<T*>(output->specialBuffer());
        int const* pRows = reinterpret_cast<int const*>(rowP->getSpecialBuffer());
        int const* pCols = reinterpret_cast<int const*>(colP->getSpecialBuffer());
        int colCount = data->columns();
        //auto shift = 0;
        auto rowSize = sizeof(T) * colCount;
        auto stream = output->getContext()->getCudaStream();
        edgeForcesKernel<T><<<1, 128, 1024, *stream>>>(pRows, pCols, dataP, vals, outputP, N, colCount, rowSize);
        NDArray::registerSpecialUse({output}, {rowP, colP, valP, data});
    }

    void barnes_edge_forces(const NDArray* rowP, NDArray const* colP, NDArray const* valP, int N, NDArray* output, NDArray const& data) {
        // Loop over all edges in the graph
        BUILD_SINGLE_SELECTOR(output->dataType(), barnes_edge_forces_, (rowP, colP, valP, N, &data, output), FLOAT_TYPES);
    }
    BUILD_SINGLE_TEMPLATE(template void barnes_edge_forces_, (const NDArray* rowP, NDArray const* colP, NDArray const* valP, int N, NDArray const* data, NDArray* output), FLOAT_TYPES);

    template <typename T>
    void barnes_gains_(NDArray* input, NDArray* gradX, NDArray* epsilon, NDArray* output) {
        auto gainsInternal = LAMBDA_TTT(x, grad, eps) {
//            return T((x + 2.) * nd4j::math::nd4j_sign<T,T>(grad) != nd4j::math::nd4j_sign<T,T>(eps)) + T(x * 0.8 * nd4j::math::nd4j_sign<T,T>(grad) != nd4j::math::nd4j_sign<T,T>(eps));
            //return T((x + 2.) * nd4j::math::nd4j_sign<T,T>(grad) == nd4j::math::nd4j_sign<T,T>(eps)) + T(x * 0.8 * nd4j::math::nd4j_sign<T,T>(grad) == nd4j::math::nd4j_sign<T,T>(eps));
            T res = nd4j::math::nd4j_sign<T,T>(grad) != nd4j::math::nd4j_sign<T,T>(eps) ? x + T(.2) : x * T(.8);
            if(res < .01) res = .01;
            return res;
        };

        input->applyTriplewiseLambda(gradX, epsilon, gainsInternal, output);
    }

    void barnes_gains(NDArray* input, NDArray* gradX, NDArray* epsilon, NDArray* output) {
        BUILD_SINGLE_SELECTOR(input->dataType(), barnes_gains_, (input, gradX, epsilon, output), NUMERIC_TYPES);
    }
    BUILD_SINGLE_TEMPLATE(template void barnes_gains_, (NDArray* input, NDArray* gradX, NDArray* epsilon, NDArray* output), NUMERIC_TYPES);

    bool cell_contains(NDArray* corner, NDArray* width, NDArray* point, Nd4jLong dimension) {
        auto  cornerMinusWidth = *corner - *width;
        auto cornerPlusWidth = *corner + *width;
        cornerMinusWidth.syncToHost();
        cornerPlusWidth.syncToHost();
        for (Nd4jLong i = 0; i < dimension; i++) {
            if (cornerMinusWidth.e<double>(i) > point->e<double>(i))
                return false;
            if (cornerPlusWidth.e<double>(i) < point->e<double>(i))
                return false;
        }

        return true;
    }
}
}
}

