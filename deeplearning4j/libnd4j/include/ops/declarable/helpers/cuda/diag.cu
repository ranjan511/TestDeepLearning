#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// Created by GS <sgazeos@gmail.com> on 4/6/2018.
//

#include "ResultSet.h"
#include <ops/declarable/helpers/diag.h>

namespace nd4j {
namespace ops {
namespace helpers {

template <typename T>
static __global__ void diagFunctorKernel(void* outputBuffer, Nd4jLong* outputShape, void const* inputBuffer, Nd4jLong* inputShape, Nd4jLong inputLength) {
    __shared__ T *z;
    __shared__ T const* x;
    __shared__ Nd4jLong outputLength;

    if (threadIdx.x == 0) {
        z = reinterpret_cast<T*>(outputBuffer);
        x = reinterpret_cast<T const*>(inputBuffer);

        outputLength = shape::length(outputShape);
    }
    __syncthreads();

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
    const auto step = gridDim.x * blockDim.x;
    for (int t = tid; t < inputLength; t += step) {
        z[shape::getIndexOffset(t * (inputLength + 1), outputShape, outputLength)] = x[shape::getIndexOffset(t, inputShape, inputLength)]; //tX];
    }

}

    template <typename T>
    static __global__ void diagPartFunctorKernel(void* outputBuffer, Nd4jLong* outputShape, void const* inputBuffer, Nd4jLong* inputShape, Nd4jLong outputLength, Nd4jLong inputLength) {
        __shared__ T *z;
        __shared__ T const* x;

        if (threadIdx.x == 0) {
            z = reinterpret_cast<T*>(outputBuffer);
            x = reinterpret_cast<T const*>(inputBuffer);

        }
        __syncthreads();

        const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
        const auto step = gridDim.x * blockDim.x;
        Nd4jLong i = threadIdx.x * (outputLength + 1);
        for (int t = tid; t < outputLength && i < inputLength; t += step) {
            z[shape::getIndexOffset(t, outputShape, outputLength)] = x[shape::getIndexOffset(i, inputShape, inputLength)]; //tX];
            i += outputLength + 1;
        }
    }

//////////////////////////////////////////////////////////////////////////
// Returns a batched matrix tensor with new batched diagonal values.
// for detailed explanations please take a look on web page: https://www.tensorflow.org/api_docs/python/tf/matrix_set_diag
    template <typename T>
    static void _diagFunctor(nd4j::LaunchContext * context, const NDArray* input, NDArray* output) {
        auto stream = context->getCudaStream();
        auto inputLength = input->lengthOf();
        dim3 launchDims(256, 512, 8192);
        if (!input->isActualOnDeviceSide())
            input->syncToDevice();
        diagFunctorKernel<T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(output->specialBuffer(), output->specialShapeInfo(), input->getSpecialBuffer(), input->getSpecialShapeInfo(), inputLength);
    }

    void diagFunctor(nd4j::LaunchContext * context, const NDArray* input, NDArray* output) {
        auto xType = input->dataType();

        BUILD_SINGLE_SELECTOR(xType, _diagFunctor, (context, input, output), LIBND4J_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template void _diagFunctor, (nd4j::LaunchContext * context, const NDArray* input, NDArray* output);, LIBND4J_TYPES);

    template <typename T>
    void _diagPartFunctor(nd4j::LaunchContext * context, NDArray const* input, NDArray* output) {
        const int outLen = output->lengthOf();
        const int inLen = input->lengthOf();
        auto stream = context->getCudaStream();

        dim3 launchDims(256, 512, 8192);
        if (!input->isActualOnDeviceSide())
            input->syncToDevice();

        diagPartFunctorKernel<T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(output->specialBuffer(), output->specialShapeInfo(), input->getSpecialBuffer(), input->getSpecialShapeInfo(), outLen, inLen);
    }


    void diagPartFunctor(nd4j::LaunchContext * context, NDArray const* input, NDArray* output) {
        auto zType = output->dataType();
        BUILD_SINGLE_SELECTOR(zType, _diagPartFunctor, (context, input, output), NUMERIC_TYPES);

    }

}
}
}