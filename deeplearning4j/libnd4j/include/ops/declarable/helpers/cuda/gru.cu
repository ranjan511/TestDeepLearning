#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com), created on 15.02.2018
//

// implementation of gated Recurrent Unit cell
// (cf. http://arxiv.org/abs/1406.1078).
// Kyunghyun Cho, Bart van Merrienboer, Caglar Gulcehre, Dzmitry Bahdanau, Fethi Bougares, Holger Schwenk, Yoshua Bengio
// "Learning Phrase Representations using RNN Encoder-Decoder for Statistical Machine Translation"


#include<ops/declarable/helpers/gru.h>
#include <ops/declarable/CustomOperations.h>
#include<ops/declarable/helpers/transforms.h>
#include <MmulHelper.h>

namespace nd4j 	  {
namespace ops 	  {
namespace helpers {


//////////////////////////////////////////////////////////////////////////
void gruCell(nd4j::LaunchContext * context, const NDArray* x, const NDArray* hLast, const NDArray* W, const NDArray* Wc,
             const NDArray* b, const NDArray* bc,
             NDArray* r, NDArray* u, NDArray* c, NDArray* h) {

    //Inputs:
    // x        input [bS, iS], iS - input size
    // hLast    previous cell output [bS, nU],  that is at previous time step t-1, nU - number of units
    // W        RU weights - [iS+nU, 2*nU] - reset and update gates
    // Wc       C weights - [iS+nU, nU] - cell gate
    // b        r and u biases, [2*nU] - reset and update gates
    // bc       c biases, [nU] - cell gate

    //Outputs:
    // r        Reset gate output [bS, nU]
    // u        Update gate output [bS, nU]
    // c        Cell gate output [bS, nU]
    // h        current cell output [bS, nU]

    /***************************************************************************************/
    /************************ THIS IS NOT OPTIMAZED CODE ***********************************/
    /** however it is more math-friendly and convenient for backprop formulas derivation) **/

    const int bS  = x->sizeAt(0);
    const int iS = x->sizeAt(1);
    const int nU = hLast->sizeAt(1);

    NDArray Wrx = (*W)({0,iS,     0,nU});       // [iS, nU]
    NDArray Wux = (*W)({0,iS,     nU,2*nU});    // [iS, nU]
    NDArray Wrh = (*W)({iS,iS+nU, 0,nU});       // [nU, nU]
    NDArray Wuh = (*W)({iS,iS+nU, nU,2*nU});    // [nU, nU]

    NDArray Wcx = (*Wc)({0,iS,     0,0});       // reset cell weights    [iS, nU]
    NDArray Wch = (*Wc)({iS,iS+nU, 0,0});       // updates cell weights  [nU, nU]

    NDArray br = (*b)({0,  nU});                // [nU]
    NDArray bu = (*b)({nU, 2*nU});              // [nU]

    // × means matrix multipication
    // * means element-wise product or so called Hadamard product

    // reset gate
    r->assign(mmul(*x, Wrx) + mmul(*hLast, Wrh) + br);         // [bS, iS] × [iS, nU] + [bS, nU] × [nU, nU] + [nU] = [bS, nU]
    r->applyTransform(transform::Sigmoid);

    // update gate
    u->assign(mmul(*x, Wux) + mmul(*hLast, Wuh) + bu);         // [bS, iS] × [iS, nU] + [bS, nU] × [nU, nU] + [nU] = [bS, nU]
    u->applyTransform(transform::Sigmoid);

    // cell gate c = activation(x × Wcx + (r * hlast) × Wch + bc)
    c->assign(mmul(*x, Wcx) + mmul(*r * *hLast, Wch) + *bc);    // [bS, iS] × [iS, nU] + [bS, nU] × [nU, nU] + [nU] = [bS, nU]
    c->applyTransform(transform::Tanh);

    NDArray temp = 1.f - *c * *c;

    // cell output
    h->assign(*u * *hLast + (1.f - *u) * *c);


    /***************************************************************************************/
    /*************** THIS IS MORE OPTIMAZED CODE (should think about concat) ***************/
    /***************************************************************************************/
/*
    //Concat inputs: x + hLast : [bs, iS + nU]
    NDArray xhConcat(x->ordering(), {bS, iS + nU}, x->dataType(), context);  // concat([bs, iS], [bs, nU]) -> [bs, iS + nU]
    helpers::concat(context, {const_cast<NDArray*>(x), const_cast<NDArray*>(hLast)},  xhConcat, {1});

    //mmul for reset and update gates: (x × weight_ux + hLast × weight_xr + b_u)
    auto m = mmul(xhConcat, *W) + *b ;    // [bs, iS+nU] * [iS+nU, 2*nU] = [bs, 2*nU]
    // m += *bru;

    m.applyTransform(transform::Sigmoid);  //sigmoid(rz) and sigmoid(uz)

    r->assign(m({0,0, 0, nU}));
    u->assign(m({0,0, nU, 2*nU}));

    // hLast = hLast * r
    xhConcat({0,0, iS, iS+nU}) *= *r;

    //c = tanh(x × weight_cx + (hLast * r) × weight_cr + b_c)
    MmulHelper::mmul(&xhConcat, Wc, c, 1.0, 0.0);       //c = 1.0 * xhConcat * Wc + 0.0 * c
    *c += *bc;
    c->applyTransform(transform::Tanh);

    //Output: h = (1-u).*c + u .* hPrev
    //auto hResult = (*u) * (*hLast) + (1.0f - *u) * (*c); const_cast<NDArray*>(h)->assign(&hResult);
    u->applyPairwiseTransform(pairwise::Multiply, hLast, h, nullptr);        //h = u * hLast
    auto temp = (1.0f - *u);
    temp *= (*c);
    (*h) += temp;
*/
}

//////////////////////////////////////////////////////////////////////////
void gruTimeLoop(nd4j::LaunchContext * context, const NDArray* x, const NDArray* hLast, const NDArray* Wx, const NDArray* Wh, const NDArray* b, NDArray* h) {

    // x   input [time, bS, iS]
    // hLast  initial cell output (at time step = 0) [bS, nU]
    // Wx  input-to-hidden  weights, [iS, 3*nU]
    // Wh  hidden-to-hidden weights, [nU, 3*nU]
    // b   biases, [3*nU]

    // h is cell outputs at each time step [time, bS, nU]

    const int time = x->sizeAt(0);

    NDArray ht_1(*hLast);

    // loop through time steps
    for (int t = 0; t < time; ++t) {

        auto xt = (*x)({t,t+1, 0,0, 0,0});
        auto ht = (*h)({t,t+1, 0,0, 0,0});

        // helpers::gruCell(&xt, &ht_1, Wx, Wh, b, &ht);
        // ht_1.assign(ht);
    }
}

//////////////////////////////////////////////////////////////////////////
void gruCellBP(nd4j::LaunchContext* context,
              const NDArray* x,    const NDArray* hLast,
              const NDArray* W,    const NDArray* Wc,        const NDArray* b,    const NDArray* bc,
              const NDArray* dLdr, const NDArray* dLdu,      const NDArray* dLdc, const NDArray* dLdh,
                    NDArray* dLdx,       NDArray* dLdhLast,
                    NDArray* dLdW,       NDArray* dLdWc,
                    NDArray* dLdb,       NDArray* dLdbc) {

    //Inputs:
    // x              input [bS, iS]
    // hLast          previous cell output [bS, nU],  that is at previous time step t-1
    // W              weights - [iS+nU, 2*nU] - reset and update gates
    // Wc             C weights - [iS+nU, nU] - cell gate
    // b              r and u biases, [2*nU] - reset and update gates
    // bc             c biases, [nU] - cell gate
    // dLdr           gradient wrt reset gate, [bS, nU]
    // dLdu           gradient wrt update gate, [bS, nU]
    // dLdc           gradient wrt cell state, [bS, nU]
    // dLdh           gradient wrt current cell output, [bS, nU]

    //Outputs:
    // dLdx           gradient wrt x,  [bS, iS],
    // dLdhLast       gradient wrt hLast, [bS, nU]
    // dLdW           gradient wrt W,  [iS+nU, 2*nU]
    // dLdWc          gradient wrt Wc, [iS+nU, nU]
    // dLdb           gradient wrt bru [2*nU]
    // dLdbc          gradient wrt bc  [nU]

    // * means element-wise product or so called Hadamard product
    // × means matrix multiplication

    /************************************************************************************************/
    /******************************* THIS IS NOT OPTIMAZED CODE *************************************/
    /*** aim is to have math-readable code in order to keep track of backprop formulas derivation ***/

    const int bS  = x->sizeAt(0);
    const int iS = x->sizeAt(1);
    const int nU = hLast->sizeAt(1);

    NDArray xT     = x->transpose();            // [iS, bS]
    NDArray hLastT = hLast->transpose();        // [nU, bS]

    NDArray Wrx = (*W)({0,iS,     0,nU});       // [iS, nU]
    NDArray Wux = (*W)({0,iS,     nU,2*nU});    // [iS, nU]
    NDArray Wrh = (*W)({iS,iS+nU, 0,nU});       // [nU, nU]
    NDArray Wuh = (*W)({iS,iS+nU, nU,2*nU});    // [nU, nU]

    NDArray Wcx = (*Wc)({0,iS,     0,0});       // reset cell weights    [iS, nU]
    NDArray Wch = (*Wc)({iS,iS+nU, 0,0});       // updates cell weights  [nU, nU]

    NDArray br = (*b)({0,  nU});                // [nU]
    NDArray bu = (*b)({nU, 2*nU});              // [nU]

    NDArray WrxT = Wrx.transpose();             // [nU, iS]
    NDArray WuxT = Wux.transpose();             // [nU, iS]
    NDArray WrhT = Wrh.transpose();             // [nU, nU]
    NDArray WuhT = Wuh.transpose();             // [nU, nU]

    NDArray WcxT = Wcx.transpose();             // [nU, iS]
    NDArray WchT = Wch.transpose();             // [nU, nU]

    NDArray dLdWrx = (*dLdW)({0,iS,     0,nU});     // [iS, nU]
    NDArray dLdWux = (*dLdW)({0,iS,     nU,2*nU});  // [iS, nU]
    NDArray dLdWrh = (*dLdW)({iS,iS+nU, 0,nU});     // [nU, nU]
    NDArray dLdWuh = (*dLdW)({iS,iS+nU, nU,2*nU});  // [nU, nU]

    NDArray dLdWcx = (*dLdWc)({0,iS,     0,0});     // [iS, nU]
    NDArray dLdWch = (*dLdWc)({iS,iS+nU, 0,0});     // [nU, nU]

    NDArray dLdbr = (*dLdb)({0,  nU});              // [nU]
    NDArray dLdbu = (*dLdb)({nU, 2*nU});            // [nU]


    // ***** feed forward step ***** //

    // reset gate
    NDArray r = mmul(*x, Wrx) + mmul(*hLast, Wrh) + br;         // [bS, iS] × [iS, nU] + [bS, nU] × [nU, nU] + [nU] = [bS, nU]
    r.applyTransform(transform::Sigmoid);

    // update gate
    NDArray u = mmul(*x, Wux) + mmul(*hLast, Wuh) + bu;         // [bS, iS] × [iS, nU] + [bS, nU] × [nU, nU] + [nU] = [bS, nU]
    u.applyTransform(transform::Sigmoid);

    // cell gate c = activation(x×Wcx + (r*hlast)×Wcu + bc)
    NDArray c = mmul(*x, Wcx) + mmul(r * *hLast, Wch) + *bc;    // [bS, iS] × [iS, nU] + [bS, nU] × [nU, nU] + [nU] = [bS, nU]
    c.applyTransform(transform::Tanh);

    // h = (1 - u) * c + u * hPrev


    // ***** back prop step ***** //

    // notations:
    // Zr = x × Wrx + hLast × Wrh + br
    // Zu = x × Wux + hLast × Wuh + bu
    // Sr = sigmoid(Zr)
    // Su = sigmoid(Zu)
    // Zc = x × Wcx + (r * hlast) × Wch + bc


    // dLdx = dLdh * dhdx = dLdh * (dhdu * dudx + dhdc * dcdx) = (dLdh * dhdu) * dudx + (dLdh * dhdc) * dcdx = dLdu * dudx + dLdc * dcdx
    //      = dLdx_u + dLdx_c
    // dLdx_u = dLdu * dudx = dLdu * dudZu * dZudx = |dZudx = ... × WuxT| = (dLdu * dudZu) × WuxT
    // dLdx_c = dLdc * dcdx = dLdc * dcdZc * (dZcdx + dZcdr * drdx) = dLdc * dcdZc * dZcdx + dLdc * dcdZc * dZcdr * drdx = dLdx_c0 + dLdx_c1
    // dLdx_c0 = dLdc * dcdZc * dZcdx = |dZcdx = ... × WcxT| = (dLdc * dcdZc) × WcxT
    // dZcdr = (... * hLast) × WchT
    // dLdc * dcdZc * dZcdr = dLdr = (dLdc * dcdZc * hLast) × WchT
    // drdx = drdZr * dZrdx
    // dZrdx = ... × WrxT
    // dLdx_c1 = dLdc * dcdZc * dZcdr * drdx = dLdr * drdx = (dLdr * drdZr) × WrxT
    // finally dLdx = dLdx_u + dLdx_c0 + dLdx_c1 = (dLdu * dudZu) × WuxT + (dLdc * dcdZc) × WcxT + (dLdr * drdZr) × WrxT


    // dLdhLast    = dLdh * (dhdhLast + dhdu * dudhLast + dhdc * dcdhLast) = dLdh * dhdhLast + dLdu * dudhLast + dLdc * dcdhLast
    //             = dLdhLast_h + dLdhLast_u + dLdhLast_c
    // dLdhLast_h  = dLdh * dhdhLas = dLdh * u
    // dLdhLast_u  = dLdu * dudhLast = |dudhLast = dudZu * dZudhLast , dZudhLast = ... × WuhT| = (dLdu * dudZu) × WuhT
    // dLdhLast_c  = dLdc * dcdhLast  = dLdc * (dcdZc * dZcdhLast + dcdZc * dZcdr * drdhLast) =
    //             = dLdc * dcdZc * dZcdhLast + dLdc * dcdZc * dZcdr * drdhLast =
    //             = dLdc * dcdZc * dZcdhLast + dLdr * drdhLast = dLdhLast_c0 + dLdhLast_c1
    // dLdhLast_c0 = dLdc * dcdZc * dZcdhLast = |dZcdhLast = (... * r) × WchT| = (dLdc * dcdZc * r) × WchT
    // dLdhLast_c1 = dLdr * drdhLast = |drdhLast  = drdZr * dZrdhLast, dZrdhLast = ... × WrhT| = (dLdr * drdZr) × WrhT
    // finally dLdhLast = dLdhLast_h + dLdhLast_u + dLdhLast_c0 + dLdhLast_c1 =
    //                  = dLdh * u + (dLdu * dudZu) × WuhT + (dLdc * dcdZc * r) × WchT + (dLdr * drdZr) × WrhT


    // dLdWrx = dLdh * dhdWrx = (dLdh * dhdc) * dcdWrx = dLdc * dcdZc * dZcdWrx = dLdc * dcdZc * dZcdr * drdWrx =
    //        = dLdc * dcdZc * dZcdr * drdZr * dZrdWrx = dLdr * drdZr * dZrdWrx
    // dZrdWrx = xT × ...
    // finally dLdWrx = xT × (dLdr * drdZr)


    // dLdWrh = dLdh * dhdWrh = (dLdh * dhdc) * dcdWrh = dLdc * dcdZc * dZcdWrh = dLdc * dcdZc * dZcdr * drdWrh =
    //        = dLdc * dcdZc * dZcdr * drdZr * dZrdWrh = dLdr * drdZr * dZrdWrh
    // dZrdWrh = hLastT × ...
    // finally dLdWrh = hLastT × (dLdr * drdZr)


    // dLdWux = dLdh * dhdWux = (dLdh * dhdu) * dudWux = dLdu * dudZu * dZudWux
    // dZudWux = xT × ...
    // dLdu * dudZu * dZudWux = xT × (dLdu * dudZu)


    // dLdWuh = dLdh * dhdWuh = (dLdh * dhdu) * dudWuh = dLdh * dhdu * dudZu * dZudWuh = dLdu * dudZu * dZudWuh
    // dZudWuh = hLastT × ...
    // finally dLdWuh = hLastT × (dLdu * dudZu)


    // dLdWcx = dLdh * dhdWcx = dLdh * dhdc * dcdWcx = (dLdh * dhdc) * dcdZc * dZcdWcx = dLdc * dcdZc * dZcdWcx
    // dZcdWcx = xT × ...
    // finally dLdWcx = xT × (dLdc * dcdZc)


    // dLdWch = dLdh * dhdWch = dLdh * dhdc * dcdWch = (dLdh * dhdc) * dcdZc * dZcdWch = dLdc * dcdZc * dZcdWch
    // dZcdWch = (r*hLast)^T × ...
    // finally dLdWch = (r*hLast)^T × (dLdc * dcdZc)


    // dLdbr = dLdh * dhdbr = (dLdh * dhdc) * dcdbr = dLdc * dcdbr = dLdc * dcdZc * dZcdbr = dLdc * dcdZc * dZcdr * drdbr =
    //       = dLdr * drdZr * dZrdbr
    // dZrdbr = 1
    // finally dLdbr = dLdr * drdZr


    // dLdbu = dLdh * dhdbu = (dLdh * dhdu) * dudbu = dLdu * dudZu * dZudbu
    // dZudbu = 1
    // finally dLdbu = dLdu * dudZu


    // dLdbc = dLdh * dhdbc = (dLdh * dhdc) * dcdbc = dLdc * dcdZc * dZcdbc
    // dZcdbc = 1
    // finally dLdbc = dLdc * dcdZc

    NDArray dhdc  = 1.f - u;           // [bS, nU]
    NDArray dhdu  = *hLast - c;        // [bS, nU]
    NDArray dudZu = u * dhdc;          // [bS, nU]
    NDArray drdZr = r * (1.f - r);     // [bS, nU]
    NDArray dcdZc = 1.f - c * c;       // [bS, nU]
    NDArray dLdZc = *dLdc * dcdZc;     // [bS, nU]
    NDArray dLdZu = *dLdu * dudZu;     // [bS, nU]
    NDArray dLdZr = *dLdr * drdZr;     // [bS, nU]

    // NDArray dLdc  = *dLdh * dhdc;                       // [bS, nU]
    // NDArray dLdu  = *dLdh * dhdu;                       // [bS, nU]
    // NDArray dLdr  = mmul(dLdc * dcdZc * *hLast, WchT);  // [bS, nU]

    dLdx->assign(mmul(dLdZu, WuxT) + mmul(dLdZc, WcxT) + mmul(dLdZr, WrxT));                        // [bS, iS]

    dLdhLast->assign(*dLdh * u + mmul(dLdZu, WuhT) + mmul(dLdZc * r, WchT) + mmul(dLdZr, WrhT));    // [bS, nU]

    dLdWrx.assign(mmul(xT,     dLdZr));     // [iS, bS] × [bS, nU] = [iS, nU]
    dLdWrh.assign(mmul(hLastT, dLdZr));     // [nU, bS] × [bS, nU] = [nU, nU]
    dLdWux.assign(mmul(xT,     dLdZu));     // [iS, bS] × [bS, nU] = [iS, nU]
    dLdWuh.assign(mmul(hLastT, dLdZu));     // [nU, bS] × [bS, nU] = [nU, nU]

    dLdWcx.assign(mmul(xT, dLdZc));                          // [iS, bS] × [bS, nU] = [iS, nU]
    dLdWch.assign(mmul((r * *hLast).transpose(), dLdZc));    // [nU, bS] × [bS, nU] = [nU, nU]

    dLdbr.assign(dLdZr.reduceAlongDims(reduce::Sum, {0}));  // [nU]
    dLdbu.assign(dLdZu.reduceAlongDims(reduce::Sum, {0}));  // [nU]

    dLdbc->assign(dLdZc.reduceAlongDims(reduce::Sum, {0})); // [nU]
}


}
}
}

