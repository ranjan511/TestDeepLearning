#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author raver119@gmail.com
//

#include <ops/declarable/helpers/top_k.h>
#include <MmulHelper.h>
#include <NDArrayFactory.h>
#include <Status.h>
#include <ConstantTadHelper.h>
#include <ShapeUtils.h>

#include <hipsolver.h>
#include <cuda_exception.h>

namespace nd4j {
namespace ops {
namespace helpers {
//    template <typename T>
//    static __device__ void swapRows_(T* matrix, Nd4jLong* shape, int theFirst, int theSecond, Nd4jLong N) {
//        if (theFirst != theSecond) {
//            auto start = threadIdx.x + blockIdx.x * blockDim.x;
//            auto step = blockDim.x * gridDim.x;
//            for (auto i = start; i < N; i += step) {
//                Nd4jLong iCoord1[] = {theFirst, i};
//                Nd4jLong iCoord2[] = {theSecond, i};
//                auto iIndex1 = shape::getOffset(0, shape::shapeOf(shape), shape::stride(shape), iCoord1, 2);
//                auto iIndex2 = shape::getOffset(0, shape::shapeOf(shape), shape::stride(shape), iCoord2, 2);
//                //atomicExch(&matrix[iIndex1], matrix[iIndex2]);
//                T e0 = matrix[iIndex1];
//                T e1 = matrix[iIndex2];
//                matrix[iIndex1] = e0;
//                matrix[iIndex2] = e1;
//            }
//        }
//    }
//    BUILD_SINGLE_TEMPLATE(template void swapRows_, (NDArray* matrix, int theFirst, int theSecond), FLOAT_TYPES);
//
//    void swapRows(NDArray* matrix, int theFirst, int theSecond) {
//        BUILD_SINGLE_SELECTOR(matrix->dataType(), swapRows_, (matrix, theFirst, theSecond), FLOAT_TYPES);
//    }
    template<typename T>
    static __global__ void
    invertKernelLow(void *invertedBuf, Nd4jLong *invertedShape, void *inputBuf, Nd4jLong *inputShape, Nd4jLong n) {
        T *inverted = reinterpret_cast<T *>(invertedBuf);
        T *input = reinterpret_cast<T *>(inputBuf);

        auto start = threadIdx.x + blockIdx.x * blockDim.x;
        auto step = blockDim.x * gridDim.x;

        for (int i = start + 1; i < n; i += step) {
            Nd4jLong pos[] = {i, i - 1};
            Nd4jLong posX[] = {i, i};
            Nd4jLong posY[] = {i - 1, i - 1};
            auto xIndex = shape::getOffset(0, shape::shapeOf(inputShape), shape::stride(inputShape), pos, 2);
            auto dxIndex = shape::getOffset(0, shape::shapeOf(inputShape), shape::stride(inputShape), posX, 2);
            auto dyIndex = shape::getOffset(0, shape::shapeOf(inputShape), shape::stride(inputShape), posY, 2);
            auto zIndex = shape::getOffset(0, shape::shapeOf(invertedShape), shape::stride(invertedShape), pos, 2);
            inverted[zIndex] = -input[xIndex] / (input[dxIndex] * input[dyIndex]);
//            math::atomics::nd4j_atomicAdd(&inverted[zIndex], - input[xIndex] * inverted[iIndex] / input[dIndex]);
        }
    }

    template<typename T>
    static __global__ void
    upvertKernel(void *invertedBuf, Nd4jLong *invertedShape, void *inputBuf, Nd4jLong *inputShape, Nd4jLong n) {
        T *inverted = reinterpret_cast<T *>(invertedBuf);
        T *input = reinterpret_cast<T *>(inputBuf);

        auto start = threadIdx.x + blockIdx.x * blockDim.x;
        auto step = blockDim.x * gridDim.x;

        for (int i = start; i < n; i += step) {
            Nd4jLong pos[] = {i, i};
            auto xIndex = shape::getOffset(0, shape::shapeOf(inputShape), shape::stride(inputShape), pos, 2);
            auto zIndex = shape::getOffset(0, shape::shapeOf(invertedShape), shape::stride(invertedShape), pos, 2);
//            math::atomics::nd4j_atomicDiv(&inverted[zIndex], input[xIndex]);
            inverted[zIndex] /= input[xIndex];
        }
    }

    template<typename T>
    static __global__ void
    upvertKernelUp(void *invertedBuf, Nd4jLong *invertedShape, void *inputBuf, Nd4jLong *inputShape, Nd4jLong n) {

        __shared__ T* inverted;
        __shared__ T* input;
        __shared__ Nd4jLong* inputStride;
        __shared__ Nd4jLong* invertedStride;
        __shared__ Nd4jLong* invertedShapeOf;
        __shared__ Nd4jLong* inputShapeOf;
        if (threadIdx.x == 0) {
            inverted = reinterpret_cast<T *>(invertedBuf);
            input = reinterpret_cast<T *>(inputBuf);
            inputStride = shape::stride(inputShape);
            invertedStride = shape::stride(invertedShape);
            invertedShapeOf = shape::shapeOf(invertedShape);
            inputShapeOf = shape::shapeOf(inputShape);

        }
        __syncthreads();

        auto start = threadIdx.x + blockIdx.x * blockDim.x;
        auto step = blockDim.x * gridDim.x;

        for (int i = start; i < n - 1; i += step) {
            Nd4jLong pos[] = {i, i + 1};
            //Nd4jLong posY[] = {i, i};
            Nd4jLong posX[] = {i + 1, i + 1};
            auto xIndex = shape::getOffset(0, inputShapeOf, shape::stride(inputShape), pos, 2);
//            auto yIndex = shape::getOffset(0, shape::shapeOf(inputShape), shape::stride(inputShape), posY, 2);
//            auto yIndex = shape::getOffset(0, shape::shapeOf(inputShape), shape::stride(inputShape), pos, 2);
            auto iIndex = shape::getOffset(0, invertedShapeOf, invertedStride, posX, 2);
            auto zIndex = shape::getOffset(0, invertedShapeOf, invertedStride, pos, 2);
            math::atomics::nd4j_atomicAdd(&inverted[zIndex], -input[xIndex] * inverted[iIndex]); // / input[yIndex]);
            //inputMatrix->t<T>(i, i + 1) * invertedMatrix->t<T>(i + 1, i + 1) / inputMatrix->t<T>(i, i)
        }
    }

    template<typename T>
    static __global__ void
    invertLowKernel(void *invertedBuf, Nd4jLong *invertedShape, void *inputBuf, Nd4jLong *inputShape, Nd4jLong n) {
        T *inverted = reinterpret_cast<T *>(invertedBuf);
        T *input = reinterpret_cast<T *>(inputBuf);

        for (int i = blockIdx.x + 2; i < n; i += gridDim.x) {
            for (int j = i - 2; j >= 0; --j)
                for (int k = threadIdx.x; k < i; k += blockDim.x) {
                    Nd4jLong posZ[] = {i, j};
                    Nd4jLong posY[] = {k, j};
                    Nd4jLong posX[] = {i, k};
                    Nd4jLong posD[] = {i, i};

                    auto xIndex = shape::getOffset(0, shape::shapeOf(inputShape), shape::stride(inputShape), posX, 2);
                    auto yIndex = shape::getOffset(0, shape::shapeOf(invertedShape), shape::stride(invertedShape), posY,
                                                   2);
                    auto dIndex = shape::getOffset(0, shape::shapeOf(inputShape), shape::stride(inputShape), posD, 2);
                    auto zIndex = shape::getOffset(0, shape::shapeOf(invertedShape), shape::stride(invertedShape), posZ,
                                                   2);
                    math::atomics::nd4j_atomicAdd(&inverted[zIndex], -inverted[yIndex] * input[xIndex] / input[dIndex]);
                }
        }
    }

    template<typename T>
    static __global__ void
    invertUpKernel(void *invertedBuf, Nd4jLong *invertedShape, void *inputBuf, Nd4jLong *inputShape, Nd4jLong n) {
        __shared__ T* inverted;
        __shared__ T* input;
        __shared__ Nd4jLong* inputShapeOf;
        __shared__ Nd4jLong* invertedShapeOf;
        __shared__ Nd4jLong* invertedStrideOf;
        __shared__ Nd4jLong* inputStrideOf;

        if (threadIdx.x == 0) {
            inverted = reinterpret_cast<T *>(invertedBuf);;
            input = reinterpret_cast<T *>(inputBuf);
            inputShapeOf = shape::shapeOf(inputShape);
            invertedShapeOf = shape::shapeOf(invertedShape);
            inputStrideOf = shape::stride(inputShape);
            invertedStrideOf = shape::stride(invertedShape);
        }
        __syncthreads();

        for (int i = (int)n - blockIdx.x - 2; i >= 0; i -= gridDim.x) {
            for (int j = i + 2; j < (int)n; j++)
                for (int k = i + threadIdx.x; k < (int)n; k += blockDim.x) {
                    Nd4jLong posZ[] = {i, j};
                    Nd4jLong posY[] = {k, j};
                    Nd4jLong posX[] = {i, k};
//                    Nd4jLong posD[] = {i, i};

                    auto xIndex = shape::getOffset(0, inputShapeOf, inputStrideOf, posX, 2);
                    auto yIndex = shape::getOffset(0, invertedShapeOf, invertedStrideOf, posY, 2);
                    //                  auto dIndex = shape::getOffset(0, shape::shapeOf(inputShape), shape::stride(inputShape), posD, 2);
                    auto zIndex = shape::getOffset(0, invertedShapeOf, invertedStrideOf, posZ, 2);
                    math::atomics::nd4j_atomicAdd(&inverted[zIndex], -inverted[yIndex] * input[xIndex]);// / input[dIndex]);
//                    printf("(%d, %d) inverted[%lld] = %lf (-inverted[%lld] * input[%lld]\n", blockIdx.x, threadIdx.x, zIndex, inverted[zIndex], yIndex, xIndex);
                }
        }
    }

    template<typename T>
    static void invertLowerMatrix_(LaunchContext *context, NDArray *inputMatrix, NDArray *invertedMatrix) {
        int n = inputMatrix->rows();
        invertedMatrix->setIdentity();

        if (inputMatrix->isIdentityMatrix()) return;

        auto stream = context->getCudaStream();

        // invert main diagonal
        upvertKernel<T><<<1, n, 512, *stream>>>(invertedMatrix->specialBuffer(), invertedMatrix->specialShapeInfo(), inputMatrix->specialBuffer(), inputMatrix->specialShapeInfo(), n);
        // invert the second diagonal
        invertKernelLow<T><<<1, n, 512, *stream>>>(invertedMatrix->specialBuffer(), invertedMatrix->specialShapeInfo(), inputMatrix->specialBuffer(), inputMatrix->specialShapeInfo(), n);
//        invertKernelLow<T><<<1, n, 128, *stream>>>(invertedMatrix->specialBuffer(), invertedMatrix->specialShapeInfo(), inputMatrix->specialBuffer(), inputMatrix->specialShapeInfo(), n);
        invertLowKernel<T><<<n, n, 512, *stream>>>(invertedMatrix->specialBuffer(), invertedMatrix->specialShapeInfo(), inputMatrix->specialBuffer(), inputMatrix->specialShapeInfo(), n);
    }

    void invertLowerMatrix(LaunchContext *context, NDArray *inputMatrix, NDArray *invertedMatrix) {
        NDArray::prepareSpecialUse({invertedMatrix}, {inputMatrix});
        BUILD_SINGLE_SELECTOR(inputMatrix->dataType(), invertLowerMatrix_, (context, inputMatrix, invertedMatrix), FLOAT_NATIVE);
        NDArray::registerSpecialUse({invertedMatrix}, {inputMatrix});
    }

    template<typename T>
    static void invertUpperMatrix_(LaunchContext *context, NDArray* inputMatrix, NDArray* invertedMatrix) {
        int n = inputMatrix->rows();
        invertedMatrix->setIdentity();
        auto stream = context->getCudaStream();
        if (inputMatrix->isIdentityMatrix()) { // the inverse for I is I
            return;
        }

        //upvertKernel<T><<<1, n, 128, *stream>>>(invertedMatrix->specialBuffer(), invertedMatrix->specialShapeInfo(), inputMatrix->specialBuffer(), inputMatrix->specialShapeInfo(), n);
        upvertKernelUp<T><<<1, n, 512, *stream >>>(invertedMatrix->specialBuffer(), invertedMatrix->specialShapeInfo(),
                inputMatrix->specialBuffer(), inputMatrix->specialShapeInfo(), n);
        invertedMatrix->tickWriteDevice();
        invertedMatrix->printIndexedBuffer("Step1 UP inversion");
        invertUpKernel<T><<<n, n, 512, *stream >>>(invertedMatrix->specialBuffer(), invertedMatrix->specialShapeInfo(),inputMatrix->specialBuffer(), inputMatrix->specialShapeInfo(), n);
    }

    void invertUpperMatrix(LaunchContext *context, NDArray *inputMatrix, NDArray *invertedMatrix) {
        NDArray::prepareSpecialUse({invertedMatrix}, {inputMatrix});
        BUILD_SINGLE_SELECTOR(invertedMatrix->dataType(), invertUpperMatrix_, (context, inputMatrix, invertedMatrix), FLOAT_NATIVE);
        NDArray::prepareSpecialUse({invertedMatrix}, {inputMatrix});
    }

//    template <typename T>
//    static __global__ void lupKernel(T* compound, Nd4jLong* compoundShape, T* permutation, Nd4jLong* permutationShape, Nd4jLong rowNum) {
//        int swapCount = 0;
//        for(int i = blockIdx.x; i < rowNum; i += gridDim.x ) {
//            auto pivotValue = T(0.0);
//            auto pivot = -1;
//
//            for(int rowCounter = i; rowCounter < rowNum; rowCounter++ ) {
//                Nd4jLong rowCoord[] = {rowCounter, i};
//                auto rowPos = shape::getOffset(0, shape::shapeOf(compoundShape), shape::stride(compoundShape), rowCoord, 2);
//                if(nd4j::math::nd4j_abs(compound[rowPos]) > pivotValue ) {
//                    pivotValue = nd4j::math::nd4j_abs(compound[rowPos]);
//                    pivot = rowCounter;
//                }
//            }
//
//            if( pivotValue != T(0.0) ) {
//                swapRows_<T>(compound, compoundShape, pivot, i, rowNum);
//                swapRows_<T>(permutation, permutationShape, pivot, i, rowNum);
//                if (pivot != i)
//                    swapCount++;
//
//                for( int j = i + 1; j < rowNum; j++ ) {
//                    Nd4jLong posJIbuf[] = {j, i};
//                    Nd4jLong posIIbuf[] = {i, i};
//                    auto posJI = shape::getOffset(0, shape::shapeOf(compoundShape), shape::stride(compoundShape), posJIbuf, 2);
//                    auto posII = shape::getOffset(0, shape::shapeOf(compoundShape), shape::stride(compoundShape), posIIbuf, 2);
//
//                    compound[posJI] /= compound[posII];
//                    for( int k = i + 1; k < rowNum; k++ ) {
//                        Nd4jLong posJKbuf[] = {j, k};
//                        Nd4jLong posIKbuf[] = {i, k};
//                        auto posJK = shape::getOffset(0, shape::shapeOf(compoundShape), shape::stride(compoundShape), posJKbuf, 2);
//                        auto posIK = shape::getOffset(0, shape::shapeOf(compoundShape), shape::stride(compoundShape), posIKbuf, 2);
//                        T arg = compound[posJI] * compound[posIK];
//                        compound[posJK] -= arg;
//                    }
//                }
//            }
//        }
//    }

//    template <typename T, typename F>
        template<typename T>
        static __global__ void determinantKernel(T *compound, T *result, Nd4jLong len) {
            //F tempRes = result[0];

            auto start = blockIdx.x * blockDim.x + threadIdx.x;
            auto step = blockDim.x * gridDim.x;
            for (auto i = start; i < len; i += step) {
                auto pos = i * len + i; //shape::getOffset(0, shape::shapeOf(shape), shape::stride(shape), di, 2);
                math::atomics::nd4j_atomicMul(&result[0], compound[pos]);
            }
        }

        template<typename T>
        static __global__ void determinantLogKernel(T *compound, T *result, Nd4jLong len) {
//            F tempRes =  (F)result[0];

            auto start = blockIdx.x * blockDim.x + threadIdx.x;
            auto step = blockDim.x * gridDim.x;
            for (auto i = start; i < len; i += step) {
                auto pos = i * len + i; //shape::getOffset(0, shape::shapeOf(shape), shape::stride(shape), di, 2);
                math::atomics::nd4j_atomicAdd(result, math::nd4j_log<T,T>(math::nd4j_abs(compound[pos])));
            }
//            __syncthreads();
//
//            if (threadIdx.x == 0) {
//                result[0] = (T)math::nd4j_log<F,F>(math::nd4j_abs(tempRes));
//            }
        }

        template<typename T, typename F>
        static __global__ void
        fillMatrix(void *output, Nd4jLong *outShape, void *input, Nd4jLong *inputShape, Nd4jLong pos, Nd4jLong rowLen) {
            __shared__
            F *matrix;
            __shared__
            T *inputBuf;
            __shared__
            Nd4jLong inputLen;
            __shared__
            Nd4jLong n2;

            if (threadIdx.x == 0) {
                matrix = reinterpret_cast<F *>(output);
                inputBuf = reinterpret_cast<T *>(input);
                inputLen = shape::length(inputShape);
                n2 = rowLen * rowLen;
            }
            __syncthreads();
            auto start = blockIdx.x * blockDim.x + threadIdx.x;
            auto step = blockDim.x * gridDim.x;

            for (int k = pos + start, j = start; j < n2; k += step, j += step) {
                auto xIndex = shape::getIndexOffset(k, inputShape, inputLen);
                matrix[j] = (F) inputBuf[xIndex];
            }
        }

        template<typename T>
        static __global__ void
        returnMatrix(void *output, Nd4jLong *outputShape, void *input, Nd4jLong *inputShape, Nd4jLong pos,
                     Nd4jLong rowLen) {
            __shared__            T *matrix;
            __shared__            T *outputBuf;
            __shared__            Nd4jLong outputLen;
            __shared__            Nd4jLong n2;

            if (threadIdx.x == 0) {
                matrix = reinterpret_cast<T *>(input);
                outputBuf = reinterpret_cast<T *>(output);
                outputLen = shape::length(inputShape);
                n2 = rowLen * rowLen;
            }
            __syncthreads();
            auto start = blockIdx.x * blockDim.x + threadIdx.x;
            auto step = blockDim.x * gridDim.x;

            for (int k = pos + start, j = start; j < n2; k += step, j += step) {
                auto zIndex = shape::getIndexOffset(k, outputShape, outputLen);
                outputBuf[zIndex] = (T) matrix[j];
            }
        }

        template<typename F>
        static __global__ void fillUpPermutation(void *output, Nd4jLong *shape, int *source, int rowNum) {
            F *permutation = reinterpret_cast<F *>(output);

            auto start = blockIdx.x * blockDim.x + threadIdx.x;
            auto step = blockDim.x * gridDim.x;
            for (auto i = start; i < rowNum; i += step) {
                int val = source[i] - 1;
                Nd4jLong posF[] = {i, val};
                auto pos = shape::getOffset(0, shape::shapeOf(shape), shape::stride(shape), posF, 2);
                permutation[pos] = F(1.f);
            }
        }

        template<typename T>
        static void lup_(LaunchContext *context, NDArray *input, NDArray *compound, NDArray *permutation) {
            auto stream = context->getCudaStream();
            auto n = input->rows();
            hipsolverHandle_t cusolverH = nullptr;
            hipsolverStatus_t status = hipsolverDnCreate(&cusolverH);
            if (HIPSOLVER_STATUS_SUCCESS != status) {
                throw cuda_exception::build("Cannot create cuSolver handle", status);
            }
            status = hipsolverSetStream(cusolverH, *stream);
            if (HIPSOLVER_STATUS_SUCCESS != status) {
                throw cuda_exception::build("Cannot set up stream for cuda solver", status);
            }
            int lwork = 0;
            int *d_info = nullptr;

            auto err = hipMalloc((void **) &d_info, sizeof(int));
            if (err) {
                throw cuda_exception::build("helpers::lup_: Cannot allocate memory for solver info buffer", err);
            }

            DataType dtype = input->dataType();
            switch (dtype) {

                case DataType::DOUBLE: {
                    double *d_work = nullptr;
                    err = hipMalloc((void **) &d_work, sizeof(float) * lwork);
                    if (err) {
                        throw cuda_exception::build("helpers::lup_: Cannot allocate memory for solver data buffer",
                                                    err);
                    }
                    double *matrix = reinterpret_cast<double *>(input->specialBuffer());
                    status = hipsolverDnDgetrf_bufferSize(
                            cusolverH,
                            n,
                            n,
                            matrix,
                            n,
                            &lwork);
                    if (HIPSOLVER_STATUS_SUCCESS != status) {
                        throw cuda_exception::build("helpers::lup_: Cannot create cuSolver handle", status);
                    }
                    if (permutation == nullptr)
                        status = hipsolverDnDgetrf(
                                cusolverH,
                                n,
                                n,
                                matrix,
                                n,
                                d_work,
                                nullptr,
                                d_info);
                    else {
                        NDArray permutVector('c', {n}, nd4j::DataType::INT32, context);
                        int *permutationBuf = reinterpret_cast<int *>(permutVector.specialBuffer());
                        status = hipsolverDnDgetrf(
                                cusolverH,
                                n,
                                n,
                                matrix,
                                n,
                                d_work,
                                permutationBuf,
                                d_info);
                        fillUpPermutation<double> << < n, n, 1024, *stream >> >
                                                                  (permutation->specialBuffer(), permutation->specialShapeInfo(), permutationBuf, n);
                        permutation->tickWriteDevice();
                    }
                    err = hipFree(d_work);
                    if (err) {
                        throw cuda_exception::build("helpers::lup_: Cannot deallocate memory for solver data buffer",
                                                    err);
                    }
                }
                    break;
                case DataType::FLOAT32: {
                    float *matrix = reinterpret_cast<float *>(input->specialBuffer());
                    float *d_work = nullptr;
                    err = hipMalloc((void **) &d_work, sizeof(float) * lwork);
                    if (err) {
                        throw cuda_exception::build("helpers::lup_: Cannot allocate memory for solver data buffer",
                                                    err);
                    }

                    status = hipsolverDnSgetrf_bufferSize(
                            cusolverH,
                            n,
                            n,
                            matrix,
                            n,
                            &lwork);
                    if (HIPSOLVER_STATUS_SUCCESS != status) {
                        throw cuda_exception::build("helpers::lup_: Cannot create cuSolver handle", status);
                    }

                    if (permutation == nullptr)
                        status = hipsolverDnSgetrf(
                                cusolverH,
                                n,
                                n,
                                matrix,
                                n,
                                d_work,
                                nullptr,
                                d_info);
                    else {
                        NDArray permutVector('c', {n}, nd4j::DataType::INT32, context);
                        int *permutationBuf = reinterpret_cast<int *>(permutVector.specialBuffer());
                        status = hipsolverDnSgetrf(
                                cusolverH,
                                n,
                                n,
                                matrix,
                                n,
                                d_work,
                                permutationBuf,
                                d_info);
                        fillUpPermutation<T> <<< n, n, 128, *stream >> >
                                                             (permutation->specialBuffer(), permutation->specialShapeInfo(), permutationBuf, n);
                        permutation->tickWriteDevice();
                    }
                    err = hipFree(d_work);
                    if (err) {
                        throw cuda_exception::build("helpers::lup_: Cannot deallocate memory for solver data buffer",
                                                    err);
                    }

                }
            }
            if (HIPSOLVER_STATUS_SUCCESS != status) {
                throw cuda_exception::build("helpers::lup_: Cannot make LU decomposition", status);
            }
            err = hipFree(d_info);
            if (err) {
                throw cuda_exception::build("helpers::lup_: Cannot deallocate memory for solver info buffer", err);
            }
            hipsolverDnDestroy(cusolverH);
//        NDArray::registerSpecialUse({input}, {input});
            input->tickWriteDevice();
        }

        BUILD_SINGLE_TEMPLATE(template void lup_,(LaunchContext * context, NDArray * input, NDArray * output, NDArray * permutation), FLOAT_NATIVE);

        template<typename T>
        static int determinant_(nd4j::LaunchContext *context, NDArray *input, NDArray *output) {
            Nd4jLong n = input->sizeAt(-1);
            Nd4jLong n2 = n * n;
            std::vector<int> dims();
            auto packX = ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), {input->rankOf() - 2, input->rankOf() - 1});
            //auto packZ = ConstantTadHelper::getInstance()->tadForDimensions(output->shapeInfo(), {output->rankOf() - 1});
//        DataType dtype = input->dataType();
//        if (dtype != DataType::DOUBLE)
//            dtype = DataType::FLOAT32;
            auto matrix = NDArrayFactory::create(input->ordering(), {n, n}, DataTypeUtils::fromT<T>(), context); //, block.getWorkspace());
            auto det = NDArrayFactory::create<T>(1);
            auto stream = context->getCudaStream();
            NDArray::prepareSpecialUse({output}, {input});
            dim3 launchDims(256, 256, 1024);
            output->assign(1.f);
            for (int e = 0; e < output->lengthOf(); e++) {
                Nd4jLong pos = e * n2;
//            if (matrix.dataType() == input->dataType())
                fillMatrix<T, T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(matrix.specialBuffer(), matrix.specialShapeInfo(), input->specialBuffer(), input->specialShapeInfo(), pos, n);
//            else
//                fillMatrix<T, float><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(matrix.specialBuffer(), matrix.specialShapeInfo(), input->specialBuffer(), input->specialShapeInfo(), pos, n);

//            if (matrix.dataType() == input->dataType())
                lup_<T>(context, &matrix, nullptr, nullptr);
//            else
//                lup_<float>(context, &matrix, nullptr, nullptr);
                auto offset = shape::getIndexOffset(e, output->shapeInfo(), output->lengthOf());
                auto inputBuf = reinterpret_cast<T *>(matrix.specialBuffer());
                auto outputBuf = reinterpret_cast<T *>(output->specialBuffer()) + offset;
//            if (matrix.dataType() == input->dataType())
                determinantKernel<T> << < launchDims.x, launchDims.y, launchDims.z, *stream >> >
                                                                                    (inputBuf, outputBuf, n);
//            else
//                determinantKernel<T, float><<<launchDims.x, launchDims.y, launchDims.z, *stream >>> (inputBuf, outputBuf, n);
            }
            NDArray::registerSpecialUse({output}, {input});

            return Status::OK();
        }

        int determinant(nd4j::LaunchContext *context, NDArray *input, NDArray *output) {
            NDArray::prepareSpecialUse({output}, {input});
            BUILD_SINGLE_SELECTOR(input->dataType(), return determinant_, (context, input, output), FLOAT_NATIVE);
            NDArray::registerSpecialUse({output}, {input});
        }

        template<typename T>
        int logAbsDeterminant_(LaunchContext *context, NDArray *input, NDArray *output) {
            Nd4jLong n = input->sizeAt(-1);
            Nd4jLong n2 = n * n;
            std::vector<int> dims();
            auto packX = ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), {input->rankOf() - 2, input->rankOf() - 1});
            //auto packZ = ConstantTadHelper::getInstance()->tadForDimensions(output->shapeInfo(), {output->rankOf() - 1});
            DataType dtype = input->dataType();
            if (dtype != DataType::DOUBLE)
                dtype = DataType::FLOAT32;

            auto matrix = NDArrayFactory::create(input->ordering(), {n, n}, dtype, context); //, block.getWorkspace());
            auto det = NDArrayFactory::create<T>(1);
            auto stream = context->getCudaStream();
            NDArray::prepareSpecialUse({output}, {input});
            dim3 launchDims(256, 256, 1024);
            output->assign(0.f);
            for (int e = 0; e < output->lengthOf(); e++) {
                Nd4jLong pos = e * n2;
//            if (matrix.dataType() == input->dataType())
                fillMatrix<T, T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(matrix.specialBuffer(), matrix.specialShapeInfo(), input->specialBuffer(), input->specialShapeInfo(), pos, n);
//            else
//                fillMatrix<T, float><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(matrix.specialBuffer(), matrix.specialShapeInfo(), input->specialBuffer(), input->specialShapeInfo(), pos, n);

//            if (matrix.dataType() == input->dataType())
                lup_<T>(context, &matrix, nullptr, nullptr);
//            else
//                lup_<float>(context, &matrix, nullptr, nullptr);
                auto offset = shape::getIndexOffset(e, output->shapeInfo(), output->lengthOf());
                auto inputBuf = reinterpret_cast<T *>(matrix.specialBuffer());
                auto outputBuf = reinterpret_cast<T *>(output->specialBuffer()) + offset;
//            if (matrix.dataType() == input->dataType())
                determinantLogKernel<T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(inputBuf, outputBuf, n);
//            else
//                determinantLogKernel<T, float><<<launchDims.x, launchDims.y, launchDims.z, *stream >>> (inputBuf, outputBuf, n);
            }
            NDArray::registerSpecialUse({output}, {input});

            return Status::OK();

            return ND4J_STATUS_OK;
        }

        int logAbsDeterminant(nd4j::LaunchContext *context, NDArray *input, NDArray *output) {
            NDArray::prepareSpecialUse({output}, {input});
            BUILD_SINGLE_SELECTOR(input->dataType(), return logAbsDeterminant_, (context, input, output), FLOAT_NATIVE);
            NDArray::registerSpecialUse({output}, {input});
        }

        template<typename T>
        static __global__ void
        fillLowerUpperKernel(void *lowerBuf, Nd4jLong *lowerShape, void *upperBuf, Nd4jLong *upperShape,
                             void *matrixBuf, Nd4jLong *matrixShape, Nd4jLong n) {

            __shared__
            Nd4jLong *xShapeOf;
            __shared__
            Nd4jLong *yShapeOf;
            __shared__
            Nd4jLong *zShapeOf;
            __shared__
            Nd4jLong *xStrideOf;
            __shared__
            Nd4jLong *yStrideOf;
            __shared__
            Nd4jLong *zStrideOf;
            __shared__
            T *lowerMatrix;
            __shared__
            T *upperMatrix;
            __shared__
            T *matrix;

            if (threadIdx.x == 0) {
                xShapeOf = shape::shapeOf(lowerShape);
                xStrideOf = shape::stride(lowerShape);

                yShapeOf = shape::shapeOf(upperShape);
                yStrideOf = shape::stride(upperShape);

                zShapeOf = shape::shapeOf(matrixShape);
                zStrideOf = shape::stride(matrixShape);
                lowerMatrix = reinterpret_cast<T *>(lowerBuf);
                upperMatrix = reinterpret_cast<T *>(upperBuf);
                matrix = reinterpret_cast<T *>(matrixBuf);
            }
            __syncthreads();

            for (int k = blockIdx.x; k < n; k += gridDim.x) {  // and then put all values under main diagonal on to it
                for (int j = threadIdx.x; j < n; j += blockDim.x) {
                    Nd4jLong posX[] = {k, j};
                    Nd4jLong posD[] = {j, j};
                    auto xPos = shape::getOffset(0, xShapeOf, xStrideOf, posX, 2);
                    auto yPos = shape::getOffset(0, yShapeOf, yStrideOf, posX, 2);
                    auto iPos = shape::getOffset(0, zShapeOf, zStrideOf, posX, 2);
                    auto dPos = shape::getOffset(0, zShapeOf, zStrideOf, posD, 2);
                    if (k >= j)
                        lowerMatrix[xPos] = matrix[iPos];//(k, j);
                    else
                        upperMatrix[yPos] = matrix[iPos]; //k, j);
                }
            }
        }

        template<typename T>
        static int inverse_(nd4j::LaunchContext *context, NDArray *input, NDArray *output) {
            auto n = input->sizeAt(-1);
            auto n2 = n * n;
            auto dtype = DataTypeUtils::fromT<T>(); //input->dataType();
//            if (dtype != DataType::DOUBLE)
//                dtype = DataType::FLOAT32;
            NDArray matrix = NDArrayFactory::create('c', {n, n}, dtype, context);
            NDArray upper = NDArrayFactory::create('c', {n, n}, dtype, context);
            NDArray lower = NDArrayFactory::create('c', {n, n}, dtype, context);
            NDArray compound = NDArrayFactory::create('c', {n, n}, dtype, context);
            NDArray permutation = NDArrayFactory::create('c', {n, n}, dtype, context);
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(),
                                                                                  {input->rankOf() - 2,
                                                                                   input->rankOf() - 1});
            auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(),
                                                                                  {output->rankOf() - 2,
                                                                                   output->rankOf() - 1});
            auto stream = context->getCudaStream();

            for (auto i = 0LL; i < packX.numberOfTads(); i++) {
                fillMatrix<T, T><<<1, n2, 1024, *stream>>>(matrix.specialBuffer(), matrix.specialShapeInfo(), input->specialBuffer(), input->specialShapeInfo(), i * n2, n);
                matrix.tickWriteDevice();
                compound.assign(matrix);
                lup_<T>(context, &compound, nullptr, nullptr);
                fillLowerUpperKernel<T><<<n, n, 1024, *stream>>>(lower.specialBuffer(), lower.specialShapeInfo(), upper.specialBuffer(), upper.specialShapeInfo(), compound.specialBuffer(), compound.specialShapeInfo(), n);
                matrix.assign(0);
                invertUpperMatrix(context, &upper, &matrix); // U^{-1}
                matrix.tickWriteDevice();
//                matrix.printIndexedBuffer("Upper Inverted");
                compound.assign(0);
                invertLowerMatrix(context, &lower, &compound); // L{-1}
                compound.tickWriteDevice();
//                compound.printIndexedBuffer("Lower Inverted");
//                matrix.tickWriteDevice();
//                compound.tickWriteDevice();
                nd4j::MmulHelper::mmul(&matrix, &compound, &upper, 1.0, 0.0);
                upper.tickWriteDevice();
//                upper.printIndexedBuffer("Full inverted");
                returnMatrix<T> <<<1, n2, 1024, *stream>>>(output->specialBuffer(), output->specialShapeInfo(), upper.specialBuffer(), upper.specialShapeInfo(), i * n2, n);
            }
            return Status::OK();
        }

        int inverse(nd4j::LaunchContext *context, NDArray *input, NDArray *output) {
            NDArray::prepareSpecialUse({output}, {input});
            BUILD_SINGLE_SELECTOR(input->dataType(), return inverse_, (context, input, output), FLOAT_NATIVE);
            NDArray::registerSpecialUse({output}, {input});
        }

        bool checkCholeskyInput(nd4j::LaunchContext *context, NDArray const *input) {
            return true;
        }

        template<typename F>
        __global__ void fillBatchKernel(F **dArrayBatch, F *buf, Nd4jLong *offsets, Nd4jLong batchSize) {
            auto start = blockIdx.x * blockDim.x + threadIdx.x;
            auto step = blockDim.x * gridDim.x;

            for (auto i = start; i < batchSize; i += step) {
                dArrayBatch[i] = buf + offsets[i];
            }
        }

        template<typename F>
        __global__ void
        adjustResultsKernel(F *dArray, Nd4jLong *shape, Nd4jLong *offsets, Nd4jLong batchSize, Nd4jLong n) {
            //auto i = blockIdx.x * blockDim.x + threadIdx.x;
            Nd4jLong *shapeOf = shape::shapeOf(shape);
            Nd4jLong *strideOf = shape::stride(shape);

            for (auto i = blockIdx.x; i < batchSize; i += gridDim.x) {
                auto current = dArray + offsets[i];
                for (auto r = threadIdx.x; r < n; r += blockDim.x) {
                    for (auto c = r + 1; c < n; c++) {
                        Nd4jLong posRC[] = {r, c};
                        auto pos = r * n + c; //shape::getOffset(0, shapeOf, strideOf, posRC, 2);
                        current[pos] = 0.;
                    }
                }
            }
        }

        template<typename F>
        int cholesky__(LaunchContext *context, NDArray *input, NDArray *output, bool inplace) {
            if (!inplace)
                output->assign(input);
            std::unique_ptr<NDArray> tempOutput(output->dup());
            hipsolverHandle_t handle = nullptr;
            auto n = input->sizeAt(-1);
            auto n2 = n * n;
            NDArray::prepareSpecialUse({output}, {input});
            auto status = hipsolverDnCreate(&handle);
            if (HIPSOLVER_STATUS_SUCCESS != status) {
                throw cuda_exception::build("helpers::cholesky_: Cannot create solver handle", status);
            }
            F **dArrayBatch = nullptr;
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(tempOutput->getShapeInfo(),
                                                                                  {tempOutput->rankOf() - 2,
                                                                                   tempOutput->rankOf() - 1});
            const Nd4jLong batchSize = packX.numberOfTads();
            int *dInfoArray = nullptr;
            auto err = hipMalloc((void **) &dArrayBatch, sizeof(F *) * batchSize);
            if (err) {
                throw cuda_exception::build("helpers::cholesky_: Cannot allocate memory for solver batch data buffer",
                                            err);
            }
            err = hipMalloc((void **) &dInfoArray, sizeof(int) * batchSize);
            if (err) {
                throw cuda_exception::build("helpers::cholesky_: Cannot allocate memory for solver errors buffer", err);
            }
            auto stream = context->getCudaStream();
            fillBatchKernel<F> << < 1, batchSize, 128, *stream >> >
                                                       (dArrayBatch, reinterpret_cast<F *>(tempOutput->specialBuffer()), packX.specialOffsets(), batchSize);

            status = hipsolverSetStream(handle, *stream);
            if (HIPSOLVER_STATUS_SUCCESS != status) {
                throw cuda_exception::build("helpers::cholesky_: Cannot set stream to solver handle", status);
            }
            const hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_UPPER;
            if (input->dataType() == DataType::DOUBLE)
                status = hipsolverDnDpotrfBatched(
                        handle,
                        uplo,
                        n,
                        (double **) dArrayBatch,
                        n,
                        dInfoArray,
                        batchSize);
            else
                status = hipsolverDnSpotrfBatched(
                        handle,
                        uplo,
                        n,
                        (float **) dArrayBatch,
                        n,
                        dInfoArray,
                        batchSize);

            if (HIPSOLVER_STATUS_SUCCESS != status) {
                throw cuda_exception::build("helpers::cholesky_: Cholesky factorization failed for batch", status);
            }
            adjustResultsKernel<F> << < batchSize, n2, 128, *stream >> >
                                                            (reinterpret_cast<F *>(tempOutput->specialBuffer()), packX.specialShapeInfo(), packX.specialOffsets(), batchSize, n);

            err = hipFree(dArrayBatch);
            if (err) {
                throw cuda_exception::build("helpers::cholesky_: Cannot deallocate memory for solver batch data buffer",
                                            err);
            }
            err = hipFree(dInfoArray);
            if (err) {
                throw cuda_exception::build("helpers::cholesky_: Cannot allocate memory for solver errors buffer", err);
            }

            if (!inplace)
                output->assign(tempOutput.get());
            else
                input->assign(tempOutput.get());

            NDArray::registerSpecialUse({output}, {input});
            return Status::OK();
        }

//    template <typename T>
        int cholesky_(LaunchContext *context, NDArray *input, NDArray *output, bool inplace) {
            NDArray::prepareSpecialUse({output}, {input});
            if (input->dataType() == DataType::DOUBLE)
                cholesky__<double>(context, input, output, inplace);
            else if (input->dataType() == DataType::FLOAT32)
                cholesky__<float>(context, input, output, inplace);
            else {
                std::unique_ptr<NDArray> tempOutput(
                        NDArrayFactory::create_('c', input->getShapeAsVector(), DataType::FLOAT32, context));
                tempOutput->assign(input);
                cholesky__<float>(context, tempOutput.get(), tempOutput.get(), true);
                output->assign(tempOutput.get());
            }
            NDArray::registerSpecialUse({output}, {input});
            return Status::OK();
        }

        int cholesky(nd4j::LaunchContext *context, NDArray *input, NDArray *output, bool inplace) {
//        BUILD_SINGLE_SELECTOR(input->dataType(), return cholesky_, (context, input, output, inplace), FLOAT_TYPES);
            return cholesky_(context, input, output, inplace);
        }
//    BUILD_SINGLE_TEMPLATE(template int cholesky_, (LaunchContext* context, NDArray* input, NDArray* output, bool inplace), FLOAT_TYPES);
        BUILD_SINGLE_TEMPLATE(template int inverse_, (nd4j::LaunchContext * context, NDArray * input, NDArray * output),
                              FLOAT_NATIVE);

        template<typename T>
        __global__ void
        logDetKernel(T *inputBuf, Nd4jLong *inputShape, Nd4jLong batchNum, Nd4jLong *tadShape, Nd4jLong *tadOffsets,
                     T *outputBuf, Nd4jLong *outputShape) {

            __shared__ int n;
            if (threadIdx.x == 0) {
                n = shape::sizeAt(inputShape, -1); // * shape::sizeAt(inputShape, -1);
            }
            __syncthreads();

            T *output = outputBuf;
            T *input = inputBuf;

            Nd4jLong *shapeOf = shape::shapeOf(tadShape);
            Nd4jLong *strideOf = shape::stride(tadShape);

            for (auto i = blockIdx.x; i < batchNum; i += gridDim.x) {
                T *current = input + tadOffsets[i];

                auto zIndex = shape::getIndexOffset(i, outputShape, batchNum);
                for (auto e = threadIdx.x; e < n; e += blockDim.x) {
                    Nd4jLong diag[] = {e, e};
                    auto xIndex = shape::getOffset(0, shapeOf, strideOf, diag, 2);
                    math::atomics::nd4j_atomicAdd(&output[zIndex],
                                                  math::nd4j_log<T, T>(current[xIndex] * current[xIndex]));
                }
            }
        }

        template<typename T>
        int logdetFunctor_(nd4j::LaunchContext *context, NDArray *input, NDArray *output) {
            NDArray::prepareSpecialUse({output}, {input});
            auto n2 = input->sizeAt(-1) * input->sizeAt(-2);
            auto stream = context->getCudaStream();
            std::unique_ptr<NDArray> tempOutput(input->dup());
//        auto inputs = tempOutput->allTensorsAlongDimension({input->rankOf() - 2, input->rankOf() - 1});
//        for (Nd4jLong e = 0; e < packX.numberOfTads(); e++) {
//            auto subArray = inputs->at(e);
//            cholesky(context, subArray, subArray, true);
//        }
//        delete inputs;
            cholesky(context, input, tempOutput.get(), false);
            tempOutput->syncToHost();
            tempOutput->printIndexedBuffer("Cholesky res!!!");
            auto outputBuf = reinterpret_cast<T*>(output->specialBuffer()); // + e * n2; // + e * n2;
            auto inputBuf = reinterpret_cast<T*>(tempOutput->specialBuffer());
            output->assign(0);
            output->syncToDevice();
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(tempOutput->getShapeInfo(),
                                                                                  {input->rankOf() - 2,
                                                                                   input->rankOf() - 1});
            logDetKernel<T> << < packX.numberOfTads(), n2, 128, *stream >> >
                                                                (inputBuf, tempOutput->specialShapeInfo(), packX.numberOfTads(), packX.specialShapeInfo(), packX.specialOffsets(), outputBuf, output->specialShapeInfo());
//        }
            NDArray::registerSpecialUse({output}, {input});
            //delete tempOutput;
            return Status::OK();
        }

        int logdetFunctor(nd4j::LaunchContext *context, NDArray *input, NDArray *output) {
            BUILD_SINGLE_SELECTOR(output->dataType(), logdetFunctor_, (context, input, output), FLOAT_NATIVE);
        }

        BUILD_SINGLE_TEMPLATE(template int logdetFunctor_,
                              (nd4j::LaunchContext * context, NDArray * input, NDArray * output), FLOAT_NATIVE);
    }
}
}
