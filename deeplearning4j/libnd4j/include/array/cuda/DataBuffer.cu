/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma (iuriish@yahoo.com)
//

#include "../DataBuffer.h"
#include <DataTypeUtils.h>
#include <op_boilerplate.h>
#include <exceptions/cuda_exception.h>

namespace nd4j {

////////////////////////////////////////////////////////////////////////
void DataBuffer::allocateSpecial() {

    if (_specialBuffer == nullptr && getLenInBytes() > 0) {
        ALLOCATE_SPECIAL(_specialBuffer, _workspace, getLenInBytes(), int8_t);
        _isOwnerSpecial = true;
    }
}

////////////////////////////////////////////////////////////////////////
void DataBuffer::syncToPrimary(const LaunchContext* context, const bool forceSync) {
    if(isPrimaryActual() && !forceSync)
        return;

    allocatePrimary();

    auto res = hipStreamSynchronize(*context->getCudaStream());
    if (res != 0)
        throw cuda_exception::build("DataBuffer::syncToPrimary failed to to some previous kernel failre", res);

    hipMemcpy(_primaryBuffer, _specialBuffer, getLenInBytes(), hipMemcpyDeviceToHost);

    readPrimary();
}


////////////////////////////////////////////////////////////////////////
void DataBuffer::syncToSpecial(const bool forceSync) {

    if(isSpecialActual() && !forceSync)
        return;

    allocateSpecial();

    hipMemcpy(_specialBuffer, _primaryBuffer, getLenInBytes(), hipMemcpyHostToDevice);

    readSpecial();
}


////////////////////////////////////////////////////////////////////////
void DataBuffer::deleteSpecial() {

    if(_isOwnerSpecial && _specialBuffer != nullptr && getLenInBytes() != 0) {
        auto p = reinterpret_cast<int8_t*>(_specialBuffer);
        RELEASE_SPECIAL(p, _workspace);
        _specialBuffer = nullptr;
        _isOwnerSpecial = false;
    }
}


////////////////////////////////////////////////////////////////////////
void DataBuffer::setCountersToZero() {

    _counter.store(0L);
    _writePrimary.store(0L);
    _writeSpecial.store(0L);
    _readPrimary.store(0L);
    _readSpecial.store(0L);
}

////////////////////////////////////////////////////////////////////////
void DataBuffer::copyCounters(const DataBuffer& other) {

    _counter.store(other._counter);
    _writePrimary.store(other._readSpecial);
    _writeSpecial.store(other._readPrimary);
    _readPrimary.store(other._writeSpecial);
    _readSpecial.store(other._writePrimary);
}

////////////////////////////////////////////////////////////////////////
void DataBuffer::copyBufferFrom(const DataBuffer& other, size_t sizeToCopyinBytes, const Nd4jLong offsetThis, const Nd4jLong offsetOther) {     // copies only to special buffer

    if(other._primaryBuffer == nullptr && other._specialBuffer == nullptr)
        return;

    if(sizeToCopyinBytes == 0)
        sizeToCopyinBytes = other.getLenInBytes();
    if(sizeToCopyinBytes == 0)
        return;

    if(other.isPrimaryActual()) {
        auto res = hipMemcpy(static_cast<int8_t*>(_specialBuffer) + offsetThis * DataTypeUtils::sizeOfElement(_dataType), static_cast<const int8_t*>(other._primaryBuffer) + offsetOther * DataTypeUtils::sizeOfElement(other._dataType), sizeToCopyinBytes, hipMemcpyHostToDevice);
        if (res != 0)
            throw cuda_exception::build("DataBuffer::copyBufferFrom: cudaMemcpy_cudaMemcpyHostToDevice failed!", res);
        other.readPrimary();
    }
    else {
        auto res = hipMemcpy(static_cast<int8_t*>(_specialBuffer) + offsetThis * DataTypeUtils::sizeOfElement(_dataType), static_cast<const int8_t*>(other._specialBuffer) + offsetOther * DataTypeUtils::sizeOfElement(other._dataType), sizeToCopyinBytes, hipMemcpyDeviceToDevice);
        if (res != 0)
            throw cuda_exception::build("DataBuffer::copyBufferFrom: cudaMemcpy_cudaMemcpyDeviceToDevice failed!", res);
        other.readSpecial();
    }

    writeSpecial();
}

////////////////////////////////////////////////////////////////////////
void DataBuffer::copyBufferFromHost(const void* hostBuffer, size_t sizeToCopyinBytes, const Nd4jLong offsetThis, const Nd4jLong offsetHostBuffer) {     // copies only to special buffer

    if(hostBuffer == nullptr)
        return;

    if(sizeToCopyinBytes == 0)
        sizeToCopyinBytes = getLenInBytes();
    if(sizeToCopyinBytes == 0)
        return;

    auto res = hipMemcpy(static_cast<int8_t*>(_specialBuffer) + offsetThis * DataTypeUtils::sizeOfElement(_dataType), static_cast<const int8_t*>(hostBuffer) + offsetHostBuffer * DataTypeUtils::sizeOfElement(_dataType), sizeToCopyinBytes, hipMemcpyHostToDevice);
    if (res != 0)
        throw cuda_exception::build("DataBuffer::copyBufferFromHost: cudaMemcpy_cudaMemcpyHostToDevice failed!", res);

    writeSpecial();
}

////////////////////////////////////////////////////////////////////////
void DataBuffer::setSpecial(void* special, const bool isOwnerSpecial) {

    deleteSpecial();
    _specialBuffer = special;
    _isOwnerSpecial = isOwnerSpecial;
}


////////////////////////////////////////////////////////////////////////
void DataBuffer::allocateBuffers(const bool allocBoth) {    // always allocate special buffer only (cuda case)

    allocateSpecial();

    if(allocBoth)
        allocatePrimary();
}

////////////////////////////////////////////////////////////////////////
void DataBuffer::setToZeroBuffers(const bool both) {

    hipMemset(special(), 0, getLenInBytes());
    writeSpecial();

    if(both) {
        memset(primary(), 0, getLenInBytes());
        readPrimary();
    }
}

////////////////////////////////////////////////////////////////////////
void DataBuffer::migrate() {
    memory::Workspace* newWorkspace = nullptr;
    void* newBuffer;
    ALLOCATE_SPECIAL(newBuffer, newWorkspace, getLenInBytes(), int8_t);
    hipMemcpy(newBuffer, _specialBuffer, getLenInBytes(), hipMemcpyDeviceToDevice);

    if (_isOwnerSpecial) {
        // now we're releasing original buffer
        RELEASE_SPECIAL(_specialBuffer, _workspace);
    }

    _isOwnerSpecial = true;
    _specialBuffer = newBuffer;
}

////////////////////////////////////////////////////////////////////////
void DataBuffer::writePrimary() const    { _writePrimary = ++_counter; }
void DataBuffer::writeSpecial() const    { _writeSpecial = ++_counter; }
void DataBuffer::readPrimary()  const    { _readPrimary  = ++_counter; }
void DataBuffer::readSpecial()  const    { _readSpecial  = ++_counter; }
bool DataBuffer::isPrimaryActual() const { return (_writePrimary.load() > _writeSpecial.load() || _readPrimary.load() > _writeSpecial.load()); }
bool DataBuffer::isSpecialActual() const { return (_writeSpecial.load() > _writePrimary.load() || _readSpecial.load() > _writePrimary.load()); }

}
