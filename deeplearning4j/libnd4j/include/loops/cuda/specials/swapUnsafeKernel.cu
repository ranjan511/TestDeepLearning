#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author GS <sgazeos@gmail.com>, created on 25.01.2019
//

#include <loops/special_kernels.h>

namespace nd4j {

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    template <typename T>
    static __global__ void swapUnsafeKernel(void* theFirstBuffer, Nd4jLong* theFirstShape, void* theSecondBuffer, Nd4jLong* theSecondShape) {
        auto tid = blockIdx.x * blockDim.x + threadIdx.x;
        int totalThreads = gridDim.x * blockDim.x;
        Nd4jLong resultLength = shape::length(theFirstShape);
        //const auto resultLength = shape::length(outputShape);
//        if (shape::order(outputShape) == 'c') {           //  ews == 1 always here
        for (int i = tid; i < resultLength; i += totalThreads) {
            auto xEws = shape::order(theFirstShape)  == 'c'? shape::elementWiseStride(theFirstShape) :1;
            auto yEws = shape::order(theSecondShape) == 'c'? shape::elementWiseStride(theSecondShape):1;
            //if (shape::order(theFirstShape) ==)
            auto xOffset = shape::getIndexOffset(i * xEws, theFirstShape, resultLength);
            auto yOffset = shape::getIndexOffset(i * yEws, theSecondShape, resultLength);
            T temp = *(reinterpret_cast<T*>(theFirstBuffer) + xOffset);
            *(reinterpret_cast<T*>(theFirstBuffer) + xOffset) = *(reinterpret_cast<T*>(theSecondBuffer) + yOffset);
            *(reinterpret_cast<T*>(theSecondBuffer) + yOffset) = temp;
        }
    }

    BUILD_SINGLE_TEMPLATE(template __global__ void swapUnsafeKernel, (void* theFirstBuffer, Nd4jLong* theFirstShape, void* theSecondBuffer, Nd4jLong* theSecondShape), LIBND4J_TYPES);

    template <typename T>
    void templatedSwapUnsafe(void* theFirstBuffer, Nd4jLong* theFirstShape, void* theSecondBuffer, Nd4jLong* theSecondShape, hipStream_t* theStream) {
        swapUnsafeKernel<T><<<256, 512, 8192, *theStream>>>(theFirstBuffer, theFirstShape, theSecondBuffer, theSecondShape);
    }
    BUILD_SINGLE_TEMPLATE(template void templatedSwapUnsafe, (void* theFirstBuffer, Nd4jLong* theFirstShape, void* theSecondBuffer, Nd4jLong* theSecondShape, hipStream_t* theStream), LIBND4J_TYPES);

}