#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

 //
 // @author raver119@gmail.com
 //

#include "testlayers.h"
#include <NDArray.h>
#include <NDArrayFactory.h>
#include <Context.h>
#include <Node.h>
#include <graph/Variable.h>
#include <graph/VariableSpace.h>
#include <specials_cuda.h>
#include <TAD.h>
#include <MmulHelper.h>
#include <helpers/PointersManager.h>
#include <hip/hip_runtime.h>
#include <helpers/RandomLauncher.h>
#include <ConstantShapeHelper.h>
#include <ConstantTadHelper.h>
#include <ShapeDescriptor.h>
#include <array/ConstantDataBuffer.h>

using namespace nd4j;
using namespace nd4j::graph;

class CudaBasicsTests1 : public testing::Test {
public:

};


//////////////////////////////////////////////////////////////////////////
static hipError_t allocateDeviceMem(LaunchContext& lc, std::vector<void*>& devicePtrs, const std::vector<std::pair<void*,size_t>>& hostData) { 

	if(devicePtrs.size() != hostData.size())
		throw std::invalid_argument("prepareDataForCuda: two input sts::vectors should same sizes !");

	hipError_t cudaResult;

	void* reductionPointer;
    cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer),  1024*1024);			if(cudaResult != 0) return cudaResult;
    int* allocationPointer;
	cudaResult = hipMalloc(reinterpret_cast<void **>(&allocationPointer), 1024*1024);			if(cudaResult != 0) return cudaResult;

	lc.setReductionPointer(reductionPointer);
	lc.setAllocationPointer(allocationPointer);
	hipStream_t stream = *lc.getCudaStream();

	for(int i = 0; i < devicePtrs.size(); ++i) {
		
		cudaResult = hipMalloc(reinterpret_cast<void **>(&devicePtrs[i]), hostData[i].second); if(cudaResult != 0) return cudaResult;
		hipMemcpyAsync(devicePtrs[i], hostData[i].first, hostData[i].second, hipMemcpyHostToDevice, stream);				
	}
	return cudaResult;
}

//////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, TestPairwise_1) {
	// allocating host-side arrays
	auto x = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
	auto z = NDArrayFactory::create<double>('c', { 5 }, {0,0,0,0,0});

	auto exp = NDArrayFactory::create<double>('c', { 5 }, { 2, 4, 6, 8, 10 });

	// making raw buffers
	Nd4jPointer devBufferPtrX, devBufferPtrZ, devShapePtrX;
	hipError_t res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrX), x.lengthOf() * x.sizeOfT());
	ASSERT_EQ(0, res);
	res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrZ), x.lengthOf() * x.sizeOfT());
	ASSERT_EQ(0, res);
	res = hipMalloc(reinterpret_cast<void **>(&devShapePtrX), shape::shapeInfoByteLength(x.shapeInfo()));
	ASSERT_EQ(0, res);

	Nd4jPointer nativeStream = (Nd4jPointer)malloc(sizeof(hipStream_t));
	CHECK_ALLOC(nativeStream, "Failed to allocate memory for new CUDA stream", sizeof(hipStream_t));
	hipError_t dZ = hipStreamCreate(reinterpret_cast<hipStream_t *>(&nativeStream));
	auto stream = reinterpret_cast<hipStream_t *>(&nativeStream);

    x.dataBuffer()->allocatePrimary();
    x.syncToHost();

	hipMemcpyAsync(devBufferPtrX, x.buffer(), x.lengthOf() * x.sizeOfT(), hipMemcpyHostToDevice, *stream);
	hipMemcpyAsync(devShapePtrX, x.shapeInfo(), shape::shapeInfoByteLength(x.shapeInfo()), hipMemcpyHostToDevice, *stream);
	
	LaunchContext lc(stream, nullptr, nullptr);
	NativeOpExecutioner::execPairwiseTransform(&lc, pairwise::Add, nullptr, x.shapeInfo(), devBufferPtrX, reinterpret_cast<Nd4jLong*>(devShapePtrX), nullptr, x.shapeInfo(), devBufferPtrX, reinterpret_cast<Nd4jLong*>(devShapePtrX), nullptr, z.shapeInfo(), devBufferPtrZ, reinterpret_cast<Nd4jLong*>(devShapePtrX), nullptr);
	res = hipStreamSynchronize(*stream);
	ASSERT_EQ(0, res);

	z.dataBuffer()->allocatePrimary();

	hipMemcpyAsync(z.buffer(), devBufferPtrZ, z.lengthOf() * x.sizeOfT(), hipMemcpyDeviceToHost, *stream);
	res = hipStreamSynchronize(*stream);
	ASSERT_EQ(0, res);

	hipFree(devBufferPtrX);
	hipFree(devBufferPtrZ);
	hipFree(devShapePtrX);

	// needed due to memcpy
    z.tickWriteHost();

	for (int e = 0; e < z.lengthOf(); e++) {
		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
	}
}


////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execIndexReduceScalar_1) {

    NDArray x1('c', {2,2}, {0, 1, 2, 3}, nd4j::DataType::INT32);
    NDArray x2('c', {2,2}, {0.5, 1.5, -4.5, 3.5}, nd4j::DataType::BFLOAT16);    
    NDArray x3('c', {2,2}, {0, -1, 0, 1}, nd4j::DataType::BOOL);
    
    NDArray scalar('c', {}, {0}, nd4j::DataType::INT64);

    NDArray exp1('c', {}, {3}, nd4j::DataType::INT64);
    NDArray exp2('c', {}, {2}, nd4j::DataType::INT64);
    NDArray exp3('c', {}, {1}, nd4j::DataType::INT64);

    void *dX1, *dX2, *dX3, *dZ; 
    Nd4jLong *dX1ShapeInfo, *dX2ShapeInfo, *dX3ShapeInfo, *dZShapeInfo;

    hipError_t cudaResult;

    cudaResult = hipMalloc(reinterpret_cast<void **>(&dX1), x1.lengthOf() * x1.sizeOfT()); 		   		         	 ASSERT_EQ(0, cudaResult);
    cudaResult = hipMalloc(reinterpret_cast<void **>(&dX2), x2.lengthOf() * x2.sizeOfT()); 		   		         	 ASSERT_EQ(0, cudaResult);    
    cudaResult = hipMalloc(reinterpret_cast<void **>(&dX3), x3.lengthOf() * x3.sizeOfT()); 		   		         	 ASSERT_EQ(0, cudaResult);    
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dZ), scalar.lengthOf() * scalar.sizeOfT()); 				         ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dX1ShapeInfo), shape::shapeInfoByteLength(x1.getShapeInfo()));    ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dX2ShapeInfo), shape::shapeInfoByteLength(x2.getShapeInfo()));    ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dX3ShapeInfo), shape::shapeInfoByteLength(x3.getShapeInfo()));    ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dZShapeInfo), shape::shapeInfoByteLength(scalar.getShapeInfo())); ASSERT_EQ(0, cudaResult);	

    hipStream_t stream;
	cudaResult = hipStreamCreate(&stream); 
	ASSERT_EQ(0, cudaResult);

	x1.syncToHost();
	x2.syncToHost();
	x3.syncToHost();
	scalar.syncToHost();
	
	hipMemcpyAsync(dX1, x1.buffer(), x1.lengthOf() * x1.sizeOfT(), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dX2, x2.buffer(), x2.lengthOf() * x2.sizeOfT(), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dX3, x3.buffer(), x3.lengthOf() * x3.sizeOfT(), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dX1ShapeInfo, x1.getShapeInfo(), shape::shapeInfoByteLength(x1.getShapeInfo()), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dX2ShapeInfo, x2.getShapeInfo(), shape::shapeInfoByteLength(x2.getShapeInfo()), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dX3ShapeInfo, x3.getShapeInfo(), shape::shapeInfoByteLength(x3.getShapeInfo()), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dZShapeInfo, scalar.getShapeInfo(), shape::shapeInfoByteLength(scalar.getShapeInfo()), hipMemcpyHostToDevice, stream);
	
	void* reductionPointer = nullptr;
	cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer), 1024*1024);
	ASSERT_EQ(0, cudaResult);

	LaunchContext lc(&stream, LaunchContext::defaultContext()->getReductionPointer(), LaunchContext::defaultContext()->getScalarPointer(), LaunchContext::defaultContext()->getAllocationPointer());

	/***************************************/
	
    NativeOpExecutioner::execIndexReduceScalar(&lc,
    											nd4j::indexreduce::IndexAbsoluteMax, 
    											x1.buffer(), x1.getShapeInfo(),
    	                                       	dX1, dX1ShapeInfo, 
    	                                       	nullptr, 
    	                                       	scalar.buffer(), scalar.getShapeInfo(),
    	                                       	dZ, dZShapeInfo);

    cudaResult = hipStreamSynchronize(stream); 
    ASSERT_EQ(0, cudaResult);

    hipMemcpyAsync(scalar.buffer(), dZ, scalar.lengthOf() * scalar.sizeOfT(), hipMemcpyDeviceToHost, stream);

    cudaResult = hipStreamSynchronize(stream); 
    ASSERT_EQ(0, cudaResult);

    scalar.tickWriteHost();

	ASSERT_NEAR(exp1.e<float>(0), scalar.e<float>(0), 1e-5);

    /***************************************/
    
    NativeOpExecutioner::execIndexReduceScalar(&lc,
    											nd4j::indexreduce::IndexAbsoluteMax, 
    											nullptr, x2.getShapeInfo(),
    	                                       	dX2, dX2ShapeInfo, 
    	                                       	nullptr, 
    	                                       	nullptr, scalar.getShapeInfo(),
    	                                       	dZ, dZShapeInfo);

    cudaResult = hipStreamSynchronize(stream); 
    ASSERT_EQ(0, cudaResult);

    hipMemcpyAsync(scalar.buffer(), dZ, scalar.lengthOf() * scalar.sizeOfT(), hipMemcpyDeviceToHost, stream);

    cudaResult = hipStreamSynchronize(stream); 
    ASSERT_EQ(0, cudaResult);

    ASSERT_NEAR(exp2.e<float>(0), scalar.e<float>(0), 1e-5);

    // *************************************

    NativeOpExecutioner::execIndexReduceScalar(&lc, 
    											nd4j::indexreduce::IndexAbsoluteMax, 
    											nullptr, x3.getShapeInfo(),
    	                                       	dX3, dX3ShapeInfo, 
    	                                       	nullptr, 
    	                                       	nullptr, scalar.getShapeInfo(),
    	                                       	dZ, dZShapeInfo);

    cudaResult = hipStreamSynchronize(stream); 
    ASSERT_EQ(0, cudaResult);

    hipMemcpyAsync(scalar.buffer(), dZ, scalar.lengthOf() * scalar.sizeOfT(), hipMemcpyDeviceToHost, stream);

    cudaResult = hipStreamSynchronize(stream); 
    ASSERT_EQ(0, cudaResult);

    ASSERT_NEAR(exp3.e<float>(0), scalar.e<float>(0), 1e-5);
    
	/***************************************/

	hipFree(dX1); 			hipFree(dX2); 			hipFree(dX3); 			hipFree(dZ);
	hipFree(dX1ShapeInfo); hipFree(dX2ShapeInfo); hipFree(dX3ShapeInfo); hipFree(dZShapeInfo); 

	/***************************************/	

	cudaResult = hipStreamDestroy(stream); 
	ASSERT_EQ(0, cudaResult);
	
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduce3Scalar_1) {

	 if (!Environment::getInstance()->isExperimentalBuild())
        return;

    NDArray x1('c', {2,2}, {1,2,3,4}, nd4j::DataType::INT32);
    NDArray x2('c', {2,2}, {-1,-2,-3,-4}, nd4j::DataType::INT32);
    NDArray x3('c', {2,2}, {1.5,1.5,1.5,1.5}, nd4j::DataType::DOUBLE);
    NDArray x4('c', {2,2}, {1,2,3,4}, nd4j::DataType::DOUBLE);

    NDArray exp1('c', {}, {-30.f}, nd4j::DataType::FLOAT32);
    NDArray exp2('c', {}, {15.}, nd4j::DataType::DOUBLE);
    
	NDArray scalar1('c', {}, {100.f}, nd4j::DataType::FLOAT32);
    NDArray scalar2('c', {}, {100.}, nd4j::DataType::DOUBLE);

    void *dX1, *dX2, *dX3, *dX4, *dZ1, *dZ2; 
    Nd4jLong *dX1ShapeInfo, *dX3ShapeInfo, *dZ1ShapeInfo, *dZ2ShapeInfo;

    hipError_t cudaResult;

    cudaResult = hipMalloc(reinterpret_cast<void **>(&dX1), x1.lengthOf() * x1.sizeOfT()); 		   		         	 	ASSERT_EQ(0, cudaResult);
    cudaResult = hipMalloc(reinterpret_cast<void **>(&dX2), x2.lengthOf() * x2.sizeOfT()); 		   		         	 	ASSERT_EQ(0, cudaResult);
    cudaResult = hipMalloc(reinterpret_cast<void **>(&dX3), x3.lengthOf() * x3.sizeOfT()); 		   		         	 	ASSERT_EQ(0, cudaResult);
    cudaResult = hipMalloc(reinterpret_cast<void **>(&dX4), x4.lengthOf() * x4.sizeOfT()); 		   		         	 	ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dZ1), scalar1.lengthOf() * scalar1.sizeOfT());			         	ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dZ2), scalar2.lengthOf() * scalar2.sizeOfT());			         	ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dX1ShapeInfo), shape::shapeInfoByteLength(x1.getShapeInfo()));    	ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dX3ShapeInfo), shape::shapeInfoByteLength(x3.getShapeInfo()));    	ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dZ1ShapeInfo), shape::shapeInfoByteLength(scalar1.getShapeInfo())); 	ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dZ2ShapeInfo), shape::shapeInfoByteLength(scalar2.getShapeInfo())); 	ASSERT_EQ(0, cudaResult);

    hipStream_t stream;
	cudaResult = hipStreamCreate(&stream); 
	ASSERT_EQ(0, cudaResult);

	x1.syncToHost();
	x2.syncToHost();
	x3.syncToHost();
	x4.syncToHost();
	scalar1.syncToHost();
	scalar2.syncToHost();
	
	hipMemcpyAsync(dX1, x1.buffer(), x1.lengthOf() * x1.sizeOfT(), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dX2, x2.buffer(), x2.lengthOf() * x2.sizeOfT(), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dX3, x3.buffer(), x3.lengthOf() * x3.sizeOfT(), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dX4, x4.buffer(), x4.lengthOf() * x4.sizeOfT(), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dX1ShapeInfo, x1.getShapeInfo(), shape::shapeInfoByteLength(x1.getShapeInfo()), hipMemcpyHostToDevice, stream);	
	hipMemcpyAsync(dX3ShapeInfo, x3.getShapeInfo(), shape::shapeInfoByteLength(x3.getShapeInfo()), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dZ1ShapeInfo, scalar1.getShapeInfo(), shape::shapeInfoByteLength(scalar1.getShapeInfo()), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dZ2ShapeInfo, scalar2.getShapeInfo(), shape::shapeInfoByteLength(scalar2.getShapeInfo()), hipMemcpyHostToDevice, stream);

	/***************************************/

	void* reductionPointer  = nullptr;
	int*  allocationPointer = nullptr;	

	cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer),  1024*1024);		ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&allocationPointer), 1024*1024);		ASSERT_EQ(0, cudaResult);

	LaunchContext lc(&stream, reductionPointer, nullptr, allocationPointer);

	/***************************************/
	
    NativeOpExecutioner::execReduce3Scalar(&lc, nd4j::reduce3::Dot,nullptr, x1.getShapeInfo(),dX1, dX1ShapeInfo, nullptr, nullptr, x2.getShapeInfo(),dX2, dX1ShapeInfo,nullptr, scalar1.getShapeInfo(),dZ1, dZ1ShapeInfo);

    cudaResult = hipStreamSynchronize(stream);     
    ASSERT_EQ(0, cudaResult);

    scalar1.tickWriteHost();
    scalar2.tickWriteHost();

    hipMemcpyAsync(scalar1.buffer(), dZ1, scalar1.lengthOf() * scalar1.sizeOfT(), hipMemcpyDeviceToHost, stream);

    cudaResult = hipStreamSynchronize(stream); 
    ASSERT_EQ(0, cudaResult);

	ASSERT_NEAR(exp1.e<float>(0), scalar1.e<float>(0), 1e-5);

    /***************************************/
    
    NativeOpExecutioner::execReduce3Scalar(&lc, nd4j::reduce3::Dot,nullptr, x3.getShapeInfo(),dX3, dX3ShapeInfo, nullptr, nullptr, x4.getShapeInfo(),dX4, dX3ShapeInfo,nullptr, scalar2.getShapeInfo(),dZ2, dZ2ShapeInfo);

    cudaResult = hipStreamSynchronize(stream); 
    ASSERT_EQ(0, cudaResult);

    hipMemcpyAsync(scalar2.buffer(), dZ2, scalar2.lengthOf() * scalar2.sizeOfT(), hipMemcpyDeviceToHost, stream);

    cudaResult = hipStreamSynchronize(stream); 
    ASSERT_EQ(0, cudaResult);

	ASSERT_NEAR(exp2.e<float>(0), scalar2.e<float>(0), 1e-5);
    
	/***************************************/

	hipFree(dX1); 			hipFree(dX2); hipFree(dX3); 		   hipFree(dX4); 	hipFree(dZ1); 				hipFree(dZ2);
	hipFree(dX1ShapeInfo); 			   hipFree(dX3ShapeInfo); 					hipFree(dZ1ShapeInfo);		hipFree(dZ2ShapeInfo);

	/***************************************/	

	cudaResult = hipStreamDestroy(stream); 
	ASSERT_EQ(0, cudaResult);
}
 

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduce3_1) {

    NDArray x('c', {2,2}, {1,2,3,4}, nd4j::DataType::INT32);
    NDArray y('c', {2,2}, {-1,-2,-3,-4}, nd4j::DataType::INT32);

    NDArray exp('c', {}, {-30.f}, nd4j::DataType::FLOAT32);
    NDArray z('c', {}, {100.f},  nd4j::DataType::FLOAT32);

    std::vector<int> dimensions = {0, 1};

    x.syncToHost();
    y.syncToHost();
    z.syncToHost();

    
    std::vector<std::pair<void*,size_t>> hostData;    	
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions	
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

    hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduce3(&lc, nd4j::reduce3::Dot, 
								nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
								nullptr, 
								nullptr, y.getShapeInfo(), y.specialBuffer(), y.specialShapeInfo(), 
								nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
								(int*)devicePtrs[0], dimensions.size(), 
								nullptr, nullptr, nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();
 	
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}


////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduce3_2) {
    
	NDArray x('c', {2,2}, {1.5,1.5,1.5,1.5}, nd4j::DataType::DOUBLE);
    NDArray y('c', {2,2}, {1,2,3,4}, nd4j::DataType::DOUBLE);

    NDArray exp('c', {}, {15.}, nd4j::DataType::DOUBLE);
    NDArray z('c', {}, {100.},  nd4j::DataType::DOUBLE);
   
    std::vector<int> dimensions = {0, 1};   

    // prepare input arrays for prepareDataForCuda function       
    std::vector<std::pair<void*,size_t>> hostData;    	
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions	
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result	
	NativeOpExecutioner::execReduce3(&lc, nd4j::reduce3::Dot, 
								nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
								nullptr, 
								nullptr, y.getShapeInfo(), y.specialBuffer(), y.specialShapeInfo(), 
								nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
								(int*)devicePtrs[0], dimensions.size(), 
								nullptr, nullptr, nullptr, nullptr);


	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();
 	
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduce3_3) {
    
	NDArray x('c', {2,3}, {1,2,3,4,5,6}, nd4j::DataType::INT32);
    NDArray y('c', {2,3}, {-6,-5,-4,-3,-2,-1}, nd4j::DataType::INT32);        

    NDArray exp('c', {3}, {-18,-20,-18}, nd4j::DataType::FLOAT32);
    NDArray z('c', {3}, {100,100,100}, nd4j::DataType::FLOAT32);
   
    std::vector<int> dimensions = {0};

    // evaluate xTad data 
    shape::TAD xTad;
    xTad.init(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // evaluate yTad data
    shape::TAD yTad;
    yTad.init(y.getShapeInfo(), dimensions.data(), dimensions.size());
    yTad.createTadOnlyShapeInfo();
    yTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function    
    std::vector<std::pair<void*,size_t>> hostData;    
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	hostData.emplace_back(yTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(yTad.tadOnlyShapeInfo));// 3 -- yTadShapeInfo
	hostData.emplace_back(yTad.tadOffsets, yTad.numTads * sizeof(Nd4jLong));						// 4-- yTadOffsets	
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		

	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result	
	NativeOpExecutioner::execReduce3(&lc, nd4j::reduce3::Dot, 
								nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
								nullptr, 
								nullptr, y.getShapeInfo(), y.specialBuffer(), y.specialShapeInfo(), 
								nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
								(int*)devicePtrs[0], dimensions.size(), 
								(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2], 
								(Nd4jLong*)devicePtrs[3], (Nd4jLong*)devicePtrs[4]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
	z.tickWriteDevice();
 	
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduce3_4) {
    	
    NDArray x('c', {2,3}, {1,2,3,4,5,6}, nd4j::DataType::DOUBLE);
    NDArray y('c', {2,3}, {1.5,1.5,1.5,1.5,1.5,1.5}, nd4j::DataType::DOUBLE);

    NDArray exp('c', {2}, {9,22.5}, nd4j::DataType::DOUBLE);
    NDArray z('c', {2}, {100,100}, nd4j::DataType::DOUBLE);
   
    std::vector<int> dimensions = {1};

    // evaluate xTad data 
    shape::TAD xTad;
    xTad.init(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // evaluate yTad data
    shape::TAD yTad;
    yTad.init(y.getShapeInfo(), dimensions.data(), dimensions.size());
    yTad.createTadOnlyShapeInfo();
    yTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;    	
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	hostData.emplace_back(yTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(yTad.tadOnlyShapeInfo));// 3 -- yTadShapeInfo
	hostData.emplace_back(yTad.tadOffsets, yTad.numTads * sizeof(Nd4jLong));						// 4-- yTadOffsets	
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		

	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result	
	NativeOpExecutioner::execReduce3(&lc, nd4j::reduce3::Dot,
								nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
								nullptr, 
								nullptr, y.getShapeInfo(), y.specialBuffer(), y.specialShapeInfo(), 
								nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
								(int*)devicePtrs[0], dimensions.size(), 
								(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2], 
								(Nd4jLong*)devicePtrs[3], (Nd4jLong*)devicePtrs[4]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();
 	
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduce3_5) {
    	
    NDArray x('c', {2,2,3}, {1.5,1.5,1.5,1.5,1.5,1.5,1.5,1.5,1.5,1.5,1.5,1.5}, nd4j::DataType::FLOAT32);
    NDArray y('c', {2,2,3}, {1,2,3,4,5,6,7,8,9,10,11,12}, nd4j::DataType::FLOAT32);

    NDArray exp('c', {2,3}, {7.5, 10.5, 13.5, 25.5, 28.5, 31.5}, nd4j::DataType::FLOAT32);
    NDArray z('c', {2,3}, {100,100,100,100,100,100}, nd4j::DataType::FLOAT32);
   
    std::vector<int> dimensions = {1};

    // evaluate xTad data 
    shape::TAD xTad;
    xTad.init(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // evaluate yTad data
    shape::TAD yTad;
    yTad.init(y.getShapeInfo(), dimensions.data(), dimensions.size());
    yTad.createTadOnlyShapeInfo();
    yTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;    	
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	hostData.emplace_back(yTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(yTad.tadOnlyShapeInfo));// 3 -- yTadShapeInfo
	hostData.emplace_back(yTad.tadOffsets, yTad.numTads * sizeof(Nd4jLong));						// 4-- yTadOffsets	
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		

	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduce3(&lc, nd4j::reduce3::Dot,
								nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
								nullptr, 
								nullptr, y.getShapeInfo(), y.specialBuffer(), y.specialShapeInfo(), 
								nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
								(int*)devicePtrs[0], dimensions.size(), 
								(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2], 
								(Nd4jLong*)devicePtrs[3], (Nd4jLong*)devicePtrs[4]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();
 	
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduce3All_1) {
    	
    NDArray x('c', {2,2}, {1,2,3,4}, nd4j::DataType::INT32);
    NDArray y('c', {2,3}, {-1,1,-1,1,-1,1}, nd4j::DataType::INT32);

    NDArray exp('c', {2,3}, {2,-2,2,2,-2,2}, nd4j::DataType::FLOAT32);
    NDArray z('c', {2,3}, {100,100,100,100,100,100}, nd4j::DataType::FLOAT32);
   
    std::vector<int> dimensions = {0};

    // evaluate xTad data 
    shape::TAD xTad;
    xTad.init(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // evaluate yTad data
    shape::TAD yTad;
    yTad.init(y.getShapeInfo(), dimensions.data(), dimensions.size());
    yTad.createTadOnlyShapeInfo();
    yTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function    
    std::vector<std::pair<void*,size_t>> hostData;    	
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	hostData.emplace_back(yTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(yTad.tadOnlyShapeInfo));// 3 -- yTadShapeInfo
	hostData.emplace_back(yTad.tadOffsets, yTad.numTads * sizeof(Nd4jLong));						// 4 -- yTadOffsets	
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		

	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduce3All(&lc, nd4j::reduce3::Dot, 
										nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
										nullptr, 
										nullptr, y.getShapeInfo(), y.specialBuffer(), y.specialShapeInfo(), 
										nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
										(int*)devicePtrs[0], dimensions.size(), 
										(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2], 
										(Nd4jLong*)devicePtrs[3], (Nd4jLong*)devicePtrs[4]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
	z.tickWriteDevice();
 	
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduce3All_2) {
    	
    NDArray x('c', {2,2}, {1,2,3,4}, nd4j::DataType::DOUBLE);
    NDArray y('c', {2,3}, {1.5,1.5,1.5,1.5,1.5,1.5}, nd4j::DataType::DOUBLE);    

    NDArray exp('c', {2,3}, {6,6,6,9,9,9}, nd4j::DataType::DOUBLE);    
    NDArray z('c', {2,3}, {100,100,100,100,100,100,},nd4j::DataType::DOUBLE);    
   
    std::vector<int> dimensions = {0};

    // evaluate xTad data 
    shape::TAD xTad;
    xTad.init(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // evaluate yTad data
    shape::TAD yTad;
    yTad.init(y.getShapeInfo(), dimensions.data(), dimensions.size());
    yTad.createTadOnlyShapeInfo();
    yTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function    
    std::vector<std::pair<void*,size_t>> hostData;    
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	hostData.emplace_back(yTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(yTad.tadOnlyShapeInfo));// 3 -- yTadShapeInfo
	hostData.emplace_back(yTad.tadOffsets, yTad.numTads * sizeof(Nd4jLong));						// 4-- yTadOffsets	
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		

	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduce3All(&lc, nd4j::reduce3::Dot, 
										nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
										nullptr, 
										nullptr, y.getShapeInfo(), y.specialBuffer(), y.specialShapeInfo(), 
										nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
										(int*)devicePtrs[0], dimensions.size(), 
										(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2], 
										(Nd4jLong*)devicePtrs[3], (Nd4jLong*)devicePtrs[4]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();
 	
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execIndexReduce_1) {
    	
    NDArray x('c', {2,3}, {100,100,100,100,100,100}, nd4j::DataType::DOUBLE);
    x.linspace(-2.); x.syncToDevice();
    NDArray exp('c', {2}, {2, 2}, nd4j::DataType::INT64);
    NDArray z('c', {2}, {100,100}, nd4j::DataType::INT64);
    
    std::vector<int> dimensions = {1};          

    // evaluate xTad data 
    shape::TAD xTad;
    xTad.init(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function        
    std::vector<std::pair<void*,size_t>> hostData;    	
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		

	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execIndexReduce(&lc, nd4j::indexreduce::IndexMax, 
										nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
										nullptr, 
										nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
										(int*)devicePtrs[0], dimensions.size(), 
										(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();
 	
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) 
		hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execIndexReduce_2) {
    	
    NDArray x('c', {2,3,4,5}, {100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,
    						  	100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,
    							100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,
    							100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,
    							100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,
    							100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100}, nd4j::DataType::FLOAT32);
    x.linspace(-2.f); x.syncToDevice();
    NDArray exp('c', {2,5}, {11,11,11,11,11,11,11,11,11,11}, nd4j::DataType::INT64);    
    NDArray z('c', {2,5}, {100,100,100,100,100,100,100,100,100,100}, nd4j::DataType::INT64);
    
    std::vector<int> dimensions = {1,2};     

    // evaluate xTad data 
    shape::TAD xTad;
    xTad.init(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function    
    
    std::vector<std::pair<void*,size_t>> hostData;    
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		

	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execIndexReduce(&lc, nd4j::indexreduce::IndexMax, 
										nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
										nullptr, 
										nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
										(int*)devicePtrs[0], dimensions.size(), 
										(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();
 	
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) 
		hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execIndexReduce_3) {
    	
    NDArray x('c', {2,3,4,5}, {100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,
    						  	100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,
    							100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,
    							100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,
    							100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,
    							100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100}, nd4j::DataType::DOUBLE);
    x.linspace(-2.); x.syncToDevice();
    NDArray exp('c', {3}, {39, 39, 39}, nd4j::DataType::INT64);    
    NDArray z('c', {3}, {100,100,100}, nd4j::DataType::INT64);
    
    std::vector<int> dimensions = {0,2,3};

    // evaluate xTad data 
    shape::TAD xTad;
    xTad.init(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function        
    std::vector<std::pair<void*,size_t>> hostData;   
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	std::vector<void*> devicePtrs(hostData.size(), nullptr);
	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		

	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execIndexReduce(&lc, nd4j::indexreduce::IndexMax, 
										nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
										nullptr, 
										nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
										(int*)devicePtrs[0], dimensions.size(), 
										(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();
 	
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) 
		hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execScalar_1) {

	if (!Environment::getInstance()->isExperimentalBuild())
        return;
    	
    NDArray x('c', {2,3},  {0,1,2,3,4,5}, nd4j::DataType::INT64); 
    NDArray exp('c',{2,3}, {0,0,1,1,2,2}, nd4j::DataType::INT64);
    NDArray scalar('c',{}, {2.f}, nd4j::DataType::FLOAT32);
    NDArray z('c', {2,3}, {100,100,100,100,100,100}, nd4j::DataType::INT64);
    
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);
	
	// call cuda kernel which calculates result
	NativeOpExecutioner::execScalar(&lc, nd4j::scalar::Divide, 
									nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
									nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
									nullptr, scalar.getShapeInfo(), scalar.specialBuffer(), scalar.specialShapeInfo(), 
									nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();
 	
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execScalar_2) {

	if (!Environment::getInstance()->isExperimentalBuild())
        return;
    	
    NDArray x('c', {2,3},  {-1,-2,-3,-4,-5,-6}, nd4j::DataType::INT64); 
    NDArray exp('c',{2,3}, {10,10,10,10,10,10}, nd4j::DataType::FLOAT32);
    NDArray scalar('c',{}, {10.f}, nd4j::DataType::FLOAT32);
    NDArray z('c', {2,3}, {100,100,100,100,100,100}, nd4j::DataType::FLOAT32);
    
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execScalar(&lc, nd4j::scalar::CopyPws, 
									nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
									nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
									nullptr, scalar.getShapeInfo(), scalar.specialBuffer(), scalar.specialShapeInfo(), 
									nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();
 	
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);


	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execScalar_3) {

	if (!Environment::getInstance()->isExperimentalBuild())
        return;
    	
    NDArray x('c', {2,3,2},  {0,1,2,3,4,5,6,7,8,9,10,11}, nd4j::DataType::INT64); 
    NDArray scalars('c',{2,2}, {1,2,3,4}, nd4j::DataType::FLOAT32);
    NDArray exp('c', {2,3,2},  {0,0,2,1,4,2, 2,1,2,2,3,2}, nd4j::DataType::INT64);     
    NDArray z('c', {2,3,2}, {100,100,100,100,100,100,100,100,100,100,100,100}, nd4j::DataType::INT64);

    std::vector<int> dimensions = {1};

    // evaluate xTad data 
    shape::TAD xTad;
    xTad.init(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function       
    std::vector<std::pair<void*,size_t>> hostData;   
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));							// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));	// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));							// 2 -- xTadOffsets	
	std::vector<void*> devicePtrs(hostData.size(), nullptr);
	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		

	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execScalar(&lc, nd4j::scalar::Divide, 
									nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
									nullptr,
									nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
									nullptr, scalars.getShapeInfo(), scalars.specialBuffer(), scalars.specialShapeInfo(),
									(int*)devicePtrs[0], dimensions.size(), 
									(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2], 
									nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();
 	
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5); 		

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) 
		hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execScalarBool_1) {
    	
    NDArray x('c', {2,3},  {-1,-2,0,1,2,3}, nd4j::DataType::BFLOAT16); 
    NDArray scalar('c',{}, {0}, nd4j::DataType::BFLOAT16);
    NDArray exp('c',{2,3}, {0,0,0,1,1,1}, nd4j::DataType::BOOL);    
    NDArray z('c', {2,3}, {100,100,100,100,100,100,}, nd4j::DataType::BOOL);    
	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);
		
	// call cuda kernel which calculates result
	// call cuda kernel which calculates result
	NativeOpExecutioner::execScalarBool(&lc, nd4j::scalar::GreaterThan, 
									nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
									nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
									nullptr, scalar.getShapeInfo(), scalar.specialBuffer(), scalar.specialShapeInfo(), 
									nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();
 	
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execScalarBool_2) {
    	
    NDArray x('c', {2,3},  {0,1,2,3,4,5}, nd4j::DataType::FLOAT32); 
    NDArray scalars('c',{2}, {-1,4}, nd4j::DataType::FLOAT32);
    NDArray exp('c', {2,3},  {1,1,1,0,0,1}, nd4j::DataType::BOOL);
    NDArray z('c', {2,3}, {100,100,100,100,100,100}, nd4j::DataType::BOOL);

    std::vector<int> dimensions = {1};

    // evaluate xTad data 
    shape::TAD xTad;
    xTad.init(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function       
    std::vector<std::pair<void*,size_t>> hostData;   
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));							// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));	// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));							// 2 -- xTadOffsets	
	std::vector<void*> devicePtrs(hostData.size(), nullptr);
	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
			
	// call cuda kernel which calculates result
	NativeOpExecutioner::execScalarBool(&lc, nd4j::scalar::GreaterThan, 
									nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
									nullptr,
									nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
									nullptr, scalars.getShapeInfo(), scalars.specialBuffer(), scalars.specialShapeInfo(),
									(int*)devicePtrs[0], dimensions.size(), 
									(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2], 
									nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();
 	
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5); 		

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) 
		hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execBroadcast_1) {

	if (!Environment::getInstance()->isExperimentalBuild())
        return;
    	
	NDArray x('c', {2,3,4}, {100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100}, nd4j::DataType::INT32);
    NDArray y('c', {3},   {10, 20, 30}, nd4j::DataType::INT64);
    NDArray z('c', {2,3,4}, {100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100}, nd4j::DataType::INT32);	
	NDArray exp('c', {2,3,4}, {10, 11, 12, 13,24, 25, 26, 27,38, 39, 40, 41,22, 23, 24, 25,36, 37, 38, 39,50, 51, 52, 53}, nd4j::DataType::INT32);
	x.linspace(0); x.syncToDevice();

    std::vector<int> dimensions = {1};

    // evaluate xTad data 
    shape::TAD xTad;
    xTad.init(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function       
    std::vector<std::pair<void*,size_t>> hostData;   
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));							// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));	// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));							// 2 -- xTadOffsets	
	std::vector<void*> devicePtrs(hostData.size(), nullptr);
	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execBroadcast(&lc, nd4j::broadcast::Add,
										nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
										nullptr, y.getShapeInfo(), y.specialBuffer(), y.specialShapeInfo(),
										nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
										(int*)devicePtrs[0], dimensions.size(), 
										(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2], 
										nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();
 	
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5); 		

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) 
		hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execBroadcast_2) {

	if (!Environment::getInstance()->isExperimentalBuild())
        return;
    	
	NDArray x('c', {2,3,4}, {100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100}, nd4j::DataType::INT32);
    NDArray y('c', {2,4},   {10,20,30,40,50,60,70,80}, nd4j::DataType::FLOAT32);
    NDArray z('c', {2,3,4}, {100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100}, nd4j::DataType::FLOAT32);	
	NDArray exp('c', {2,3,4}, {10., 21., 32., 43., 14., 25., 36., 47., 18., 29., 40., 51., 62., 73., 84., 95., 66., 77., 88., 99., 70., 81., 92., 103}, nd4j::DataType::FLOAT32);
	x.linspace(0); x.syncToDevice();

    std::vector<int> dimensions = {0,2};

    // evaluate xTad data 
    shape::TAD xTad;
    xTad.init(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function       
    std::vector<std::pair<void*,size_t>> hostData;   
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));							// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));	// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));							// 2 -- xTadOffsets	
	std::vector<void*> devicePtrs(hostData.size(), nullptr);
	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execBroadcast(&lc, nd4j::broadcast::Add,
										nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
										nullptr, y.getShapeInfo(), y.specialBuffer(), y.specialShapeInfo(),
										nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
										(int*)devicePtrs[0], dimensions.size(), 
										(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2], 
										nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();
 	
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5); 		

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) 
		hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execBroadcastBool_1) {
    	
	NDArray x('c', {2,3,4}, {100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100}, nd4j::DataType::INT32);
    NDArray y('c', {3},   {2, 12, 22}, nd4j::DataType::INT32);
    NDArray z('c', {2,3,4}, {100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,}, nd4j::DataType::BOOL);	
	NDArray exp('c', {2,3,4}, {0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0}, nd4j::DataType::BOOL);
	x.linspace(1); x.syncToDevice();

    std::vector<int> dimensions = {1};

    // evaluate xTad data 
    shape::TAD xTad;
    xTad.init(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function       
    std::vector<std::pair<void*,size_t>> hostData;   
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));							// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));	// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));							// 2 -- xTadOffsets	
	std::vector<void*> devicePtrs(hostData.size(), nullptr);
	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execBroadcastBool(&lc, nd4j::broadcast::EqualTo,
										nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
										nullptr, y.getShapeInfo(), y.specialBuffer(), y.specialShapeInfo(),
										nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
										(int*)devicePtrs[0], dimensions.size(), 
										(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2], 
										nullptr, nullptr);	

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();
 	
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5); 		

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) 
		hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execBroadcastBool_2) {
    	
	NDArray x('c', {2,3,4}, {100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100},nd4j::DataType::FLOAT32);
    NDArray y('c', {2,4},   {1,10,10,15,20,20,20,24}, nd4j::DataType::FLOAT32);
    NDArray z('c', {2,3,4}, {100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100}, nd4j::DataType::BOOL);	
	NDArray exp('c', {2,3,4}, {1, 0, 0, 0,0, 0, 0, 0,0, 1, 0, 0,0, 0, 0, 0,0, 0, 0, 0,0, 0, 0, 1}, nd4j::DataType::BOOL);
	x.linspace(1); x.syncToDevice();

    std::vector<int> dimensions = {0,2};

    // evaluate xTad data 
    shape::TAD xTad;
    xTad.init(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function       
    std::vector<std::pair<void*,size_t>> hostData;   
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));							// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));	// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));							// 2 -- xTadOffsets
	
	std::vector<void*> devicePtrs(hostData.size(), nullptr);
	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execBroadcastBool(&lc, nd4j::broadcast::EqualTo,
										nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
										nullptr, y.getShapeInfo(), y.specialBuffer(), y.specialShapeInfo(),
										nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
										(int*)devicePtrs[0], dimensions.size(), 
										(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2], 
										nullptr, nullptr);	

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();
 	
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5); 		

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) 
		hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execPairwiseTransform_1) {

	if (!Environment::getInstance()->isExperimentalBuild())
        return;
    	
	NDArray x('c', {2,2,2}, {1,5,3,7,2,6,4,8}, nd4j::DataType::INT32);
    NDArray y('c', {4,2}, {0.1,0.2,0.3,0.4,1.5,0.6,0.7,1.8}, nd4j::DataType::DOUBLE);
    NDArray z('c', {8}, {100,100,100,100,100,100,100,100}, nd4j::DataType::INT32);	
	NDArray exp('c', {8}, {0,1,2,3,3,5,6,6}, nd4j::DataType::INT32);
	x.permutei({2,1,0});	// -> {1,2,3,4,5,6,7,8}
    x.syncShape();

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);
	
	// call cuda kernel which calculates result
	NativeOpExecutioner::execPairwiseTransform(&lc, nd4j::pairwise::Subtract,
												nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
												nullptr, y.getShapeInfo(), y.specialBuffer(), y.specialShapeInfo(), 
												nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
												nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();
    
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
	
	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execPairwiseBoolTransform_1) {
    	
	NDArray x('c', {2,2,2}, {1,5,3,7,2,6,4,8}, nd4j::DataType::INT64);
    NDArray y('c', {4,2}, {0,2,0,4,0,6,0,8}, nd4j::DataType::INT64);
    NDArray z('c', {8}, {100,100,100,100,100,100,100,100}, nd4j::DataType::BOOL);	
	NDArray exp('c', {8}, {0,1,0,1,0,1,0,1}, nd4j::DataType::BOOL);
	x.permutei({2,1,0});	// -> {1,2,3,4,5,6,7,8}
	x.syncShape();
        
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);	

	// call cuda kernel which calculates result
	NativeOpExecutioner::execPairwiseBoolTransform(&lc, nd4j::pairwise::EqualTo,
													nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
													nullptr, y.getShapeInfo(), y.specialBuffer(), y.specialShapeInfo(), 
													nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
													nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();
    
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
	
	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}


////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execTransformFloat_1) {
    	
	NDArray x('c', {2,2}, {0, 6.25, 2.25, 12.25}, nd4j::DataType::DOUBLE);    
    NDArray z('c', {4}, {100,100,100,100}, nd4j::DataType::FLOAT32);	
	NDArray exp('c', {4}, {0, 1.5, 2.5, 3.5}, nd4j::DataType::FLOAT32);
	x.permutei({1,0});
	x.syncShape();
        
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execTransformFloat(&lc, nd4j::transform::Sqrt,
		nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
		nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
		nullptr, nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();
    
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execTransformFloat_2) {
    	
	NDArray x('c', {1,4}, {0, 4, 9, 16}, nd4j::DataType::INT64);
    NDArray z('c', {2,2}, {100,100,100,100}, nd4j::DataType::DOUBLE);	
	NDArray exp('c', {2,2}, {0, 2, 3, 4}, nd4j::DataType::DOUBLE);	       
	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);
	
	// call cuda kernel which calculates result
	NativeOpExecutioner::execTransformFloat(&lc, nd4j::transform::Sqrt,
		nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
		nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
		nullptr, nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();
    
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execTransformAny_1) {
    	
	NDArray x('c', {2,2}, {0, 6.25, 2.25, 12.25}, nd4j::DataType::DOUBLE);    
    NDArray z('c', {4,1}, {100,100,100,100}, nd4j::DataType::INT32);	
	NDArray exp('c', {4,1}, {0, 2, 6, 12}, nd4j::DataType::INT32);
	x.permutei({1,0});
        
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execTransformAny(&lc, nd4j::transform::Assign,
		nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
		nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
		nullptr, nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();
    
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execTransformAny_2) {
    	
	NDArray x('c', {1,4}, {0, 6.25, 2.25, 12.25}, nd4j::DataType::BFLOAT16);
    NDArray z('c', {2,2}, {100,100,100,100}, nd4j::DataType::FLOAT32);	
	NDArray exp('c', {2,2}, {0, 6.25, 2.25, 12.25}, nd4j::DataType::FLOAT32);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execTransformAny(&lc, nd4j::transform::Assign,
		nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
		nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
		nullptr, nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();
    
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execTransformStrict_1) {
    	
	NDArray x('c', {2,3}, {0,2,4,1,3,5}, nd4j::DataType::DOUBLE);
    NDArray z('c', {3,2}, {100,100,100,100,100,100}, nd4j::DataType::DOUBLE);	
	NDArray exp('c', {3,2}, {0, 3, 12, 27, 48, 75}, nd4j::DataType::DOUBLE);
	x.permutei({1,0});
	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execTransformStrict(&lc, nd4j::transform::CubeDerivative,
		nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
		nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
		nullptr, nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();
    
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execTransformStrict_2) {
    	
	NDArray x('c', {6}, {0,1,2,3,4,5}, nd4j::DataType::FLOAT32);
    NDArray z('c', {3,2}, {100,100,100,100,100,100}, nd4j::DataType::FLOAT32);	
	NDArray exp('c', {3,2}, {0, 3, 12, 27, 48, 75}, nd4j::DataType::FLOAT32);	
    	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execTransformStrict(&lc, nd4j::transform::CubeDerivative,
		nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
		nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
		nullptr, nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
	z.tickWriteDevice();
    
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execTransformSame_1) {
    
	NDArray x('c', {2,3}, {0,2.5,4.5,1.5,3.5,5.5}, nd4j::DataType::DOUBLE);	
    NDArray z('c', {1,6}, {100,100,100,100,100,100}, nd4j::DataType::DOUBLE);	
	NDArray exp('c', {1,6}, {0,2.25,6.25,12.25,20.25,30.25}, nd4j::DataType::DOUBLE);
	x.permutei({1,0});
    	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execTransformSame(&lc, nd4j::transform::Square,
		nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
		nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
		nullptr, nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();
        
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execTransformSame_2) {
    	
	NDArray x('c', {6}, {0,1,2,3,4,5}, nd4j::DataType::INT32);
    NDArray z('c', {3,2}, {100,100,100,100,100,100}, nd4j::DataType::INT32);	
	NDArray exp('c', {3,2}, {0,1,4,9,16,25}, nd4j::DataType::INT32);	
    	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);
	
	// call cuda kernel which calculates result
	NativeOpExecutioner::execTransformSame(&lc, nd4j::transform::Square,
		nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
		nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
		nullptr, nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();
    
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execTransformBool_1) {
    
	NDArray x('c', {2,3}, {0,2,4,-1,-3,-5}, nd4j::DataType::DOUBLE);	
    NDArray z('c', {1,6}, {100,100,100,100,100,100}, nd4j::DataType::BOOL);	    
	NDArray exp('c', {1,6}, {0,0,1,0,1,0}, nd4j::DataType::BOOL);
	x.permutei({1,0});
    
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execTransformBool(&lc, nd4j::transform::IsPositive,
		nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
		nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
		nullptr, nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();
         	
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execTransformBool_2) {
    	
	NDArray x('c', {6}, {0,-1,2,-3,4,-5}, nd4j::DataType::INT32);
    NDArray z('c', {3,2}, {100,100,100,100,100,100}, nd4j::DataType::BOOL);	
	NDArray exp('c', {3,2}, {0,0,1,0,1,0}, nd4j::DataType::BOOL);
    	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execTransformBool(&lc, nd4j::transform::IsPositive,
		nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
		nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
		nullptr, nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();
    
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduceFloat_1) {
    	   	
    NDArray x('c', {2,3,4}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18}, nd4j::DataType::INT32);
    NDArray z('c', {3}, {100,100,100}, nd4j::DataType::FLOAT32);
    NDArray exp('c', {3}, {2.5, 6.5, 10.5}, nd4j::DataType::FLOAT32);
    x.permutei({2,1,0});
    
    std::vector<int> dimensions = {0,2};

    // evaluate xTad data 
    shape::TAD xTad;
    xTad.init(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function       
    std::vector<std::pair<void*,size_t>> hostData;    	
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	std::vector<void*> devicePtrs(hostData.size(), nullptr);
	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduceFloat(&lc, nd4j::reduce::Mean, 
					nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
					nullptr, 
					nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
					(int*)devicePtrs[0], dimensions.size(), 
					(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)  		
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) 
		hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduceFloat_2) {
    	   	
    NDArray x('c', {2,3,4}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18}, nd4j::DataType::INT32);
    NDArray z('c', {2,4}, {100,100,100,100,100,100,100,100}, nd4j::DataType::DOUBLE);
    NDArray exp('c', {2,4}, {-1., 0., 1., 2.,11., 12., 13., 14.}, nd4j::DataType::DOUBLE);
    
    std::vector<int> dimensions = {1};

    // evaluate xTad data 
    shape::TAD xTad;
    xTad.init(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function       
    std::vector<std::pair<void*,size_t>> hostData;    	
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	std::vector<void*> devicePtrs(hostData.size(), nullptr);
	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduceFloat(&lc, nd4j::reduce::Mean, 
					nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
					nullptr, 
					nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
					(int*)devicePtrs[0], dimensions.size(), 
					(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)  		
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) 
		hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduceSame_1) {
    	   	
    NDArray x('c', {2,3,4}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18}, nd4j::DataType::INT32);
    NDArray z('c', {3}, {100,100,100}, nd4j::DataType::INT32);
    NDArray exp('c', {3}, {20, 52, 84}, nd4j::DataType::INT32);
    x.permutei({2,1,0});
    
    std::vector<int> dimensions = {0,2};

    // evaluate xTad data 
    shape::TAD xTad;
    xTad.init(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function       
    std::vector<std::pair<void*,size_t>> hostData;    	
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	std::vector<void*> devicePtrs(hostData.size(), nullptr);
	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduceSame(&lc, nd4j::reduce::Sum, 
					nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
					nullptr, 
					nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
					(int*)devicePtrs[0], dimensions.size(), 
					(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)  		
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) 
		hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduceSame_2) {
    	   	
    NDArray x('c', {2,3,4}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18}, nd4j::DataType::FLOAT32);
    NDArray z('c', {2,4}, {100,100,100,100,100,100,100,100}, nd4j::DataType::FLOAT32);
    NDArray exp('c', {2,4}, {-3., 0., 3., 6.,33., 36., 39., 42.}, nd4j::DataType::FLOAT32);
    
    std::vector<int> dimensions = {1};

    // evaluate xTad data 
    shape::TAD xTad;
    xTad.init(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function       
    std::vector<std::pair<void*,size_t>> hostData;    	
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	std::vector<void*> devicePtrs(hostData.size(), nullptr);
	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduceSame(&lc, nd4j::reduce::Sum, 
					nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
					nullptr, 
					nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
					(int*)devicePtrs[0], dimensions.size(), 
					(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)  		
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) 
		hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduceBool_1) {
    	   	
    NDArray x('c', {2,3,4}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6,-7,-8,-9,-10,-11,-12,-13,-14,-15,-16,-17,-18}, nd4j::DataType::INT32);
    NDArray z('c', {3}, {100,100,100}, nd4j::DataType::BOOL);
    NDArray exp('c', {3}, {0, 1, 1}, nd4j::DataType::BOOL);
    x.permutei({2,1,0});

    
    std::vector<int> dimensions = {0,2};

    // evaluate xTad data 
    shape::TAD xTad;
    xTad.init(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function       
    std::vector<std::pair<void*,size_t>> hostData;    	
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	std::vector<void*> devicePtrs(hostData.size(), nullptr);
	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduceBool(&lc, nd4j::reduce::IsPositive, 
					nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
					nullptr, 
					nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
					(int*)devicePtrs[0], dimensions.size(), 
					(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)  		
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) 
		hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduceBool_2) {
    	   	
    NDArray x('c', {2,3,4}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6,-7,-8,-9,-10,-11,-12,-13,-14,-15,-16,-17,-18}, nd4j::DataType::FLOAT32);
    NDArray z('c', {2,4}, {100,100,100,100,100,100,100,100}, nd4j::DataType::BOOL);
    NDArray exp('c', {2,4}, {1, 1, 1, 1, 0, 0, 0, 0}, nd4j::DataType::BOOL);
    
    std::vector<int> dimensions = {1};

    // evaluate xTad data 
    shape::TAD xTad;
    xTad.init(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function       
    std::vector<std::pair<void*,size_t>> hostData;    	
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	std::vector<void*> devicePtrs(hostData.size(), nullptr);
	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduceBool(&lc, nd4j::reduce::IsPositive, 
					nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
					nullptr, 
					nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
					(int*)devicePtrs[0], dimensions.size(), 
					(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)  		
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) 
		hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduceLong_1) {
    	   	
    NDArray x('c', {2,3,4}, {-5,0,-3,0,-1,0,1,2,3,4,5,6,7,0,9,10,11,0,13,14,0,16,0,18}, nd4j::DataType::INT32);
    NDArray z('c', {3}, {100,100,100}, nd4j::DataType::INT64);
    NDArray exp('c', {3}, {5,6,6}, nd4j::DataType::INT64);
    x.permutei({2,1,0});
    
    std::vector<int> dimensions = {0,2};

    // evaluate xTad data 
    shape::TAD xTad;
    xTad.init(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function       
    std::vector<std::pair<void*,size_t>> hostData;    	
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	std::vector<void*> devicePtrs(hostData.size(), nullptr);
	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduceLong(&lc, nd4j::reduce::CountNonZero, 
					nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
					nullptr, 
					nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
					(int*)devicePtrs[0], dimensions.size(), 
					(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)  		
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) 
		hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduceLong_2) {
    	   	
    NDArray x('c', {2,3,4}, {-5,0,-3,0,-1,0,1,2,3,4,5,6,7,0,9,10,11,0,13,14,0,16,0,18}, nd4j::DataType::FLOAT32);
    NDArray z('c', {2,4}, {100,100,100,100,100,100,100,100}, nd4j::DataType::INT64);
    NDArray exp('c', {2,4}, {3, 1, 3, 2, 2, 1, 2, 3}, nd4j::DataType::INT64);    

    std::vector<int> dimensions = {1};

    // evaluate xTad data 
    shape::TAD xTad;
    xTad.init(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function       
    std::vector<std::pair<void*,size_t>> hostData;    	
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	std::vector<void*> devicePtrs(hostData.size(), nullptr);
	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduceLong(&lc, nd4j::reduce::CountNonZero, 
					nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
					nullptr, 
					nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
					(int*)devicePtrs[0], dimensions.size(), 
					(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)  		
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) 
		hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduceFloatScalar_1) {
    	   	
    NDArray x('c', {2,3,4}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18}, nd4j::DataType::INT32);
    NDArray z('c', {}, {100}, nd4j::DataType::FLOAT32);
    NDArray exp('c', {}, {6.5}, nd4j::DataType::FLOAT32);
    x.permutei({2,1,0});
       
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);
	void* reductionPointer;
    cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer),  1024*1024); ASSERT_EQ(0, cudaResult);
    int* allocationPointer;
	cudaResult = hipMalloc(reinterpret_cast<void **>(&allocationPointer), 1024*1024); ASSERT_EQ(0, cudaResult);
	lc.setReductionPointer(reductionPointer);
	lc.setAllocationPointer(allocationPointer);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduceFloatScalar(&lc, nd4j::reduce::Mean, 
					nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
					nullptr, 
					nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo());
	
	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)  		
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduceFloatScalar_2) {
    	   	
    NDArray x('c', {2,3,4}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18}, nd4j::DataType::INT32);
    NDArray z('c', {}, {100}, nd4j::DataType::DOUBLE);
    NDArray exp('c', {}, {6.5}, nd4j::DataType::DOUBLE);
	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);
	void* reductionPointer;
    cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer),  1024*1024); ASSERT_EQ(0, cudaResult);
    int* allocationPointer;
	cudaResult = hipMalloc(reinterpret_cast<void **>(&allocationPointer), 1024*1024); ASSERT_EQ(0, cudaResult);
	lc.setReductionPointer(reductionPointer);
	lc.setAllocationPointer(allocationPointer);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduceFloatScalar(&lc, nd4j::reduce::Mean, 
					nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
					nullptr, 
					nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo());

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)  		
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduceSameScalar_1) {
    	   	
    NDArray x('c', {2,3,4}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18}, nd4j::DataType::INT32);
    NDArray z('c', {}, {100}, nd4j::DataType::INT32);
    NDArray exp('c', {}, {156}, nd4j::DataType::INT32);
    x.permutei({2,1,0});
       
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);
	void* reductionPointer;
    cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer),  1024*1024); ASSERT_EQ(0, cudaResult);
    int* allocationPointer;
	cudaResult = hipMalloc(reinterpret_cast<void **>(&allocationPointer), 1024*1024); ASSERT_EQ(0, cudaResult);
	lc.setReductionPointer(reductionPointer);
	lc.setAllocationPointer(allocationPointer);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduceSameScalar(&lc, nd4j::reduce::Sum, 
					nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
					nullptr, 
					nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo());
	
	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)  		
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduceSameScalar_2) {
    	   	
    NDArray x('c', {2,3,4}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18}, nd4j::DataType::DOUBLE);
    NDArray z('c', {}, {100}, nd4j::DataType::DOUBLE);
    NDArray exp('c', {}, {156}, nd4j::DataType::DOUBLE);
	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);
	void* reductionPointer;
    cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer),  1024*1024); ASSERT_EQ(0, cudaResult);
    int* allocationPointer;
	cudaResult = hipMalloc(reinterpret_cast<void **>(&allocationPointer), 1024*1024); ASSERT_EQ(0, cudaResult);
	lc.setReductionPointer(reductionPointer);
	lc.setAllocationPointer(allocationPointer);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduceSameScalar(&lc, nd4j::reduce::Sum, 
					nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
					nullptr, 
					nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo());

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)  		
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduceBoolScalar_1) {
    	   	
    NDArray x('c', {2,3,4}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6,-7,-8,-9,-10,-11,-12,-13,-14,-15,-16,-17,-18}, nd4j::DataType::INT32);
    NDArray z('c', {}, {100}, nd4j::DataType::BOOL);
    NDArray exp('c', {}, {1}, nd4j::DataType::BOOL);
    x.permutei({2,1,0});
    x.syncShape();    
       
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);
	void* reductionPointer;
    cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer),  1024*1024); ASSERT_EQ(0, cudaResult);
    int* allocationPointer;
	cudaResult = hipMalloc(reinterpret_cast<void **>(&allocationPointer), 1024*1024); ASSERT_EQ(0, cudaResult);
	lc.setReductionPointer(reductionPointer);
	lc.setAllocationPointer(allocationPointer);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduceBoolScalar(&lc, nd4j::reduce::IsPositive, 
					nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
					nullptr, 
					nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo());
	
	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)  		
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduceBoolScalar_2) {
    	   	
    NDArray x('c', {2,3,4}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6,-7,-8,-9,-10,-11,-12,-13,-14,-15,-16,-17,-18}, nd4j::DataType::DOUBLE);
    NDArray z('c', {}, {100}, nd4j::DataType::BOOL);
    NDArray exp('c', {}, {1}, nd4j::DataType::BOOL);
    
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);
	void* reductionPointer;
    cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer),  1024*1024); ASSERT_EQ(0, cudaResult);
    int* allocationPointer;
	cudaResult = hipMalloc(reinterpret_cast<void **>(&allocationPointer), 1024*1024); ASSERT_EQ(0, cudaResult);
	lc.setReductionPointer(reductionPointer);
	lc.setAllocationPointer(allocationPointer);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduceBoolScalar(&lc, nd4j::reduce::IsPositive, 
					nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
					nullptr, 
					nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo());

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)  		
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduceLongScalar_1) {
    	   	
    NDArray x('c', {2,3,4}, {-5,0,-3,0,-1,0,1,2,3,4,5,6,7,0,9,10,11,0,13,14,0,16,0,18}, nd4j::DataType::INT32);
    NDArray z('c', {}, {100}, nd4j::DataType::INT64);
    NDArray exp('c', {}, {17}, nd4j::DataType::INT64);
    x.permutei({2,1,0});
    x.syncShape();    
       
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);
	void* reductionPointer;
    cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer),  1024*1024); ASSERT_EQ(0, cudaResult);
    int* allocationPointer;
	cudaResult = hipMalloc(reinterpret_cast<void **>(&allocationPointer), 1024*1024); ASSERT_EQ(0, cudaResult);
	lc.setReductionPointer(reductionPointer);
	lc.setAllocationPointer(allocationPointer);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduceLongScalar(&lc, nd4j::reduce::CountNonZero, 
					nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
					nullptr, 
					nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo());
	
	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)  		
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduceLongScalar_2) {
    	   	
    NDArray x('c', {2,3,4}, {-5,0,-3,0,-1,0,1,2,3,4,5,6,7,0,9,10,11,0,13,14,0,16,0,18}, nd4j::DataType::DOUBLE);
    NDArray z('c', {}, {100}, nd4j::DataType::INT64);
    NDArray exp('c', {}, {17}, nd4j::DataType::INT64);
    
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);
	void* reductionPointer;
    cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer),  1024*1024); ASSERT_EQ(0, cudaResult);
    int* allocationPointer;
	cudaResult = hipMalloc(reinterpret_cast<void **>(&allocationPointer), 1024*1024); ASSERT_EQ(0, cudaResult);
	lc.setReductionPointer(reductionPointer);
	lc.setAllocationPointer(allocationPointer);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduceLongScalar(&lc, nd4j::reduce::CountNonZero, 
					nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
					nullptr, 
					nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo());

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)  		
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduce3TAD_1) {
    	
    NDArray x('c', {2,2,3}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6}, nd4j::DataType::FLOAT32);
    NDArray y('c', {2,2}, {1,2,3,4}, nd4j::DataType::FLOAT32);
    NDArray exp('c', {3}, {10,20,30}, nd4j::DataType::DOUBLE);
    NDArray z('c', {3}, {100,100,100}, nd4j::DataType::DOUBLE);
   
    std::vector<int> dimensions = {0,1};
    auto packX = ConstantTadHelper::getInstance()->tadForDimensions(x.shapeInfo(), dimensions);
    LaunchContext* context = x.getContext();

	x.syncToDevice();
	y.syncToDevice();
	PointersManager pm(context, "execReduce3TAD_1");
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduce3TAD(context, nd4j::reduce3::Dot,
								nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
								nullptr, 
								nullptr, y.getShapeInfo(), y.specialBuffer(), y.specialShapeInfo(), 
								nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
								nullptr, dimensions.size(),
								packX.specialShapeInfo(), packX.specialOffsets(), nullptr, nullptr);
    pm.synchronize();
//	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();
//    z.printIndexedBuffer("OutputReduce3TAD");
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduce3TAD_2) {
    	
    NDArray x('c', {2,2,3}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6}, nd4j::DataType::INT64);
    NDArray y('c', {2,3}, {1,2,3,4,5,6}, nd4j::DataType::INT64);
    NDArray exp('c', {2}, {10,73}, nd4j::DataType::FLOAT32);
    NDArray z('c', {2}, {100,100}, nd4j::DataType::FLOAT32);
   
    std::vector<int> dimensions = {0,2};

    // evaluate xTad data 
    shape::TAD xTad;
    xTad.init(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;    	
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets	
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		

	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduce3TAD(&lc, nd4j::reduce3::Dot,
								nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
								nullptr, 
								nullptr, y.getShapeInfo(), y.specialBuffer(), y.specialShapeInfo(), 
								nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
								(int*)devicePtrs[0], dimensions.size(), 
								(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2], nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduce3TAD_3) {
    	
    NDArray x('c', {2,2,3}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6}, nd4j::DataType::INT64);
    NDArray y('c', {3}, {1,2,3}, nd4j::DataType::INT64);
    NDArray exp('c', {2,2}, {-22,-4,14,32}, nd4j::DataType::FLOAT32);
    NDArray z('c', {2,2}, {100,100,100,100}, nd4j::DataType::FLOAT32);
   
    std::vector<int> dimensions = {2};

    // evaluate xTad data 
    shape::TAD xTad;
    xTad.init(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;    	
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets	
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		

	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduce3TAD(&lc, nd4j::reduce3::Dot,
								nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
								nullptr, 
								nullptr, y.getShapeInfo(), y.specialBuffer(), y.specialShapeInfo(), 
								nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
								(int*)devicePtrs[0], dimensions.size(), 
								(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2], (Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execReduce3TAD_4) {
    	
    NDArray x('c', {2,2,3}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6}, nd4j::DataType::DOUBLE);
    NDArray y('c', {2,2,3}, {10,20,30,40,50,60,70,80,90,100,110,120}, nd4j::DataType::DOUBLE);
    NDArray exp('c', {}, {1820}, nd4j::DataType::FLOAT32);
    NDArray z('c', {}, {100}, nd4j::DataType::FLOAT32);

    std::vector<int> dimensions = {0,1,2};

    // evaluate xTad data 
    shape::TAD xTad;
    xTad.init(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;    	
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets	
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduce3TAD(&lc, nd4j::reduce3::Dot,
								nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
								nullptr, 
								nullptr, y.getShapeInfo(), y.specialBuffer(), y.specialShapeInfo(), 
								nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
								(int*)devicePtrs[0], dimensions.size(), 
								(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2], (Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execSummaryStats_1) {
    	
    NDArray x('c', {2,2,3}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6}, nd4j::DataType::INT64);    
    NDArray exp('c', {}, {3.605551}, nd4j::DataType::FLOAT32);
    NDArray z('c', {}, {100}, nd4j::DataType::FLOAT32);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);
	void* reductionPointer;
    cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer), 1024*1024); ASSERT_EQ(0, cudaResult);    	
	lc.setReductionPointer(reductionPointer);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execSummaryStats(&lc, nd4j::variance::SummaryStatsStandardDeviation,
								nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
								nullptr, 								
								nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
								true);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execSummaryStats_2) {
    	
    NDArray x('c', {2,2,3}, {-5,-4,-3,-20,-1,0,1,2,3,4,5,6}, nd4j::DataType::DOUBLE);    
    NDArray exp('c', {2}, {3.405877, 9.715966}, nd4j::DataType::FLOAT32);
    NDArray z('c', {2}, {100,100}, nd4j::DataType::FLOAT32);

    std::vector<int> dimensions = {0,2};

    // evaluate xTad data 
    shape::TAD xTad;
    xTad.init(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;    	
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets	
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execSummaryStats(&lc, nd4j::variance::SummaryStatsStandardDeviation,
								nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
								nullptr, 								
								nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
								(int*)devicePtrs[0], dimensions.size(), 
								(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2],
								true);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execSummaryStats_3) {
    	
    NDArray x('c', {2,2,3}, {-5,-4,-3,-20,-1,0,1,2,3,4,5,6}, nd4j::DataType::DOUBLE);    
    NDArray exp('c', {2}, {10.606602, 2.121320}, nd4j::DataType::FLOAT32);
    NDArray z('c', {2}, {100,100}, nd4j::DataType::FLOAT32);

    std::vector<int> dimensions = {1};

    // evaluate xTad data 
    shape::TAD xTad;
    xTad.init(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;    	
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets	
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execSummaryStats(&lc, nd4j::variance::SummaryStatsStandardDeviation,
								nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
								nullptr, 								
								nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
								(int*)devicePtrs[0], dimensions.size(), 
								(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2],
								true);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execSummaryStatsScalar_1) {
    	
    NDArray x('c', {2,2,3}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6}, nd4j::DataType::INT64);
    NDArray exp('c', {}, {3.605551}, nd4j::DataType::FLOAT32);
    NDArray z('c', {}, {100}, nd4j::DataType::FLOAT32);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);
	void* reductionPointer;
    cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer), 1024*1024); ASSERT_EQ(0, cudaResult);    	
	lc.setReductionPointer(reductionPointer);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execSummaryStatsScalar(&lc, nd4j::variance::SummaryStatsStandardDeviation,
								nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
								nullptr, 								
								nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
								true);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

//////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execRandom_1) {
    	   
//    NDArray z('c', {10}, {100,0,0,0,0,0,0,0,0,0}, nd4j::DataType::DOUBLE);
    NDArray z('c', {10}, {100,0,0,0,0,0,0,0,0,100}, nd4j::DataType::FLOAT32);
    NDArray exp('c', {10}, {0.050942, -0.183229, -0.093921, 0.075469, 0.257166, -0.254838, 0.342227, -0.682188, -0.004345, 0.464633}, nd4j::DataType::FLOAT32);

    nd4j::graph::RandomGenerator gen(119,5);

	hipError_t cudaResult;
    NDArray* array = &z;
    ExtraArguments arguments({0.f, 0.5f});
    auto context = z.getContext();
    PointersManager pm(context, "tests::execRandom_1");
//    z.printIndexedBuffer("Input data");
//    z.syncToDevice();
    NativeOpExecutioner::execRandom(context, random::GaussianDistribution, &gen, array->buffer(), array->shapeInfo(), array->specialBuffer(), array->specialShapeInfo(), array->buffer(), array->shapeInfo(), array->specialBuffer(), array->specialShapeInfo(), array->buffer(), array->shapeInfo(), array->specialBuffer(), array->specialShapeInfo(), arguments.argumentsAsT(array->dataType()));
    pm.synchronize();
    z.tickWriteDevice();
//	z.printIndexedBuffer("Output Gaussian");
//    RandomLauncher::fillGaussian(context, gen, &z,  0.f, 0.5f);
//    pm.synchronize();
//    z.tickWriteDevice();
//    z.printIndexedBuffer("Output Gaussian");

//    hipStream_t stream;
//    cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
//    LaunchContext lc(&stream);
//
//	//	::execRandom(extraPointers, random::GaussianDistribution, &gen, z.buffer(), z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(), &extra);
//	// call cuda kernel which calculates result
//	NativeOpExecutioner::execRandom(&lc, nd4j::random::GaussianDistribution,
//								&gen,
//								nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
//								nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
//								nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
//								extraArguments.argumentsAsT(z.dataType()));
//
//	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
//	ASSERT_EQ(cudaResult, 0);
//    z.tickWriteDevice();
//    z.syncToHost();
//    z.printIndexedBuffer("Random1");
    ASSERT_EQ(exp, z);
// 	// verify results
// 	for (int e = 0; e < z.lengthOf(); e++)
// 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
//    hipFree(dExtraArgs);
	// free allocated global device memory
//	hipFree(dGen);
	// delete cuda stream
//	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

//////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execRandom_2) {
    	   
    NDArray x('c', {10}, {0.1,0.2,0.3,0.4,0.5,0.6,0.7,0.8,0.9,1}, nd4j::DataType::DOUBLE);    
    NDArray z('c', {2,5}, {100,100,100,100,100,100,100,100,100,100}, nd4j::DataType::DOUBLE);
    NDArray exp('c', {10}, {0., 0., 0.3, 0., 0.5, 0., 0.7, 0., 0., 1.}, nd4j::DataType::DOUBLE);
    
    ExtraArguments extraArguments({0.7});
    nd4j::graph::RandomGenerator gen(119,5);
    
//    // prepare input arrays for prepareDataForCuda function
//    std::vector<std::pair<void*,size_t>> hostData;
//	hostData.emplace_back(extraArguments.data(), extraArguments.size() * sizeof(double));		// 0 -- dimensions
//	std::vector<void*> devicePtrs(hostData.size(), nullptr);
//
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
//	hipStream_t stream;
//	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext* lc = x.getContext(); //(&stream);

	// allocate required amount of global device memory and copy host data to it 		
//	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execRandom(lc, nd4j::random::DropOut,
								&gen,
								nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
								nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 								
								extraArguments.argumentsAsT(z.dataType()));

	cudaResult = hipStreamSynchronize(*lc->getCudaStream()); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();
    z.syncToHost();
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
//	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);

	// delete cuda stream
//	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

//////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execRandom_3) {
    	       
    NDArray z('c', {10}, {100,100,100,100,100,100,100,100,100,100}, nd4j::DataType::DOUBLE);    
    NDArray exp('c', {10}, {2.373649, 2.239791, 1.887353, 2.488636, 2.068904, 2.281399, 1.828228, 2.228222, 2.490847, 1.669537}, nd4j::DataType::DOUBLE);
    
    std::vector<double> extraArguments = {1.5, 2.5};
    nd4j::graph::RandomGenerator gen(119,5);
    
    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;    		
	hostData.emplace_back(extraArguments.data(), extraArguments.size() * sizeof(double));		// 0 -- dimensions		
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execRandom(&lc, nd4j::random::UniformDistribution,
								&gen,
								nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 								
								devicePtrs[0]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

//////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests1, execRandom_4) {
    	       
    NDArray z('c', {2,5}, {1,2,3,4,5,6,7,8,9,10}, nd4j::DataType::FLOAT32);
    NDArray exp('c', {10}, {2.373649, 2.281399, 2.239791, 1.828228, 1.887353, 2.228222, 2.488636, 2.490847, 2.068904, 1.669537}, nd4j::DataType::FLOAT32);
    z.permutei({1,0});        
        
    ExtraArguments extraArguments({1.5, 2.5});
    nd4j::graph::RandomGenerator gen(119,5);
    
//    // prepare input arrays for prepareDataForCuda function
//    std::vector<std::pair<void*,size_t>> hostData;
//	hostData.emplace_back(extraArguments.data(), extraArguments.size() * sizeof(double));		// 0 -- dimensions
//	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
//	hipError_t cudaResult;
//	hipStream_t stream;
//	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
//	LaunchContext lc(&stream);
//
//	// allocate required amount of global device memory and copy host data to it
//	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
    auto context = z.getContext();
    PointersManager pm(context, "execRandom4");
	// call cuda kernel which calculates result
	NativeOpExecutioner::execRandom(context, nd4j::random::UniformDistribution,
								&gen,
								nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 								
								extraArguments.argumentsAsT(z.dataType()));

//	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();
//    z.printIndexedBuffer("Output Uniform4");
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
//	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);

	// delete cuda stream
//	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

