#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

 //
 // @author raver119@gmail.com
 //

#include "testlayers.h"
#include <NDArray.h>
#include <NDArrayFactory.h>
#include <Context.h>
#include <Node.h>
#include <graph/Variable.h>
#include <graph/VariableSpace.h>
#include <execution/LaunchContext.h>
#include <specials_cuda.h>
#include <TAD.h>
#include <ops/declarable/CustomOperations.h>

#include <hip/hip_runtime.h>

using namespace nd4j;
using namespace nd4j::graph;

class NDArrayCudaBasicsTests : public testing::Test {
public:

};

//////////////////////////////////////////////////////////////////////////
static hipError_t allocateDeviceMem(LaunchContext& lc, std::vector<void*>& devicePtrs, const std::vector<std::pair<void*,size_t>>& hostData) {

    if(devicePtrs.size() != hostData.size())
        throw std::invalid_argument("prepareDataForCuda: two input sts::vectors should same sizes !");

    hipError_t cudaResult;

    void* reductionPointer;
    cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer),  1024*1024);			if(cudaResult != 0) return cudaResult;
    int* allocationPointer;
    cudaResult = hipMalloc(reinterpret_cast<void **>(&allocationPointer), 1024*1024);			if(cudaResult != 0) return cudaResult;

    lc.setReductionPointer(reductionPointer);
    lc.setAllocationPointer(allocationPointer);
    hipStream_t stream = *lc.getCudaStream();

    for(int i = 0; i < devicePtrs.size(); ++i) {

        cudaResult = hipMalloc(reinterpret_cast<void **>(&devicePtrs[i]), hostData[i].second); if(cudaResult != 0) return cudaResult;
        hipMemcpyAsync(devicePtrs[i], hostData[i].first, hostData[i].second, hipMemcpyHostToDevice, stream);
    }
    return cudaResult;
}

TEST_F(NDArrayCudaBasicsTests, Test_Registration_1) {
    auto x = NDArrayFactory::create<int>('c', {5}, {1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create<int>('c', {5}, {5, 4, 3, 2, 1});

    ASSERT_TRUE(x.isActualOnDeviceSide());
    ASSERT_FALSE(x.isActualOnHostSide());
}

TEST_F(NDArrayCudaBasicsTests, Test_Registration_2) {
    auto x = NDArrayFactory::create<int>('c', {5});
    auto y = NDArrayFactory::create<int>('c', {5});

    ASSERT_TRUE(x.isActualOnDeviceSide());
    ASSERT_FALSE(x.isActualOnHostSide());
}

TEST_F(NDArrayCudaBasicsTests, Test_Registration_3) {
    auto x = NDArrayFactory::create<int>('c', {5}, {1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create<int>('c', {5}, {5, 4, 3, 2, 1});

    ASSERT_TRUE(x.isActualOnDeviceSide());
    ASSERT_FALSE(x.isActualOnHostSide());

    NDArray::registerSpecialUse({&x}, {&y});

    ASSERT_TRUE(x.isActualOnDeviceSide());
    ASSERT_FALSE(x.isActualOnHostSide());

    ASSERT_TRUE(y.isActualOnDeviceSide());
    ASSERT_FALSE(y.isActualOnHostSide());
}

TEST_F(NDArrayCudaBasicsTests, Test_Registration_01) {
    auto x = NDArrayFactory::create_<int>('c', {5}, {1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create_<int>('c', {5}, {5, 4, 3, 2, 1});

    ASSERT_TRUE(x->isActualOnDeviceSide());
    ASSERT_FALSE(x->isActualOnHostSide());
    delete x;
    delete y;
}

TEST_F(NDArrayCudaBasicsTests, Test_Registration_02) {
    auto x = NDArrayFactory::create_<int>('c', {5});
    auto y = NDArrayFactory::create_<int>('c', {5});

    ASSERT_TRUE(x->isActualOnDeviceSide());
    ASSERT_FALSE(x->isActualOnHostSide());
    delete x;
    delete y;
}

TEST_F(NDArrayCudaBasicsTests, Test_Registration_03) {
    auto x = NDArrayFactory::create_<int>('c', {5}, {1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create_<int>('c', {5}, {5, 4, 3, 2, 1});

    ASSERT_TRUE(x->isActualOnDeviceSide());
    ASSERT_FALSE(x->isActualOnHostSide());

    NDArray::registerSpecialUse({y}, {x});
    x->applyTransform(transform::Neg, y, nullptr);
    //ASSERT_TRUE(x->isActualOnDeviceSide());
    //ASSERT_FALSE(x->isActualOnHostSide());

    //ASSERT_TRUE(y->isActualOnDeviceSide());
    //ASSERT_TRUE(y->isActualOnHostSide());
    //y->syncToHost();
    // y->printBuffer("Negatives");
    delete x;
    delete y;
}

TEST_F(NDArrayCudaBasicsTests, Test_Cosine_1) {
    auto x = NDArrayFactory::create_<double>('c', {5}, {1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create_<double>('c', {5}, {5, 4, 3, 2, 1});

    ASSERT_TRUE(x->isActualOnDeviceSide());
    ASSERT_FALSE(x->isActualOnHostSide());

    NDArray::registerSpecialUse({y}, {x});
    x->applyTransform(transform::Cosine, y, nullptr);
    //ASSERT_TRUE(x->isActualOnDeviceSide());
    //ASSERT_FALSE(x->isActualOnHostSide());

    //ASSERT_TRUE(y->isActualOnDeviceSide());
    //ASSERT_TRUE(y->isActualOnHostSide());
    //y->syncToHost();
    y->printBuffer("Cosine");
    delete x;
    delete y;
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestAdd_1) {
    // allocating host-side arrays
    auto x = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
    auto z = NDArrayFactory::create<double>('c', { 5 }, {10, 10, 10, 10, 10});

    auto exp = NDArrayFactory::create<double>('c', { 5 }, { 2, 4, 6, 8, 10 });

    // making raw buffers
    //Nd4jPointer devBufferPtrX, devBufferPtrZ, devShapePtrX;
    //hipError_t res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrX), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrZ), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devShapePtrX), shape::shapeInfoByteLength(x.shapeInfo()));
    //ASSERT_EQ(0, res);

    Nd4jPointer nativeStream = (Nd4jPointer)malloc(sizeof(hipStream_t));
    CHECK_ALLOC(nativeStream, "Failed to allocate memory for new CUDA stream", sizeof(hipStream_t));
    hipError_t dZ = hipStreamCreate(reinterpret_cast<hipStream_t *>(&nativeStream));
    auto stream = reinterpret_cast<hipStream_t *>(&nativeStream);

    //hipMemcpyAsync(devBufferPtrX, x.buffer(), x.lengthOf() * x.sizeOfT(), hipMemcpyHostToDevice, *stream);
    //hipMemcpyAsync(devShapePtrX, x.shapeInfo(), shape::shapeInfoByteLength(x.shapeInfo()), hipMemcpyHostToDevice, *stream);

    LaunchContext lc(stream, nullptr, nullptr);
    NativeOpExecutioner::execPairwiseTransform(&lc, pairwise::Add, x.buffer(), x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(), y.buffer(), y.shapeInfo(), y.specialBuffer(), y.specialShapeInfo(), z.buffer(), z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(), nullptr);
    z.tickWriteDevice();
    auto res = hipStreamSynchronize(*stream);
    ASSERT_EQ(0, res);

    for (int e = 0; e < z.lengthOf(); e++)
        ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestAdd_2) {
    // allocating host-side arrays
    NDArray x('c', { 5 }, { 1, 2, 3, 4, 5});
    NDArray y('c', { 5 }, { 1, 2, 3, 4, 5});
    NDArray z('c', { 5 }, nd4j::DataType::DOUBLE);

    NDArray exp('c', { 5 }, { 2, 4, 6, 8, 10 });

    Nd4jPointer nativeStream = (Nd4jPointer)malloc(sizeof(hipStream_t));
    CHECK_ALLOC(nativeStream, "Failed to allocate memory for new CUDA stream", sizeof(hipStream_t));
    hipError_t dZ = hipStreamCreate(reinterpret_cast<hipStream_t *>(&nativeStream));
    auto stream = reinterpret_cast<hipStream_t *>(&nativeStream);

    LaunchContext lc(stream, *stream, nullptr, nullptr);
    NativeOpExecutioner::execPairwiseTransform(&lc, pairwise::Add, nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(), nullptr, y.shapeInfo(), y.specialBuffer(), y.specialShapeInfo(), nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(), nullptr);
    auto res = hipStreamSynchronize(*stream);
    ASSERT_EQ(0, res);

    for (int e = 0; e < z.lengthOf(); e++)
        ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestAdd_3) {
    // allocating host-side arrays
    auto x = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
    auto z = NDArrayFactory::create<double>('c', { 5 }, {10, 10, 10, 10, 10});

    auto exp = NDArrayFactory::create<double>('c', { 5 }, { 2, 4, 6, 8, 10 });

    // making raw buffers
    //Nd4jPointer devBufferPtrX, devBufferPtrZ, devShapePtrX;
    //hipError_t res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrX), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrZ), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devShapePtrX), shape::shapeInfoByteLength(x.shapeInfo()));
    //ASSERT_EQ(0, res);

    Nd4jPointer nativeStream = (Nd4jPointer)malloc(sizeof(hipStream_t));
    CHECK_ALLOC(nativeStream, "Failed to allocate memory for new CUDA stream", sizeof(hipStream_t));
    hipError_t dZ = hipStreamCreate(reinterpret_cast<hipStream_t *>(&nativeStream));
    auto stream = reinterpret_cast<hipStream_t *>(&nativeStream);

    //hipMemcpyAsync(devBufferPtrX, x.buffer(), x.lengthOf() * x.sizeOfT(), hipMemcpyHostToDevice, *stream);
    //hipMemcpyAsync(devShapePtrX, x.shapeInfo(), shape::shapeInfoByteLength(x.shapeInfo()), hipMemcpyHostToDevice, *stream);

    LaunchContext lc(stream, *stream, nullptr, nullptr);
    NativeOpExecutioner::execPairwiseTransform(&lc, pairwise::Add, x.buffer(), x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(), y.buffer(), y.shapeInfo(), y.specialBuffer(), y.specialShapeInfo(), z.buffer(), z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(), nullptr);
    z.tickWriteDevice();
    auto res = hipStreamSynchronize(*stream);
    ASSERT_EQ(0, res);
    //double* localBuffer = ;
    hipMemcpy(z.buffer(), z.specialBuffer(), z.lengthOf() * z.sizeOfT(), hipMemcpyDeviceToHost);
    res = hipStreamSynchronize(*stream);
    ASSERT_EQ(0, res);
    x.printBuffer("3X = ");
    y.printBuffer("3Y = ");
    z.printBuffer("3Result out");

    //
    // hipFree(devBufferPtrX);
    //hipFree(devBufferPtrZ);
    //hipFree(devShapePtrX);

    for (int e = 0; e < z.lengthOf(); e++) {
        ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
    }
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestAdd_4) {
    // allocating host-side arrays
    auto x = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
    auto z = NDArrayFactory::create<double>('c', { 5 });

    auto exp = NDArrayFactory::create<double>('c', { 5 }, { 2, 4, 6, 8, 10 });

    // making raw buffers
    //Nd4jPointer devBufferPtrX, devBufferPtrZ, devShapePtrX;
    //hipError_t res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrX), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrZ), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devShapePtrX), shape::shapeInfoByteLength(x.shapeInfo()));
    //ASSERT_EQ(0, res);
    x.applyPairwiseTransform(pairwise::Add, &y, &z, nullptr);

    //
    // hipFree(devBufferPtrX);
    //hipFree(devBufferPtrZ);
    //hipFree(devShapePtrX);

    for (int e = 0; e < z.lengthOf(); e++) {
        ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
    }
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestAdd_5) {
    // allocating host-side arrays
    auto x = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
    //auto z = NDArrayFactory::create<double>('c', { 5 });

    auto exp = NDArrayFactory::create<double>('c', { 5 }, { 2, 4, 6, 8, 10 });

    // making raw buffers
    //Nd4jPointer devBufferPtrX, devBufferPtrZ, devShapePtrX;
    //hipError_t res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrX), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrZ), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devShapePtrX), shape::shapeInfoByteLength(x.shapeInfo()));
    //ASSERT_EQ(0, res);
    x += y;
    //x.applyPairwiseTransform(pairwise::Add, &y, &z, nullptr);
    x.syncToHost();
    //y.printBuffer("3Y = ");
    //z.printBuffer("3Result out");

    //
    // hipFree(devBufferPtrX);
    //hipFree(devBufferPtrZ);
    //hipFree(devShapePtrX);

    for (int e = 0; e < x.lengthOf(); e++) {
        ASSERT_NEAR(exp.e<double>(e), x.e<double>(e), 1e-5);
    }
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestAdd_6) {
    // allocating host-side arrays
    auto x = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create<double>(2); //.'c', { 5 }, { 1, 2, 3, 4, 5});
    //auto z = NDArrayFactory::create<double>('c', { 5 });

    auto exp = NDArrayFactory::create<double>('c', { 5 }, { 3, 4, 5, 6, 7 });

    // making raw buffers
    //Nd4jPointer devBufferPtrX, devBufferPtrZ, devShapePtrX;
    //hipError_t res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrX), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrZ), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devShapePtrX), shape::shapeInfoByteLength(x.shapeInfo()));
    //ASSERT_EQ(0, res);
    x += y;
    //x.applyPairwiseTransform(pairwise::Add, &y, &z, nullptr);
    x.syncToHost();
    x.printBuffer("6X = ");
    //y.printBuffer("3Y = ");
    //z.printBuffer("3Result out");

    //
    // hipFree(devBufferPtrX);
    //hipFree(devBufferPtrZ);
    //hipFree(devShapePtrX);

    for (int e = 0; e < x.lengthOf(); e++) {
        ASSERT_NEAR(exp.e<double>(e), x.e<double>(e), 1e-5);
    }
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestAdd_7) {
    // allocating host-side arrays
    auto x = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
    //auto y = NDArrayFactory::create<double>(2); //.'c', { 5 }, { 1, 2, 3, 4, 5});
    //auto z = NDArrayFactory::create<double>('c', { 5 });

    auto exp = NDArrayFactory::create<double>('c', { 5 }, { 3, 4, 5, 6, 7 });

    // making raw buffers
    //Nd4jPointer devBufferPtrX, devBufferPtrZ, devShapePtrX;
    //hipError_t res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrX), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrZ), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devShapePtrX), shape::shapeInfoByteLength(x.shapeInfo()));
    //ASSERT_EQ(0, res);
    x += 2.;
    //x.applyPairwiseTransform(pairwise::Add, &y, &z, nullptr);
    x.syncToHost();
    x.printBuffer("7X = ");
    //y.printBuffer("3Y = ");
    //z.printBuffer("3Result out");

    //
    // hipFree(devBufferPtrX);
    //hipFree(devBufferPtrZ);
    //hipFree(devShapePtrX);

    for (int e = 0; e < x.lengthOf(); e++) {
        ASSERT_NEAR(exp.e<double>(e), x.e<double>(e), 1e-5);
    }
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestMultiply_1) {
    // allocating host-side arrays
    auto x = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
    auto z = NDArrayFactory::create<double>('c', { 5 });

    auto exp = NDArrayFactory::create<double>('c', { 5 }, { 1, 4, 9, 16, 25 });

    // making raw buffers
    //Nd4jPointer devBufferPtrX, devBufferPtrZ, devShapePtrX;
    //hipError_t res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrX), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrZ), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devShapePtrX), shape::shapeInfoByteLength(x.shapeInfo()));
    //ASSERT_EQ(0, res);
    x.applyPairwiseTransform(pairwise::Multiply, &y, &z, nullptr);
    // x.printBuffer("3X = ");
    // y.printBuffer("3Y = ");
    // z.printBuffer("3Result out");

    //
    // hipFree(devBufferPtrX);
    //hipFree(devBufferPtrZ);
    //hipFree(devShapePtrX);

    for (int e = 0; e < z.lengthOf(); e++) {
        ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
    }
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestMultiply_2) {
    // allocating host-side arrays
    auto x = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
    NDArray z('c', { 5 }, nd4j::DataType::DOUBLE);

    auto exp = NDArrayFactory::create<double>('c', { 5 }, { 1, 4, 9, 16, 25 });

    // making raw buffers
    //Nd4jPointer devBufferPtrX, devBufferPtrZ, devShapePtrX;
    //hipError_t res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrX), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrZ), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devShapePtrX), shape::shapeInfoByteLength(x.shapeInfo()));
    //ASSERT_EQ(0, res);
    x.applyPairwiseTransform(pairwise::Multiply, &y, &z, nullptr);
    x.printBuffer("3X = ");
    y.printBuffer("3Y = ");
    z.printBuffer("3Result out");

    //
    // hipFree(devBufferPtrX);
    //hipFree(devBufferPtrZ);
    //hipFree(devShapePtrX);

    for (int e = 0; e < z.lengthOf(); e++) {
        ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
    }
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestMultiply_3) {
    // allocating host-side arrays
    NDArray x('c', { 5 }, { 1, 2, 3, 4, 5}, nd4j::DataType::DOUBLE);
    NDArray y('c', { 5 }, { 1., 2., 3., 4., 5.}, nd4j::DataType::DOUBLE);
    auto z = NDArrayFactory::create<double>('c', { 5 });

    auto exp = NDArrayFactory::create<double>('c', { 5 }, { 1, 4, 9, 16, 25 });

    // making raw buffers
    //Nd4jPointer devBufferPtrX, devBufferPtrZ, devShapePtrX;
    //hipError_t res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrX), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrZ), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devShapePtrX), shape::shapeInfoByteLength(x.shapeInfo()));
    //ASSERT_EQ(0, res);
    x.applyPairwiseTransform(pairwise::Multiply, &y, &z, nullptr);
    //x.printBuffer("23X = ");
    //y.printBuffer("23Y = ");
    // z.printBuffer("23Result out");

    //
    // hipFree(devBufferPtrX);
    //hipFree(devBufferPtrZ);
    //hipFree(devShapePtrX);

    for (int e = 0; e < z.lengthOf(); e++) {
        ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
    }
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestMultiply_4) {
    // allocating host-side arrays
    NDArray x('c', { 5 }, { 1, 2, 3, 4, 5}, nd4j::DataType::DOUBLE);
    NDArray y('c', { 5 }, { 1., 2., 3., 4., 5.}, nd4j::DataType::DOUBLE);
    //auto z = NDArrayFactory::create<double>('c', { 5 });

    auto exp = NDArrayFactory::create<double>('c', { 5 }, { 1, 4, 9, 16, 25 });

    // making raw buffers
    //Nd4jPointer devBufferPtrX, devBufferPtrZ, devShapePtrX;
    //hipError_t res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrX), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrZ), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devShapePtrX), shape::shapeInfoByteLength(x.shapeInfo()));
    //ASSERT_EQ(0, res);
    //x.applyPairwiseTransform(pairwise::Multiply, &y, &z, nullptr);
    //x.printBuffer("23X = ");
    //y.printBuffer("23Y = ");
    x *= y;
    //x.tickWriteDevice();
    // x.printBuffer("33Result out");

    //
    // hipFree(devBufferPtrX);
    //hipFree(devBufferPtrZ);
    //hipFree(devShapePtrX);

    for (int e = 0; e < x.lengthOf(); e++) {
        ASSERT_NEAR(exp.e<double>(e), x.e<double>(e), 1e-5);
    }
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestPrimitiveNeg_01) {
    // allocating host-side arrays
    auto x = NDArrayFactory::create<int>('c', { 5 }, { 1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create<int>('c', { 5 }, { 1, 2, 3, 4, 5});
    auto exp = NDArrayFactory::create<int>('c', { 5 }, { -1, -2, -3, -4, -5 });

    auto stream = x.getContext()->getCudaStream();//reinterpret_cast<hipStream_t *>(&nativeStream);

    NativeOpExecutioner::execTransformSame(x.getContext(), transform::Neg, x.buffer(), x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(), y.buffer(), y.shapeInfo(), y.specialBuffer(), y.specialShapeInfo(), nullptr, nullptr, nullptr);
    auto res = hipStreamSynchronize(*stream);
    ASSERT_EQ(0, res);
    y.tickWriteDevice();

    // x.printBuffer("X = ");
    // y.printBuffer("Y = ");

    for (int e = 0; e < y.lengthOf(); e++) {
        ASSERT_NEAR(exp.e<int>(e), y.e<int>(e), 1e-5);
    }
}

TEST_F(NDArrayCudaBasicsTests, Test_PrimitiveNeg_2) {
    auto x = NDArrayFactory::create<double>('c', {5}, {1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create<double>('c', {5});

    ASSERT_TRUE(x.isActualOnDeviceSide());
    ASSERT_FALSE(x.isActualOnHostSide());

    x.applyTransform(transform::Neg, &y, nullptr);
    //ASSERT_TRUE(x->isActualOnDeviceSide());
    //ASSERT_FALSE(x->isActualOnHostSide());

    //ASSERT_TRUE(y->isActualOnDeviceSide());
    //ASSERT_TRUE(y->isActualOnHostSide());
    //auto res = hipStreamSynchronize(*y.getContext()->getCudaStream());
    //ASSERT_EQ(0, res);
    // y.printBuffer("Negatives2");
    //delete x;
    //delete y;
}

TEST_F(NDArrayCudaBasicsTests, Test_PrimitiveSqrt_1) { // strict
    auto x = NDArrayFactory::create<double>('c', {5}, {1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create<double>('c', {5});
    auto exp = NDArrayFactory::create<double>({1.000000, 1.414214, 1.732051, 2.000000, 2.236068});
    ASSERT_TRUE(x.isActualOnDeviceSide());
    ASSERT_FALSE(x.isActualOnHostSide());

    x.applyTransform(transform::Sqrt, &y, nullptr);
    //ASSERT_TRUE(x->isActualOnDeviceSide());
    //ASSERT_FALSE(x->isActualOnHostSide());

    //ASSERT_TRUE(y->isActualOnDeviceSide());
    //ASSERT_TRUE(y->isActualOnHostSide());
    //auto res = hipStreamSynchronize(*y.getContext()->getCudaStream());
    //ASSERT_EQ(0, res);
    ASSERT_TRUE(y.equalsTo(exp));
    //y.printBuffer("SQRT output");
    //delete x;
    //delete y;
}

TEST_F(NDArrayCudaBasicsTests, Test_PrimitiveAssign_1) { // strict
    auto x = NDArrayFactory::create<double>('c', {5}, {1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create<double>('c', {5});
    //auto exp = NDArrayFactory::create<double>({1.000000, 1.414214, 1.732051, 2.000000, 2.236068});
    //ASSERT_TRUE(x.isActualOnDeviceSide());
    //ASSERT_TRUE(x.isActualOnHostSide());

    x.applyTransform(transform::Assign, &y, nullptr);
    //ASSERT_TRUE(x->isActualOnDeviceSide());
    //ASSERT_FALSE(x->isActualOnHostSide());

    //ASSERT_TRUE(y->isActualOnDeviceSide());
    //ASSERT_TRUE(y->isActualOnHostSide());
    //auto res = hipStreamSynchronize(*y.getContext()->getCudaStream());
    //ASSERT_EQ(0, res);

    // printf("Assigned to another array\n");
    // y.printBuffer("OUput");
    ASSERT_TRUE(y.equalsTo(x));
    //y.syncToHost();
    //y.printBuffer("IsMax output");
    //delete x;
    //delete y;
}

TEST_F(NDArrayCudaBasicsTests, Test_PrimitiveCosine_1) { // strict
    auto x = NDArrayFactory::create<double>('c', {5}, {1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create<double>('c', {5});
    auto exp = NDArrayFactory::create<double>('c', {5}, {0.540302, -0.416147, -0.989992, -0.653644, 0.283662});

    ASSERT_TRUE(x.isActualOnDeviceSide());
    ASSERT_FALSE(x.isActualOnHostSide());

    x.applyTransform(transform::Cosine, &y, nullptr);
    //ASSERT_TRUE(x->isActualOnDeviceSide());
    //ASSERT_FALSE(x->isActualOnHostSide());

    //ASSERT_TRUE(y->isActualOnDeviceSide());
    //ASSERT_TRUE(y->isActualOnHostSide());
    //auto res = hipStreamSynchronize(*y.getContext()->getCudaStream());
    //ASSERT_EQ(0, res);
    ASSERT_TRUE(exp.isSameShape(y));
    ASSERT_TRUE(exp.dataType() == y.dataType());
    //y.printBuffer("Cosine2");
    //delete x;
    //delete y;
}

TEST_F(NDArrayCudaBasicsTests, Test_PrimitiveCosine_2) {
    auto x = NDArrayFactory::create<double>('c', {5}, {1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create<double>('c', {5});
    auto exp = NDArrayFactory::create<double>('c', {5}, {0.540302, -0.416147, -0.989992, -0.653644, 0.283662});

    ASSERT_TRUE(x.isActualOnDeviceSide());
    ASSERT_FALSE(x.isActualOnHostSide());
    x.applyTransform(transform::Cosine, &y, nullptr);
    //ASSERT_TRUE(x->isActualOnDeviceSide());
    //ASSERT_FALSE(x->isActualOnHostSide());

    //ASSERT_TRUE(y->isActualOnDeviceSide());
    //ASSERT_TRUE(y->isActualOnHostSide());
    //auto res = hipStreamSynchronize(*y.getContext()->getCudaStream());
    //ASSERT_EQ(0, res);
    //exp.syncToHost();
    //y.printBuffer("PrimitiveCosine2");
    //exp.printBuffer("Primitive Cosine exp");
    ASSERT_TRUE(exp.isSameShape(y));
    ASSERT_TRUE(exp.dataType() == y.dataType());
    //for (int e = 0; e < y.lengthOf(); e++) {
    //    ASSERT_NEAR(exp.e<double>(e), y.e<double>(e), 1e-5);
    //}

    ASSERT_TRUE(exp.equalsTo(y));
    //delete x;
    //delete y;
}

TEST_F(NDArrayCudaBasicsTests, Test_PrimitiveCosine_3) {
    auto x = NDArrayFactory::create<double>('c', {5}, {1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create<double>('c', {5});
    auto exp = NDArrayFactory::create<double>({0.540302, -0.416147, -0.989992, -0.653644, 0.283662});

    ASSERT_TRUE(x.isActualOnDeviceSide());
    ASSERT_FALSE(x.isActualOnHostSide());
    x.applyTransform(transform::Cosine, &y, nullptr);
    //ASSERT_TRUE(x->isActualOnDeviceSide());
    //ASSERT_FALSE(x->isActualOnHostSide());

    //ASSERT_TRUE(y->isActualOnDeviceSide());
    //ASSERT_TRUE(y->isActualOnHostSide());
    //auto res = hipStreamSynchronize(*y.getContext()->getCudaStream());
    //ASSERT_EQ(0, res);
    //exp.syncToHost();
//    y.printBuffer("PrimitiveCosine3");
//    exp.printBuffer("Primitive Cosine3 exp");
//    y.printShapeInfo("Y shape");
//    exp.printShapeInfo("Exp Shape");
    ASSERT_TRUE(exp.isSameShape(y));
//
//    for (int e = 0; e < y.lengthOf(); e++) {
//        printf("%lf == %lf\n", exp.e<double>(e), y.e<double>(e));
////        ASSERT_NEAR(exp.e<double>(e), y.e<double>(e), 1e-5);
//    }

    ASSERT_TRUE(exp.equalsTo(y));
    //delete x;
    //delete y;
}

TEST_F(NDArrayCudaBasicsTests, TestRawBroadcast_2) {

    //if (!Environment::getInstance()->isExperimentalBuild())
    //    return;

    NDArray x = NDArrayFactory::create<double>('c', {2,3,4});
    NDArray y('c', {2,4},   {10,20,30,40,50,60,70,80}, nd4j::DataType::DOUBLE);
    NDArray z('c', {2,3,4}, {100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100}, nd4j::DataType::DOUBLE);
//    NDArray exp('c', {2,3,4}, {10., 21., 32., 43., 14., 25., 36., 47., 18., 29., 40., 51., 62., 73., 84., 95., 66., 77., 88., 99., 70., 81., 92., 103}, nd4j::DataType::DOUBLE);
    NDArray exp('c', {2,3,4}, {10., 40., 90., 160., 50., 120., 210., 320., 90., 200., 330., 480., 650., 840., 1050., 1280., 850., 1080., 1330., 1600., 1050., 1320., 1610., 1920.}, nd4j::DataType::DOUBLE);
    x.linspace(1); x.syncToDevice();

    std::vector<int> dimensions = {0,2};

    // evaluate xTad data
    shape::TAD xTad;
    xTad.init(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;
    hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));							// 0 -- dimensions
    hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));	// 1 -- xTadShapeInfo
    hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));							// 2 -- xTadOffsets
    std::vector<void*> devicePtrs(hostData.size(), nullptr);

    // create cuda stream and LaunchContext
    hipError_t cudaResult;
    hipStream_t stream;
    cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
    LaunchContext lc(&stream);

    // allocate required amount of global device memory and copy host data to it
    cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);

    // call cuda kernel which calculates result
    NativeOpExecutioner::execBroadcast(&lc, nd4j::broadcast::Multiply,
                                       nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
                                       nullptr, y.getShapeInfo(), y.specialBuffer(), y.specialShapeInfo(),
                                       nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
                                       (int*)devicePtrs[0], dimensions.size(),
                                       (Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2],
                                       nullptr, nullptr);

    cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.tickWriteDevice();
    z.printBuffer("Result with Broadcast2 (multiply)");
    exp.printBuffer("Expect with Broadcast2 (multiply)");
    // verify results
    for (int e = 0; e < z.lengthOf(); e++)
        ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

    // free allocated global device memory
    for(int i = 0; i < devicePtrs.size(); ++i)
        hipFree(devicePtrs[i]);

    // delete cuda stream
    cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

TEST_F(NDArrayCudaBasicsTests, TestRawBroadcast_3) {

    //if (!Environment::getInstance()->isExperimentalBuild())
    //    return;

    NDArray x('c', {2,3,4}, nd4j::DataType::DOUBLE);
    NDArray y('c', {2,4},   {10,20,30,40,50,60,70,80}, nd4j::DataType::DOUBLE);
    NDArray z('c', {2,3,4}, {100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100}, nd4j::DataType::DOUBLE);
//    NDArray exp('c', {2,3,4}, {10., 21., 32., 43., 14., 25., 36., 47., 18., 29., 40., 51., 62., 73., 84., 95., 66., 77., 88., 99., 70., 81., 92., 103}, nd4j::DataType::DOUBLE);
    NDArray exp('c', {2,3,4}, {10., 40., 90., 160., 50., 120., 210., 320., 90., 200., 330., 480., 650., 840., 1050., 1280., 850., 1080., 1330., 1600., 1050., 1320., 1610., 1920.}, nd4j::DataType::DOUBLE);
    x.linspace(1); x.syncToDevice();

    std::vector<int> dimensions = {0,2};

    // evaluate xTad data
    shape::TAD xTad;
    xTad.init(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;
    hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));							// 0 -- dimensions
    hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));	// 1 -- xTadShapeInfo
    hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));							// 2 -- xTadOffsets
    std::vector<void*> devicePtrs(hostData.size(), nullptr);

    // create cuda stream and LaunchContext
    hipError_t cudaResult;
    //hipStream_t stream;
    //cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
    LaunchContext* pLc = x.getContext();//(&stream);
    hipStream_t* stream = pLc->getCudaStream();
    // allocate required amount of global device memory and copy host data to it
//    cudaResult = allocateDeviceMem(*pLc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
    for(int i = 0; i < devicePtrs.size(); ++i) {

        cudaResult = hipMalloc(reinterpret_cast<void **>(&devicePtrs[i]), hostData[i].second); ASSERT_EQ(0, cudaResult);
        hipMemcpyAsync(devicePtrs[i], hostData[i].first, hostData[i].second, hipMemcpyHostToDevice, *stream);
    }

    NDArray::registerSpecialUse({&z}, {&x, &y});
    // call cuda kernel which calculates result
    NativeOpExecutioner::execBroadcast(pLc, nd4j::broadcast::Multiply,
                                       nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
                                       nullptr, y.getShapeInfo(), y.specialBuffer(), y.specialShapeInfo(),
                                       nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
                                       (int*)devicePtrs[0], dimensions.size(),
                                       (Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2],
                                       nullptr, nullptr);

    //cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    //z.syncToHost();
    z.printBuffer("Result with Broadcast3 (multiply)");
    // verify results
    for (int e = 0; e < z.lengthOf(); e++)
        ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

    // free allocated global device memory
    for(int i = 0; i < devicePtrs.size(); ++i)
        hipFree(devicePtrs[i]);
    ASSERT_TRUE(exp.equalsTo(z));
    // delete cuda stream
    //cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}


TEST_F(NDArrayCudaBasicsTests, TestBroadcastMultiply_1) {
    // allocating host-side arrays
    NDArray x('c', { 2, 3 }, { 1, 2, 3, 4, 5, 6}, nd4j::DataType::DOUBLE);
    NDArray y = NDArrayFactory::create<double>(3.); //'c', { 3 }, { 2., 3., 4.}, nd4j::DataType::DOUBLE);
    //auto z = NDArrayFactory::create<double>('c', { 5 });

    auto exp = NDArrayFactory::create<double>('c', { 2, 3 }, { 3, 6, 9, 12, 15, 18 });

    // making raw buffers
    //Nd4jPointer devBufferPtrX, devBufferPtrZ, devShapePtrX;
    //hipError_t res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrX), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrZ), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devShapePtrX), shape::shapeInfoByteLength(x.shapeInfo()));
    //ASSERT_EQ(0, res);
    //x.applyPairwiseTransform(pairwise::Multiply, &y, &z, nullptr);
    //x.printBuffer("23X = ");
    //y.printBuffer("23Y = ");
    x *= y;
    //x.syncToHost();
    x.printBuffer("54Result out");

    //
    // hipFree(devBufferPtrX);
    //hipFree(devBufferPtrZ);
    //hipFree(devShapePtrX);
    ASSERT_TRUE(exp.equalsTo(x));
//    for (int e = 0; e < x.lengthOf(); e++) {
//        ASSERT_NEAR(exp.e<double>(e), x.e<double>(e), 1e-5);
//    }
}

TEST_F(NDArrayCudaBasicsTests, TestBroadcastMultiply_01) {
    // allocating host-side arrays
    NDArray x('c', { 2, 3 }, { 1, 2, 3, 4, 5, 6}, nd4j::DataType::DOUBLE);
    NDArray y = NDArrayFactory::create<double>(3.); //'c', { 3 }, { 2., 3., 4.}, nd4j::DataType::DOUBLE);
    auto z = NDArrayFactory::create<double>('c', { 2, 3 });

    auto exp = NDArrayFactory::create<double>('c', { 2, 3 }, { 3, 6, 9, 12, 15, 18 });

    // making raw buffers
    //Nd4jPointer devBufferPtrX, devBufferPtrZ, devShapePtrX;
    //hipError_t res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrX), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrZ), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devShapePtrX), shape::shapeInfoByteLength(x.shapeInfo()));
    //ASSERT_EQ(0, res);
    //x.applyPairwiseTransform(pairwise::Multiply, &y, &z, nullptr);
    //x.printBuffer("23X = ");
    //y.printBuffer("23Y = ");
    x.applyTrueBroadcast(BroadcastOpsTuple::Multiply(), &y, &z);// *= y;
    // z.printBuffer("53Result out");

    //
    // hipFree(devBufferPtrX);
    //hipFree(devBufferPtrZ);
    //hipFree(devShapePtrX);
    ASSERT_TRUE(exp.equalsTo(z));

//    for (int e = 0; e < x.lengthOf(); e++) {
//        ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
//    }
}

TEST_F(NDArrayCudaBasicsTests, TestBroadcastMultiply_02) {
    // allocating host-side arrays
    auto x = NDArrayFactory::create<double>('c', { 2, 3 }, { 1, 2, 3, 4, 5, 6}); //, nd4j::DataType::DOUBLE);
    auto y = NDArrayFactory::create<double>('c', {2,3}, {3, 3, 3, 3, 3, 3}); //'c', { 3 }, { 2., 3., 4.}, nd4j::DataType::DOUBLE);
    auto z = NDArrayFactory::create<double>('c', { 2, 3 });

    auto exp = NDArrayFactory::create<double>('c', { 2, 3 }, { 3, 6, 9, 12, 15, 18 });
    //if (x.isActualOnHostSide() && !x.isActualOnDeviceSide())
    // making raw buffers
    //Nd4jPointer devBufferPtrX, devBufferPtrZ, devShapePtrX;
    //hipError_t res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrX), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrZ), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devShapePtrX), shape::shapeInfoByteLength(x.shapeInfo()));
    //ASSERT_EQ(0, res);
    //x.applyPairwiseTransform(pairwise::Multiply, &y, &z, nullptr);
    //x.printBuffer("23X = ");
    //y.printBuffer("23Y = ");
    x.applyTrueBroadcast(BroadcastOpsTuple::Multiply(), &y, &z);// *= y;

    // z.printBuffer("52Result out");

    //
    // hipFree(devBufferPtrX);
    //hipFree(devBufferPtrZ);
    //hipFree(devShapePtrX);
    ASSERT_TRUE(exp.equalsTo(z));

//    for (int e = 0; e < x.lengthOf(); e++) {
//        ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
//    }
}

TEST_F(NDArrayCudaBasicsTests, TestBroadcastMultiply_002) {
    // allocating host-side arrays
    auto x = NDArrayFactory::create<double>('c', { 2, 3 }, { 1, 2, 3, 4, 5, 6}); //, nd4j::DataType::DOUBLE);
    auto y = NDArrayFactory::create<double>('c', {2, 3}, {2., 3., 3., 3., 3., 3.}); //'c', { 3 }, { 2., 3., 4.}, nd4j::DataType::DOUBLE);
    auto z = NDArrayFactory::create<double>('c', { 2, 3 });

    auto exp = NDArrayFactory::create<double>('c', { 2, 3 }, { 2, 6, 9, 12, 15, 18 });
    //if (x.isActualOnHostSide() && !x.isActualOnDeviceSide())
    // making raw buffers
    //Nd4jPointer devBufferPtrX, devBufferPtrZ, devShapePtrX;
    //hipError_t res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrX), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrZ), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devShapePtrX), shape::shapeInfoByteLength(x.shapeInfo()));
    //ASSERT_EQ(0, res);
    //x.applyPairwiseTransform(pairwise::Multiply, &y, &z, nullptr);
    //x.printBuffer("23X = ");
    //y.printBuffer("23Y = ");
    x.applyPairwiseTransform(pairwise::Multiply, &y, &z);// *= y;

    // z.printBuffer("51Result out");

    //
    // hipFree(devBufferPtrX);
    //hipFree(devBufferPtrZ);
    //hipFree(devShapePtrX);
    ASSERT_TRUE(exp.equalsTo(z));

//    for (int e = 0; e < x.lengthOf(); e++) {
//        ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
//    }
}

////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestBroadcastRaw_1) {

    //if (!Environment::getInstance()->isExperimentalBuild())
    //    return;

    NDArray x('c', {2,3,4}, {100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100}, nd4j::DataType::INT32);
    NDArray y('c', {3},   {10, 20, 30}, nd4j::DataType::INT64);
    NDArray z('c', {2,3,4}, {100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100}, nd4j::DataType::INT32);
    NDArray exp('c', {2,3,4}, {10, 11, 12, 13,24, 25, 26, 27,38, 39, 40, 41,22, 23, 24, 25,36, 37, 38, 39,50, 51, 52, 53}, nd4j::DataType::INT32);
    //real output [10, 11, 12, 13, 4, 5, 6, 7, 28, 29, 30, 31, 22, 23, 24, 25, 16, 17, 18, 19, 40, 41, 42, 43]
    x.linspace(0); x.syncToDevice();

    std::vector<int> dimensions = {1};

    // evaluate xTad data
    shape::TAD xTad;
    xTad.init(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;
    hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(Nd4jLong));							// 0 -- dimensions
    hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));	// 1 -- xTadShapeInfo
    hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));							// 2 -- xTadOffsets
    std::vector<void*> devicePtrs(hostData.size(), nullptr);

    // create cuda stream and LaunchContext
    hipError_t cudaResult;
    hipStream_t* stream = x.getContext()->getCudaStream();
    LaunchContext* pLc = x.getContext();

    // allocate required amount of global device memory and copy host data to it
    //cudaResult = allocateDeviceMem(*pLc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
    for(size_t i = 0; i < devicePtrs.size(); ++i) {
        nd4j_printf("Allocation of %i bytes with device\n", hostData[i].second)
        cudaResult = hipMalloc(&devicePtrs[i], hostData[i].second); //if(cudaResult != 0) return cudaResult;
        ASSERT_EQ(cudaResult, 0);
        hipMemcpy(devicePtrs[i], hostData[i].first, hostData[i].second, hipMemcpyHostToDevice);
    }

    // call cuda kernel which calculates result
    NativeOpExecutioner::execBroadcast(pLc, nd4j::broadcast::Add,
                                       nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
                                       nullptr, y.getShapeInfo(), y.specialBuffer(), y.specialShapeInfo(),
                                       nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
                                       (int*)devicePtrs[0], dimensions.size(),
                                       (Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2],
                                       nullptr, nullptr);

    cudaResult = hipStreamSynchronize(*stream); ASSERT_EQ(0, cudaResult);

    // x.printIndexedBuffer(" X");
    // y.printIndexedBuffer("+Y");
    // z.printBuffer("ADD broadcasted output");
    // verify results
   // for (int e = 0; e < z.lengthOf(); e++)
   //     ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

    // free allocated global device memory
    for(int i = 0; i < devicePtrs.size(); ++i)
        hipFree(devicePtrs[i]);

    // delete cuda stream
    //cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

TEST_F(NDArrayCudaBasicsTests, TestBroadcastMultiply) {
    // allocating host-side arrays
    NDArray x('c', { 2, 3 }, { 1, 2, 3, 4, 5, 6}, nd4j::DataType::DOUBLE);
    NDArray y('c', { 3 }, { 2., 3., 4.}, nd4j::DataType::DOUBLE);
    //auto z = NDArrayFactory::create<double>('c', { 5 });

    auto exp = NDArrayFactory::create<double>('c', { 2, 3 }, { 2, 6, 12, 8, 15, 24 });

    // making raw buffers
    //Nd4jPointer devBufferPtrX, devBufferPtrZ, devShapePtrX;
    //hipError_t res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrX), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrZ), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devShapePtrX), shape::shapeInfoByteLength(x.shapeInfo()));
    //ASSERT_EQ(0, res);
    //x.applyPairwiseTransform(pairwise::Multiply, &y, &z, nullptr);
    //x.printBuffer("23X = ");
    //y.printBuffer("23Y = ");
    x *= y;
    x.printBuffer("55Result out");

    //
    // hipFree(devBufferPtrX);
    //hipFree(devBufferPtrZ);
    //hipFree(devShapePtrX);

    //for (int e = 0; e < x.lengthOf(); e++) {
    //    ASSERT_NEAR(exp.e<double>(e), x.e<double>(e), 1e-5);
    //}
}


TEST_F(NDArrayCudaBasicsTests, TestBroadcastMultiply_2) {
    // allocating host-side arrays
    NDArray x('c', { 2, 3 }, { 1, 2, 3, 4, 5, 6}, nd4j::DataType::DOUBLE);
    NDArray y('c', { 3 }, { 2., 3., 4.}, nd4j::DataType::DOUBLE);
    //auto z = NDArrayFactory::create<double>('c', { 5 });

    auto exp = NDArrayFactory::create<double>('c', { 2, 3 }, { 11,12, 13,14, 15, 16 });
    auto expZ = NDArrayFactory::create<double>('c', { 2, 3 }, { 2, 6, 12, 8, 15, 24 });

    // making raw buffers
    //Nd4jPointer devBufferPtrX, devBufferPtrZ, devShapePtrX;
    //hipError_t res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrX), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrZ), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devShapePtrX), shape::shapeInfoByteLength(x.shapeInfo()));
    //ASSERT_EQ(0, res);
    //x.applyPairwiseTransform(pairwise::Multiply, &y, &z, nullptr);
    //x.printBuffer("23X = ");
    //y.printBuffer("23Y = ");
    //void NDArray::applyTrueBroadcast(nd4j::BroadcastOpsTuple op, const NDArray* other, NDArray* target, const bool checkTargetShape, ExtraArguments *extraArgs)
    x.applyTrueBroadcast(BroadcastOpsTuple::Multiply(), &y, &exp);
    exp.printBuffer("56Result out");

    //
    // hipFree(devBufferPtrX);
    //hipFree(devBufferPtrZ);
    //hipFree(devShapePtrX);

    //for (int e = 0; e < x.lengthOf(); e++) {
    //    ASSERT_NEAR(exp.e<double>(e), x.e<double>(e), 1e-5);
    //}
    ASSERT_TRUE(exp.equalsTo(expZ));

}


//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestReduceSum_1) {
    // allocating host-side arrays
    auto x = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create<double>(15);
    auto exp = NDArrayFactory::create<double>(15);

    auto stream = x.getContext()->getCudaStream();//reinterpret_cast<hipStream_t *>(&nativeStream);

    NativeOpExecutioner::execReduceSameScalar(x.getContext(), reduce::Sum, x.buffer(), x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(), nullptr, y.buffer(), y.shapeInfo(), y.specialBuffer(), y.specialShapeInfo());
    auto res = hipStreamSynchronize(*stream);
    ASSERT_EQ(0, res);
    y.syncToHost();

    x.printBuffer("X = ");
    y.printBuffer("Y = ");
    ASSERT_NEAR(y.e<double>(0), 15, 1e-5);
}

//////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestDup1) {

    NDArray array('c', {2,3}, {1,2,3,4,5,6});
    array.printBuffer("Array at start");
    auto arrC = array.dup('c');
    auto arrF = array.dup('f');
    // arrC->printBuffer("arrC");

    // arrF->printBuffer("arrF");
    //arrC->printShapeInfo("C shape");
    //arrF->printShapeInfo("F shape");

    ASSERT_TRUE(array.equalsTo(arrF));
    ASSERT_TRUE(array.equalsTo(arrC));

    ASSERT_TRUE(arrF->equalsTo(arrC));

    delete arrC;
    delete arrF;
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, equalsTo_1) {

    NDArray x('c', {2,5}, {1,2,3,4,5,6,7,8,9,10}, nd4j::DataType::DOUBLE);
    NDArray y('c', {2,5}, {1,2,3,4,5,6,7,8,9,10}, nd4j::DataType::DOUBLE);

    ASSERT_TRUE(x.equalsTo(y));

    x.permutei({1,0});
    y.permutei({1,0});

    ASSERT_TRUE(x.equalsTo(y));
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, equalsTo_2) {

    NDArray x('c', {2,5}, {1,2,3,4,5,6,7,8,10,10}, nd4j::DataType::DOUBLE);
    NDArray y('c', {2,5}, {1,2,5,4,5,6,7,8,9,10}, nd4j::DataType::DOUBLE);

    ASSERT_FALSE(x.equalsTo(y));

    x.permutei({1,0});
    y.permutei({1,0});

    ASSERT_FALSE(x.equalsTo(y));
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, equalsTo_3) {

    NDArray x('c', {2,5}, {1,2,3,4,5,6,7,8,9,10}, nd4j::DataType::DOUBLE);
    NDArray y('c', {2,5}, {1.f,2.f,3.f,4.f,5.f,6.f,7.f,8.f,9.f,10.f}, nd4j::DataType::FLOAT32);

    ASSERT_FALSE(x.equalsTo(y));

    x.permutei({1,0});
    y.permutei({1,0});

    ASSERT_FALSE(x.equalsTo(y));
}

////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, applyReduce3_1) {

    NDArray x('c', {2,3,4}, {-10,-9,-8,-7,-6,-5,-4,-3,-2,-1,0,1,2,3,4,5,6,7,8,9,10,11,12,13}, nd4j::DataType::INT32);
    NDArray x2('c', {2,3,4}, {-10,-9,-8,-7,-6,-5,-4,-3,-2,-1,0,1,2,3,4,5,6,7,8,9,10,11,12,13}, nd4j::DataType::INT32);
    NDArray y('c', {2,3,4}, {-2,3,-4,5,-2,3,-4,5,-2,3,-4,5,-2,3,-4,5,-2,3,-4,5,-2,3,-4,5}, nd4j::DataType::INT32);
    NDArray k('c', {2,3}, {-2,3,-4,5,-2,3}, nd4j::DataType::INT32);
    NDArray k2('c', {3,2}, {-2,3,-4,5,-2,3}, nd4j::DataType::INT32);

    NDArray exp1('c', {3}, {4., 20., 36.}, nd4j::DataType::FLOAT32);
    NDArray exp2('c', {2,3}, {-10., -2., 6.,14., 22., 30.}, nd4j::DataType::FLOAT32);
    NDArray exp3('c', {4}, {38., 41., 44., 47.}, nd4j::DataType::FLOAT32);
    NDArray exp4('c', {4}, {114., 117., 120., 123.}, nd4j::DataType::FLOAT32);


    NDArray* z = x.applyReduce3(nd4j::reduce3::Dot, &y, {0,2});
    ASSERT_TRUE(z->equalsTo(&exp1));
    delete z;

    z = x.applyReduce3(nd4j::reduce3::Dot, &k, {0,1});
    ASSERT_TRUE(z->equalsTo(&exp3));
    delete z;

    x.permutei({0,2,1});
    y.permutei({0,2,1});

    z = y.applyReduce3(nd4j::reduce3::Dot, &x, {1});
    ASSERT_TRUE(z->equalsTo(&exp2));
    // printCudaGlobal<float><<<1,1,0, *y.getContext()->getCudaStream()>>>(z->specialBuffer(), 6);
    delete z;

    x2.permutei({1,0,2});

    z = x2.applyReduce3(nd4j::reduce3::Dot, &k2, {0,1});
    ASSERT_TRUE(z->equalsTo(&exp4));
    delete z;
}

////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, applyReduce3_2) {

    NDArray x('c', {2,3,4}, {-10,-9,-8.5,-7,-6,-5,-4,-3,-2,-1,0,1,2,3,4,5,6,7,8,9,10,11,12,13}, nd4j::DataType::DOUBLE);
    NDArray x2('c', {2,3,4}, {-10,-9,-8,-7,-6,-5,-4,-3,-2,-1,0.5,1,2,3,4,5,6,7,8,9,10,11,12,13}, nd4j::DataType::DOUBLE);
    NDArray y('c', {2,3,4}, {-2,3,-4,5,-2,3,-4,5,-2,3,-4,5,-2.5,3,-4,5,-2,3,-4,5,-2,3,-4,5}, nd4j::DataType::DOUBLE);
    NDArray k('c', {2,3}, {-2,3,-4,5.5,-2,3}, nd4j::DataType::DOUBLE);
    NDArray k2('c', {3,2}, {-2,3,-4,5,-2,3.5}, nd4j::DataType::DOUBLE);

    NDArray exp1('c', {3}, {5., 20., 36.}, nd4j::DataType::DOUBLE);
    NDArray exp2('c', {2,3}, {-8., -2., 6., 13., 22., 30.}, nd4j::DataType::DOUBLE);
    NDArray exp3('c', {4}, {39., 42.5, 47., 49.5}, nd4j::DataType::DOUBLE);
    NDArray exp4('c', {4}, {119., 122.5, 125., 129.5}, nd4j::DataType::DOUBLE);

    NDArray* z = x.applyReduce3(nd4j::reduce3::Dot, &y, {0,2});
    ASSERT_TRUE(z->equalsTo(&exp1));
    delete z;

    z = x.applyReduce3(nd4j::reduce3::Dot, &k, {0,1});
    ASSERT_TRUE(z->equalsTo(&exp3));
    delete z;

    x.permutei({0,2,1});
    y.permutei({0,2,1});

    z = y.applyReduce3(nd4j::reduce3::Dot, &x, {1});
    ASSERT_TRUE(z->equalsTo(&exp2));
    // printCudaGlobal<float><<<1,1,0, *y.getContext()->getCudaStream()>>>(z->specialBuffer(), 6);
    delete z;

    x2.permutei({1,0,2});

    z = x2.applyReduce3(nd4j::reduce3::Dot, &k2, {0,1});
    ASSERT_TRUE(z->equalsTo(&exp4));
    delete z;
}

////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, applyReduce3_3) {

    NDArray x1('c', {2,2,2}, {1,2,3,4,5,6,7,8}, nd4j::DataType::INT32);
    NDArray x2('c', {2,2,2}, {-1,-2,-3,-4,-5,-6,-7,-8}, nd4j::DataType::INT32);
    NDArray x3('c', {3,2}, {1.5,1.5,1.5,1.5,1.5,1.5}, nd4j::DataType::DOUBLE);
    NDArray x4('c', {3,2}, {1,2,3,4,5,6}, nd4j::DataType::DOUBLE);

    NDArray exp1('c', {}, {-204}, nd4j::DataType::FLOAT32);
    NDArray exp2('c', {}, {31.5}, nd4j::DataType::DOUBLE);


    auto z = x1.applyReduce3(reduce3::Dot, &x2);
    ASSERT_TRUE(z->equalsTo(&exp1));
    delete z;

    z = x3.applyReduce3(reduce3::Dot, &x4);
    ASSERT_TRUE(z->equalsTo(&exp2));
    delete z;

    x1.permutei({2,1,0});
    x2.permutei({2,1,0});
    x3.permutei({1,0});
    x4.permutei({1,0});

    z = x1.applyReduce3(reduce3::Dot, &x2);
    ASSERT_TRUE(z->equalsTo(&exp1));
    delete z;

    z = x3.applyReduce3(reduce3::Dot, &x4);
    ASSERT_TRUE(z->equalsTo(&exp2));
    delete z;
}

////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, applyAllReduce3_1) {

    NDArray x1('c', {2,3,2}, {1,2,3,4,5,6,7,8,-1,-2,-3,-4,}, nd4j::DataType::INT32);
    NDArray x2('c', {2,2,2}, {-1,-2,-3,-4,-5,-6,-7,-8}, nd4j::DataType::INT32);
    NDArray x3('c', {3,2}, {1.5,1.5,1.5,1.5,1.5,1.5}, nd4j::DataType::DOUBLE);
    NDArray x4('c', {3,2}, {1,2,3,4,5,6}, nd4j::DataType::DOUBLE);

    NDArray exp1('c', {3,2}, {-88., -124., 6., -2., 22., 14.}, nd4j::DataType::FLOAT32);
    NDArray exp2('c', {6,4}, {-36., -44., -52., -60.,-42., -52., -62., -72.,2., 0., -2., -4.,6., 4., 2., 0.,10., 8., 6., 4.,14., 12., 10., 8.}, nd4j::DataType::FLOAT32);
    NDArray exp3('c', {1,1}, {31.5}, nd4j::DataType::DOUBLE);
    NDArray exp4('c', {3,3}, {4.5, 10.5, 16.5,4.5, 10.5, 16.5,4.5, 10.5, 16.5}, nd4j::DataType::DOUBLE);

    auto z = x1.applyAllReduce3(reduce3::Dot, &x2, {0,2});
    ASSERT_TRUE(z->equalsTo(&exp1));
    delete z;

    z = x1.applyAllReduce3(reduce3::Dot, &x2, {0});
    ASSERT_TRUE(z->equalsTo(&exp2));
    delete z;

    z = x3.applyAllReduce3(reduce3::Dot, &x4, {0,1});
    ASSERT_TRUE(z->equalsTo(&exp3));
    delete z;

    z = x3.applyAllReduce3(reduce3::Dot, &x4, {1});
    // z->syncToHost();
    // z->printShapeInfo();
    // z->printIndexedBuffer();
    ASSERT_TRUE(z->equalsTo(&exp4));
    delete z;

    x1.permutei({2,1,0});
    x2.permutei({2,1,0});
    x3.permutei({1,0});
    x4.permutei({1,0});

    z = x1.applyAllReduce3(reduce3::Dot, &x2, {0,2});
    ASSERT_TRUE(z->equalsTo(&exp1));
    delete z;

    z = x3.applyAllReduce3(reduce3::Dot, &x4, {0});
    ASSERT_TRUE(z->equalsTo(&exp4));
    delete z;
}

//////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, applyIndexReduce_test1) {

    NDArray x('c', {2,3}, {0, 10, 1, 2, 2.5,-4}, nd4j::DataType::DOUBLE);

    NDArray scalar('c', {}, {100}, nd4j::DataType::INT64);
    NDArray vec1('c', {2}, {100,100}, nd4j::DataType::INT64);
    NDArray vec2('c', {3}, {100,100,100}, nd4j::DataType::INT64);

    NDArray exp1('c', {}, {1}, nd4j::DataType::INT64);
    NDArray exp2('c', {2}, {1,1}, nd4j::DataType::INT64);
    NDArray exp3('c', {3}, {1,0,0}, nd4j::DataType::INT64);

    NDArray exp4('c', {}, {2}, nd4j::DataType::INT64);
    NDArray exp5('c', {2}, {1,1}, nd4j::DataType::INT64);
    NDArray exp6('c', {3}, {1,0,0}, nd4j::DataType::INT64);

    x.applyIndexReduce(nd4j::indexreduce::IndexMax, &scalar, {0,1});
    ASSERT_TRUE(scalar.equalsTo(&exp1));

    x.applyIndexReduce(nd4j::indexreduce::IndexMax, &vec1, {1});
    ASSERT_TRUE(vec1.equalsTo(&exp2));

    x.applyIndexReduce(nd4j::indexreduce::IndexMax, &vec2, {0});
    ASSERT_TRUE(vec2.equalsTo(&exp3));

    x.permutei({1,0});

    x.applyIndexReduce(nd4j::indexreduce::IndexMax, &scalar, {0,1});
    ASSERT_TRUE(scalar.equalsTo(&exp4));

    x.applyIndexReduce(nd4j::indexreduce::IndexMax, &vec1, {0});
    ASSERT_TRUE(vec1.equalsTo(&exp5));

    x.applyIndexReduce(nd4j::indexreduce::IndexMax, &vec2, {1});
    ASSERT_TRUE(vec2.equalsTo(&exp6));
}


//////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, applyIndexReduce_test2) {

    NDArray x('c', {2,3}, {0, 10, 1, 2, 2.5,-4}, nd4j::DataType::DOUBLE);

    NDArray exp1('c', {}, {1}, nd4j::DataType::INT64);
    NDArray exp2('c', {2}, {1,1}, nd4j::DataType::INT64);
    NDArray exp3('c', {3}, {1,0,0}, nd4j::DataType::INT64);

    NDArray exp4('c', {}, {2}, nd4j::DataType::INT64);
    NDArray exp5('c', {2}, {1,1}, nd4j::DataType::INT64);
    NDArray exp6('c', {3}, {1,0,0}, nd4j::DataType::INT64);

    auto z = x.applyIndexReduce(nd4j::indexreduce::IndexMax, {0,1});
    ASSERT_TRUE(z->equalsTo(&exp1));
    delete z;

    z = x.applyIndexReduce(nd4j::indexreduce::IndexMax, {1});
    ASSERT_TRUE(z->equalsTo(&exp2));
    delete z;

    z = x.applyIndexReduce(nd4j::indexreduce::IndexMax, {0});
    ASSERT_TRUE(z->equalsTo(&exp3));
    delete z;

    x.permutei({1,0});

    z = x.applyIndexReduce(nd4j::indexreduce::IndexMax, {0,1});
    ASSERT_TRUE(z->equalsTo(&exp4));
    delete z;

    z = x.applyIndexReduce(nd4j::indexreduce::IndexMax, {0});
    ASSERT_TRUE(z->equalsTo(&exp5));
    delete z;

    z = x.applyIndexReduce(nd4j::indexreduce::IndexMax, {1});
    ASSERT_TRUE(z->equalsTo(&exp6));
    delete z;
}

////////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, reduceAlongDimension_float_test1) {

    NDArray x('c', {2,3,2}, {1,2,3,4,5,6,7,8,-1,-2,-3,-4,}, nd4j::DataType::INT32);

    NDArray z1('c', {}, {100}, nd4j::DataType::DOUBLE);
    NDArray z2('c', {2,2}, {100,100,100,100}, nd4j::DataType::FLOAT32);
    NDArray z3('c', {3}, {100,100,100}, nd4j::DataType::DOUBLE);
    NDArray z4('c', {3,2}, {100,100,100,100,100,100}, nd4j::DataType::FLOAT32);
    NDArray z5('c', {2}, {100,100}, nd4j::DataType::FLOAT32);

    NDArray exp1('c', {}, {2.166667}, nd4j::DataType::DOUBLE);
    NDArray exp2('c', {2,2}, {3,4,1,0.666667}, nd4j::DataType::FLOAT32);
    NDArray exp3('c', {3}, {4.5,1,1}, nd4j::DataType::DOUBLE);
    NDArray exp4('c', {3,2}, {4,5,1,1,1,1}, nd4j::DataType::FLOAT32);
    NDArray exp5('c', {2}, {3.5,0.833333}, nd4j::DataType::FLOAT32);

    x.reduceAlongDimension(nd4j::reduce::Mean, &z1, {0,1,2});
    ASSERT_TRUE(z1.equalsTo(&exp1));

    x.reduceAlongDimension(nd4j::reduce::Mean, &z2, {1});
    ASSERT_TRUE(z2.equalsTo(&exp2));

    x.reduceAlongDimension(nd4j::reduce::Mean, &z3, {0,2});
    ASSERT_TRUE(z3.equalsTo(&exp3));

    x.permutei({1,0,2});    // 3x2x2

    x.reduceAlongDimension(nd4j::reduce::Mean, &z1, {0,1,2});
    ASSERT_TRUE(z1.equalsTo(&exp1));

    x.reduceAlongDimension(nd4j::reduce::Mean, &z4, {1});
    ASSERT_TRUE(z4.equalsTo(&exp4));

    x.reduceAlongDimension(nd4j::reduce::Mean, &z5, {0,2});
    ASSERT_TRUE(z5.equalsTo(&exp5));
}

////////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, reduceAlongDimension_float_test2) {

    NDArray x('c', {2,3,2}, {1,2,3,4,5,6,7,8,-1,-2,-3,-4,}, nd4j::DataType::DOUBLE);

    NDArray exp1('c', {}, {2.166667}, nd4j::DataType::DOUBLE);
    NDArray exp2('c', {2,2}, {3,4,1,0.666667}, nd4j::DataType::DOUBLE);
    NDArray exp3('c', {3}, {4.5,1,1}, nd4j::DataType::DOUBLE);
    NDArray exp4('c', {3,2}, {4,5,1,1,1,1}, nd4j::DataType::DOUBLE);
    NDArray exp5('c', {2}, {3.5,0.833333}, nd4j::DataType::DOUBLE);

    NDArray z1 = x.reduceAlongDims(nd4j::reduce::Mean, {0,1,2});
    ASSERT_TRUE(z1.equalsTo(&exp1));

    NDArray z2 = x.reduceAlongDims(nd4j::reduce::Mean, {1});
    ASSERT_TRUE(z2.equalsTo(&exp2));

    NDArray z3 = x.reduceAlongDims(nd4j::reduce::Mean, {0,2});
    ASSERT_TRUE(z3.equalsTo(&exp3));

    x.permutei({1,0,2});    // 3x2x2

    NDArray z4 = x.reduceAlongDims(nd4j::reduce::Mean, {0,1,2});
    ASSERT_TRUE(z4.equalsTo(&exp1));

    NDArray z5 = x.reduceAlongDims(nd4j::reduce::Mean, {1});
    ASSERT_TRUE(z5.equalsTo(&exp4));

    NDArray z6 = x.reduceAlongDims(nd4j::reduce::Mean, {0,2});
    ASSERT_TRUE(z6.equalsTo(&exp5));
}

//////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, EqualityTest1) {
    auto arrayA = NDArrayFactory::create_<float>('f', {3, 5});
    auto arrayB = NDArrayFactory::create_<float>('f', {3, 5});
    auto arrayC = NDArrayFactory::create_<float>('f', {3, 5});

    auto arrayD = NDArrayFactory::create_<float>('f', {2, 4});
    auto arrayE = NDArrayFactory::create_<float>('f', {1, 15});

    for (int i = 0; i < arrayA->rows(); i++) {
        for (int k = 0; k < arrayA->columns(); k++) {
            arrayA->p(i, k, (float) i);
        }
    }
    arrayA->printBuffer("arrayA is ");
    for (int i = 0; i < arrayB->rows(); i++) {
        for (int k = 0; k < arrayB->columns(); k++) {
            arrayB->p(i, k, (float) i);
        }
    }
    arrayB->printBuffer("arrayB is ");

    for (int i = 0; i < arrayC->rows(); i++) {
        for (int k = 0; k < arrayC->columns(); k++) {
            arrayC->p(i, k, (float) i+1);
        }
    }
    arrayC->printBuffer("arrayC is ");



    ASSERT_TRUE(arrayA->equalsTo(arrayB, 1e-5));

    ASSERT_FALSE(arrayC->equalsTo(arrayB, 1e-5));

    ASSERT_FALSE(arrayD->equalsTo(arrayB, 1e-5));

    ASSERT_FALSE(arrayE->equalsTo(arrayB, 1e-5));

    delete arrayA;
    delete arrayB;
    delete arrayC;
    delete arrayD;
    delete arrayE;
}

////////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, reduceAlongDimension_same_test1) {

    NDArray x('c', {2,3,2}, {1.5,2,3,4,5,6,7.5,8,-1,-2,-3.5,-4,}, nd4j::DataType::FLOAT32);

    NDArray z1('c', {}, {100}, nd4j::DataType::FLOAT32);
    NDArray z2('c', {2,2}, {100,100,100,100}, nd4j::DataType::FLOAT32);
    NDArray z3('c', {3}, {100,100,100}, nd4j::DataType::FLOAT32);
    NDArray z4('c', {3,2}, {100,100,100,100,100,100}, nd4j::DataType::FLOAT32);
    NDArray z5('c', {2}, {100,100}, nd4j::DataType::FLOAT32);

    NDArray exp1('c', {}, {26.5}, nd4j::DataType::FLOAT32);
    NDArray exp2('c', {2,2}, {9.5,12,3,2}, nd4j::DataType::FLOAT32);
    NDArray exp3('c', {3}, {19,4,3.5}, nd4j::DataType::FLOAT32);
    NDArray exp4('c', {3,2}, {9,10,2,2,1.5,2}, nd4j::DataType::FLOAT32);
    NDArray exp5('c', {2}, {21.5,5}, nd4j::DataType::FLOAT32);

    x.reduceAlongDimension(nd4j::reduce::Sum, &z1, {0,1,2});
    ASSERT_TRUE(z1.equalsTo(&exp1));

    x.reduceAlongDimension(nd4j::reduce::Sum, &z2, {1});
    ASSERT_TRUE(z2.equalsTo(&exp2));

    x.reduceAlongDimension(nd4j::reduce::Sum, &z3, {0,2});
    ASSERT_TRUE(z3.equalsTo(&exp3));

    x.permutei({1,0,2});    // 3x2x2

    x.reduceAlongDimension(nd4j::reduce::Sum, &z1, {0,1,2});
    ASSERT_TRUE(z1.equalsTo(&exp1));

    x.reduceAlongDimension(nd4j::reduce::Sum, &z4, {1});
    ASSERT_TRUE(z4.equalsTo(&exp4));

    x.reduceAlongDimension(nd4j::reduce::Sum, &z5, {0,2});
    ASSERT_TRUE(z5.equalsTo(&exp5));
}

////////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, reduceAlongDimension_same_test2) {

    NDArray x('c', {2,3,2}, {1.5,2,3,4,5,6,7.5,8,-1,-2,-3.5,-4,}, nd4j::DataType::INT64);

    NDArray exp1('c', {}, {26}, nd4j::DataType::INT64);
    NDArray exp2('c', {2,2}, {9,12,3,2}, nd4j::DataType::INT64);
    NDArray exp3('c', {3}, {18,4,4}, nd4j::DataType::INT64);
    NDArray exp4('c', {3,2}, {8,10,2,2,2,2}, nd4j::DataType::INT64);
    NDArray exp5('c', {2}, {21,5}, nd4j::DataType::INT64);

    NDArray z1 = x.reduceAlongDims(nd4j::reduce::Sum, {0,1,2});
    ASSERT_TRUE(z1.equalsTo(&exp1));

    NDArray z2 = x.reduceAlongDims(nd4j::reduce::Sum, {1});
    ASSERT_TRUE(z2.equalsTo(&exp2));

    NDArray z3 = x.reduceAlongDims(nd4j::reduce::Sum, {0,2});
    ASSERT_TRUE(z3.equalsTo(&exp3));

    x.permutei({1,0,2});    // 3x2x2

    NDArray z4 = x.reduceAlongDims(nd4j::reduce::Sum, {0,1,2});
    ASSERT_TRUE(z4.equalsTo(&exp1));

    NDArray z5 = x.reduceAlongDims(nd4j::reduce::Sum, {1});
    ASSERT_TRUE(z5.equalsTo(&exp4));

    NDArray z6 = x.reduceAlongDims(nd4j::reduce::Sum, {0,2});
    ASSERT_TRUE(z6.equalsTo(&exp5));
}

////////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, reduceAlongDimension_bool_test1) {

    NDArray x('c', {2,3,2}, {0.5,2,3,-4,5,6,-7.5,8,-1,-0.5,-3.5,4}, nd4j::DataType::DOUBLE);

    NDArray z1('c', {}, {100}, nd4j::DataType::BOOL);
    NDArray z2('c', {2,2}, {100,100,100,100}, nd4j::DataType::BOOL);
    NDArray z3('c', {3}, {100,100,100}, nd4j::DataType::BOOL);
    NDArray z4('c', {3,2}, {100,100,100,100,100,100}, nd4j::DataType::BOOL);
    NDArray z5('c', {2}, {100,100}, nd4j::DataType::BOOL);

    NDArray exp1('c', {}, {1}, nd4j::DataType::BOOL);
    NDArray exp2('c', {2,2}, {1,1,0,1}, nd4j::DataType::BOOL);
    NDArray exp3('c', {3}, {1,1,1}, nd4j::DataType::BOOL);
    NDArray exp4('c', {3,2}, {1,1,1,0,1,1}, nd4j::DataType::BOOL);
    NDArray exp5('c', {2}, {1,1}, nd4j::DataType::BOOL);

    x.reduceAlongDimension(nd4j::reduce::IsPositive, &z1, {0,1,2});
    ASSERT_TRUE(z1.equalsTo(&exp1));

    x.reduceAlongDimension(nd4j::reduce::IsPositive, &z2, {1});
    ASSERT_TRUE(z2.equalsTo(&exp2));

    x.reduceAlongDimension(nd4j::reduce::IsPositive, &z3, {0,2});
    ASSERT_TRUE(z3.equalsTo(&exp3));

    x.permutei({1,0,2});    // 3x2x2

    x.reduceAlongDimension(nd4j::reduce::IsPositive, &z1, {0,1,2});
    ASSERT_TRUE(z1.equalsTo(&exp1));

    x.reduceAlongDimension(nd4j::reduce::IsPositive, &z4, {1});
    ASSERT_TRUE(z4.equalsTo(&exp4));

    x.reduceAlongDimension(nd4j::reduce::IsPositive, &z5, {0,2});
    ASSERT_TRUE(z5.equalsTo(&exp5));
}

////////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, reduceAlongDimension_bool_test2) {

    NDArray x('c', {2,3,2}, {0.5,2,3,-4,5,6,-7.5,8,-1,-0.5,-3.5,4}, nd4j::DataType::INT32);

    NDArray exp1('c', {}, {1}, nd4j::DataType::BOOL);
    NDArray exp2('c', {2,2}, {1,1,0,1}, nd4j::DataType::BOOL);
    NDArray exp3('c', {3}, {1,1,1}, nd4j::DataType::BOOL);
    NDArray exp4('c', {3,2}, {0,1,1,0,1,1}, nd4j::DataType::BOOL);
    NDArray exp5('c', {2}, {1,1}, nd4j::DataType::BOOL);

    NDArray z1 = x.reduceAlongDims(nd4j::reduce::IsPositive, {0,1,2});
    ASSERT_TRUE(z1.equalsTo(&exp1));

    NDArray z2 = x.reduceAlongDims(nd4j::reduce::IsPositive, {1});
    ASSERT_TRUE(z2.equalsTo(&exp2));

    NDArray z3 = x.reduceAlongDims(nd4j::reduce::IsPositive, {0,2});
    ASSERT_TRUE(z3.equalsTo(&exp3));

    x.permutei({1,0,2});    // 3x2x2

    NDArray z4 = x.reduceAlongDims(nd4j::reduce::IsPositive, {0,1,2});
    ASSERT_TRUE(z4.equalsTo(&exp1));

    NDArray z5 = x.reduceAlongDims(nd4j::reduce::IsPositive, {1});
    ASSERT_TRUE(z5.equalsTo(&exp4));

    NDArray z6 = x.reduceAlongDims(nd4j::reduce::IsPositive, {0,2});
    ASSERT_TRUE(z6.equalsTo(&exp5));
}

////////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, reduceAlongDimension_long_test1) {

    NDArray x('c', {2,3,2}, {0.5,2,3,-0,5,6,-7.5,0,-1,-0.5,-3.5,4}, nd4j::DataType::FLOAT32);

    NDArray z1('c', {}, {100}, nd4j::DataType::INT64);
    NDArray z2('c', {2,2}, {100,100,100,100}, nd4j::DataType::INT64);
    NDArray z3('c', {3}, {100,100,100}, nd4j::DataType::INT64);
    NDArray z4('c', {3,2}, {100,100,100,100,100,100}, nd4j::DataType::INT64);
    NDArray z5('c', {2}, {100,100}, nd4j::DataType::INT64);

    NDArray exp1('c', {}, {2}, nd4j::DataType::INT64);
    NDArray exp2('c', {2,2}, {0,1,0,1}, nd4j::DataType::INT64);
    NDArray exp3('c', {3}, {1,1,0}, nd4j::DataType::INT64);
    NDArray exp4('c', {3,2}, {0,1,0,1,0,0}, nd4j::DataType::INT64);
    NDArray exp5('c', {2}, {1,1}, nd4j::DataType::INT64);

    x.reduceAlongDimension(nd4j::reduce::CountZero, &z1, {0,1,2});
    ASSERT_TRUE(z1.equalsTo(&exp1));

    x.reduceAlongDimension(nd4j::reduce::CountZero, &z2, {1});
    ASSERT_TRUE(z2.equalsTo(&exp2));

    x.reduceAlongDimension(nd4j::reduce::CountZero, &z3, {0,2});
    ASSERT_TRUE(z3.equalsTo(&exp3));

    x.permutei({1,0,2});    // 3x2x2

    x.reduceAlongDimension(nd4j::reduce::CountZero, &z1, {0,1,2});
    ASSERT_TRUE(z1.equalsTo(&exp1));

    x.reduceAlongDimension(nd4j::reduce::CountZero, &z4, {1});
    ASSERT_TRUE(z4.equalsTo(&exp4));

    x.reduceAlongDimension(nd4j::reduce::CountZero, &z5, {0,2});
    ASSERT_TRUE(z5.equalsTo(&exp5));
}

////////////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, reduceAlongDimension_long_test2) {

    NDArray x('c', {2,3,2}, {0.5,2,3,-0,5,6,-7.5,0,-1,-0.5,-3.5,4}, nd4j::DataType::INT32);

    NDArray exp1('c', {}, {4}, nd4j::DataType::INT64);
    NDArray exp2('c', {2,2}, {1,1,0,2}, nd4j::DataType::INT64);
    NDArray exp3('c', {3}, {2,2,0}, nd4j::DataType::INT64);
    NDArray exp4('c', {3,2}, {1,1,0,2,0,0}, nd4j::DataType::INT64);
    NDArray exp5('c', {2}, {2,2}, nd4j::DataType::INT64);

    NDArray z1 = x.reduceAlongDims(nd4j::reduce::CountZero, {0,1,2});
    ASSERT_TRUE(z1.equalsTo(&exp1));

    NDArray z2 = x.reduceAlongDims(nd4j::reduce::CountZero, {1});
    ASSERT_TRUE(z2.equalsTo(&exp2));

    NDArray z3 = x.reduceAlongDims(nd4j::reduce::CountZero, {0,2});
    ASSERT_TRUE(z3.equalsTo(&exp3));

    x.permutei({1,0,2});    // 3x2x2

    NDArray z4 = x.reduceAlongDims(nd4j::reduce::CountZero, {0,1,2});
    ASSERT_TRUE(z4.equalsTo(&exp1));

    NDArray z5 = x.reduceAlongDims(nd4j::reduce::CountZero, {1});
    ASSERT_TRUE(z5.equalsTo(&exp4));

    NDArray z6 = x.reduceAlongDims(nd4j::reduce::CountZero, {0,2});
    ASSERT_TRUE(z6.equalsTo(&exp5));
}

TEST_F(NDArrayCudaBasicsTests, BroadcastOpsTest1) {

    auto x = NDArrayFactory::create<float>('c', {5, 5});
    auto z = NDArrayFactory::create<float>('c', {5, 5});
    auto row = NDArrayFactory::linspace(1.0f, 5.0f, 5);
    NDArray expRow('c', {1, 5,}, {1,2,3,4,5}, nd4j::DataType::FLOAT32);
    NDArray exp('c', {5,5}, {1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5}, nd4j::DataType::FLOAT32);

    ASSERT_TRUE(row->equalsTo(&expRow));

    x.applyBroadcast(broadcast::Add, {1}, row, &z, nullptr);
    x += *row;

    ASSERT_TRUE(x.equalsTo(z));
    //ASSERT_TRUE(z.equalsTo(&exp));

    delete row;
}

TEST_F(NDArrayCudaBasicsTests, BroadcastOpsTest2) {

    auto x = NDArrayFactory::create<float>('c', {5, 5});
    //auto z = NDArrayFactory::create<float>('c', {5, 5});
    auto row = NDArrayFactory::linspace(1.0f, 5.0f, 5);
    NDArray expRow('c', {1, 5,}, {1,2,3,4,5}, nd4j::DataType::FLOAT32);
    NDArray exp('c', {5,5}, {1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5}, nd4j::DataType::FLOAT32);

    ASSERT_TRUE(row->equalsTo(&expRow));
    x.applyBroadcast(broadcast::Add, {1}, row);
    ASSERT_TRUE(x.equalsTo(&exp));
}

//////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestBroadcast_1) {

    NDArray exp('c', {2, 3, 2, 2}, {1., 1., 1., 1., 2., 2., 2., 2., 3., 3., 3., 3., 1., 1., 1., 1., 2., 2., 2., 2., 3., 3., 3., 3.}, nd4j::DataType::DOUBLE);

    auto input = NDArrayFactory::create<double>('c',{ 2, 3, 2, 2});
    auto bias = NDArrayFactory::create<double>('c', {1, 3});

    bias.linspace(1);
    input.applyBroadcast(broadcast::Add, {1}, &bias);
    ASSERT_TRUE(exp.equalsTo(&input));
}

TEST_F(NDArrayCudaBasicsTests, TestFloat16_1) {
    auto x = NDArrayFactory::create<float>({1,2,3,4,5,7,8,9});
    auto y = NDArrayFactory::create<float>({1,2,3,4,5,7,8,9});
    ASSERT_TRUE(x.equalsTo(&y));
}

TEST_F(NDArrayCudaBasicsTests, TestFloat16_2) {
    auto x = NDArrayFactory::create<float16>('c', {9}, {1,2,3,4,5,6,7,8,9});
    auto y = NDArrayFactory::create<float16>('c', {9}, {1,2,3,4,5,6,7,8,9});
    ASSERT_TRUE(x.equalsTo(y));
    //for (int e = 0; e < x.lengthOf(); e++)
    //    ASSERT_NEAR(x.e<float16>(e), y.e<float16>(e), 1.e-5f);
}

TEST_F(NDArrayCudaBasicsTests, TestFloat16_3) {
    auto x = NDArrayFactory::create<bfloat16>({1,2,3,4,5,7,8,9});
    auto y = NDArrayFactory::create<bfloat16>({1,2,3,4,5,7,8,9});
    ASSERT_TRUE(x.equalsTo(&y));
}

TEST_F(NDArrayCudaBasicsTests, TestFloat_4) {
    auto x = NDArrayFactory::create<float>({1,2,3,4,5,7,8,9});
    auto y = NDArrayFactory::create<float>({2,4,5,5,6,7,8,9});
    ASSERT_FALSE(x.equalsTo(&y));
}

TEST_F(NDArrayCudaBasicsTests, TestFloat_5) {
    auto x = NDArrayFactory::create<float>('c', {3,3}, {1,2,3,4,5,6,7,8,9});
    auto y = NDArrayFactory::create<float>('c', {3,3}, {2,4,5,5,6,7,8,9, 10});
    ASSERT_FALSE(x.equalsTo(&y));
}

TEST_F(NDArrayCudaBasicsTests, TestFloat_6) {
    auto x = NDArrayFactory::create<float>('f', {3,3}, {1,2,3,4,5,6,7,8,9});
    auto y = NDArrayFactory::create<float>('f', {3,3}, {2,4,5,5,6,7,8,9,10});
    ASSERT_FALSE(x.equalsTo(&y));
}

//////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, Operator_Plus_Test_05)
{
    auto x = NDArrayFactory::create<float>('c', {8, 8, 8});
    auto y = NDArrayFactory::create<float>('c', {1, 8, 8});
    auto expected = NDArrayFactory::create<float>('c', {8, 8, 8});
    NDArray res2  = NDArrayFactory::create<float>(expected.ordering(), expected.getShapeAsVector());
    x = 1.;
    y = 2.;
    expected = 3.;
    res2 = 0.f;

    x.applyTrueBroadcast(BroadcastOpsTuple::Add(), &y, &res2);// *= y;

    ASSERT_TRUE(expected.isSameShape(&res2));
    ASSERT_TRUE(expected.equalsTo(&res2));
}

//////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, Operator_Plus_Test_5)
{
    auto x = NDArrayFactory::create<float>('c', {8, 8, 8});
    auto y = NDArrayFactory::create<float>('c', {8, 1, 8});
    auto expected = NDArrayFactory::create<float>('c', {8, 8, 8});
    NDArray res2(expected);
    x = 1.;
    y = 2.;
    expected = 3.;
    //x.printBuffer("X=");
    //y.printBuffer("Y=");
    //expected.printBuffer("EXPECTED");
    auto result = x + y;
    //result.printBuffer("1 + 2 =");
    //res2.assign(x + y);

    //x.applyTrueBroadcast(BroadcastOpsTuple::Add(), &y, &res2);
    //res2.printBuffer("Z=");
    //x.applyTrueBroadcast(BroadcastOpsTuple::Add(), &y, &res2);// *= y;
//    x += y;
    //x.printBuffer("OutputX");
    //res2.syncToHost();
    //res2.printBuffer("OUputZ");
    //x.printIndexedBuffer("OUtputX");
    ASSERT_TRUE(expected.isSameShape(&result));
    ASSERT_TRUE(expected.equalsTo(&result));
}

//////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, Operator_Plus_Test_51)
{
    auto x = NDArrayFactory::create<float>('c', {8, 8, 8});
    auto y = NDArrayFactory::create<float>('c', {8, 8});
    auto expected = NDArrayFactory::create<float>('c', {8, 8, 8});
    NDArray res2(expected);
    x = 1.;
    y = 2.;
    expected = 3.;
    //x.printBuffer("X=");
    //y.printBuffer("Y=");
    //expected.printBuffer("EXPECTED");
    auto result = x + y;
    //result.printBuffer("1 + 2 =");
    //res2.assign(x + y);

    //x.applyTrueBroadcast(BroadcastOpsTuple::Add(), &y, &res2);
    //res2.printBuffer("Z=");
    //x.applyTrueBroadcast(BroadcastOpsTuple::Add(), &y, &res2);// *= y;
//    x += y;
    //x.printBuffer("OutputX");
    //res2.syncToHost();
    //res2.printBuffer("OUputZ");
    //x.printIndexedBuffer("OUtputX");
    ASSERT_TRUE(expected.isSameShape(&result));
    ASSERT_TRUE(expected.equalsTo(&result));
}

TEST_F(NDArrayCudaBasicsTests, Tile_Test_2_1)
{
    auto x = NDArrayFactory::create<float>('c', {2, 1, 2});
    x = 10.;
    auto y = x.tile({1,2,1});
    auto exp = NDArrayFactory::create<float>('c', {2, 2, 2});
    exp = 10.;

    // y.printShapeInfo("Output SHAPE");
    // y.printBuffer("Output TILE");
    // exp.printBuffer("Expect TILE");
    ASSERT_TRUE(exp.equalsTo(y));
}

TEST_F(NDArrayCudaBasicsTests, Tile_Test_2_2)
{
    auto x = NDArrayFactory::create<float>('f', {2, 1, 2});
    x = 10.;
    auto y = x.tile({1,2,1});
    auto exp = NDArrayFactory::create<float>('f', {2, 2, 2});
    exp = 10.;
    y.printShapeInfo("Output SHAPE");
    y.printBuffer("Output TILE");
    ASSERT_TRUE(exp.equalsTo(y));
}

TEST_F(NDArrayCudaBasicsTests, Tile_Test_2_3)
{
    auto x = NDArrayFactory::create<float>('f', {2, 1, 2});
    x = 10.;
    x.p(1,0,1, 20);
    x.syncToDevice();
    auto y = x.tile({1,2,1});
    auto exp = NDArrayFactory::create<float>('f', {2, 2, 2});
    exp = 10.;
    exp.p(1,0,1, 20.);
    exp.p(1, 1, 1, 20.);
    exp.syncToDevice();
    ASSERT_TRUE(exp.equalsTo(y));
}

//////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, Operator_Plus_Test_2)
{
    double expBuff[] = {2., 3, 3., 4., 4., 5, 5., 6., 6., 7, 7., 8.};
    NDArray a('c', {4,4}, {1.,2,3,4,5,6,7,8,9,2,3,2,1,0,4,7.}, nd4j::DataType::FLOAT32);
    a.printBuffer();
    auto x = NDArrayFactory::create<double>('c', {3, 2, 1});
    auto y = NDArrayFactory::create<double>('c',    {1, 2});
    auto expected = NDArrayFactory::create<double>(expBuff, 'c', {3, 2, 2});

    x.linspace(1);
    y.linspace(1);
    x.printBuffer("X=");
    y.printBuffer("Y=");
    auto result = x + y;
    result.printIndexedBuffer("Result");

    ASSERT_TRUE(expected.isSameShape(&result));
    ASSERT_TRUE(expected.equalsTo(&result));
}

//////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, assign_2)
{
    NDArray x('c', {4}, {1.5,2.5,3.5,4.5}, nd4j::DataType::FLOAT32);
    NDArray y('c', {4}, nd4j::DataType::INT32);
    NDArray expected('c', {4}, {1,2,3,4}, nd4j::DataType::INT32);

    y.assign(x);
    // y.printBuffer("ASSIGN VECTOR");

    ASSERT_TRUE(expected.equalsTo(&y));
}

//////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, subarray_1)
{
    NDArray x('c', {2,3,4}, {1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18,19,20,21,22,23,24}, nd4j::DataType::FLOAT32);
    NDArray y('f', {2,3,4}, {1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18,19,20,21,22,23,24}, nd4j::DataType::FLOAT32);

    Nd4jLong shapeExpX0[] = {1, 2, 12, 8192, 1, 99};
    float    buffExpX0[]  = {1.000000, 13.000000};
    Nd4jLong shapeExpX1[] = {1, 2, 12, 8192, 1, 99};
    float    buffExpX1[]  = {2.000000, 14.000000};
    Nd4jLong shapeExpX2[] = {3, 2, 1, 1, 12, 4, 1, 8192, 1, 99};
    float    buffExpX2[]  = {1.000000, 13.000000};
    Nd4jLong shapeExpX3[] = {2, 2, 4, 12, 1, 8192, 1, 99};
    float    buffExpX3[]  = {9.000000, 10.000000, 11.000000, 12.000000, 21.000000, 22.000000, 23.000000, 24.000000};
    Nd4jLong shapeExpX4[] = {3, 2, 1, 4, 12, 4, 1, 8192, 1, 99};
    float    buffExpX4[]  = {9.000000, 10.000000, 11.000000, 12.000000, 21.000000, 22.000000, 23.000000, 24.000000};
    Nd4jLong shapeExpX5[] = {2, 2, 3, 12, 4, 8192, 1, 99};
    float    buffExpX5[]  = {4.000000, 8.000000, 12.000000, 16.000000, 20.000000, 24.000000};

    Nd4jLong shapeExpY0[] = {1, 2, 1, 8192, 1, 99};
    float    buffExpY0[]  = {1.000000, 2.000000};
    Nd4jLong shapeExpY1[] = {1, 2, 1, 8192, 1, 99};
    float    buffExpY1[]  = {7.000000, 8.000000};
    Nd4jLong shapeExpY2[] = {3, 2, 1, 1, 1, 2, 6, 8192, 1, 102};
    float    buffExpY2[]  = {1.000000, 2.000000};
    Nd4jLong shapeExpY3[] = {2, 2, 4, 1, 6, 8192, 1, 99};
    float    buffExpY3[]  = {5.000000, 11.000000, 17.000000, 23.000000, 6.000000, 12.000000, 18.000000, 24.000000};
    Nd4jLong shapeExpY4[] = {3, 2, 1, 4, 1, 2, 6, 8192, 1, 102};
    float    buffExpY4[]  = {5.000000, 11.000000, 17.000000, 23.000000, 6.000000, 12.000000, 18.000000, 24.000000};
    Nd4jLong shapeExpY5[] = {2, 2, 3, 1, 2, 8192, 1, 99};
    float    buffExpY5[]  = {19.000000, 21.000000, 23.000000, 20.000000, 22.000000, 24.000000};


    NDArray x0 = x(0, {1,2});
    NDArray xExp(buffExpX0, shapeExpX0);

    ASSERT_TRUE(xExp.isSameShape(x0));
    ASSERT_TRUE(xExp.equalsTo(x0));
//    for(int i = 0; i < shape::shapeInfoLength(x0.rankOf()); ++i)
//        ASSERT_TRUE(x0.getShapeInfo()[i] == shapeExpX0[i]);
//    for(int i = 0; i < x0.lengthOf(); ++i)
//        ASSERT_TRUE(x0.e<float>(i) == buffExpX0[i]);

    NDArray x1 = x(1, {1,2});
    NDArray x1Exp(buffExpX1, shapeExpX1);
    ASSERT_TRUE(x1Exp.isSameShape(x1));
    ASSERT_TRUE(x1Exp.equalsTo(x1));

//    for(int i = 0; i < shape::shapeInfoLength(x1.rankOf()); ++i)
//        ASSERT_TRUE(x1.getShapeInfo()[i] == shapeExpX1[i]);
//    for(int i = 0; i < x1.lengthOf(); ++i)
//        ASSERT_TRUE(x1.e<float>(i) == buffExpX1[i]);

    NDArray x2 = x(0, {1,2}, true);
    NDArray x2Exp(buffExpX2, shapeExpX2);
    ASSERT_TRUE(x2Exp.isSameShape(x2));
//    x2.printBuffer("X2");
//    x2Exp.printBuffer("X2 EXPECT");
    ASSERT_TRUE(x2Exp.equalsTo(x2));
//    for(int i = 0; i < shape::shapeInfoLength(x2.rankOf()); ++i)
//        ASSERT_TRUE(x2.getShapeInfo()[i] == shapeExpX2[i]);
//    for(int i = 0; i < x2.lengthOf(); ++i)
//        ASSERT_TRUE(x2.e<float>(i) == buffExpX2[i]);

    NDArray x3 = x(2, {1});
    NDArray x3Exp(buffExpX3, shapeExpX3);
    ASSERT_TRUE(x3Exp.isSameShape(x3));
    ASSERT_TRUE(x3Exp.equalsTo(x3));
//    for(int i = 0; i < shape::shapeInfoLength(x3.rankOf()); ++i)
//        ASSERT_TRUE(x3.getShapeInfo()[i] == shapeExpX3[i]);
//    for(int i = 0; i < x3.lengthOf(); ++i)
//        ASSERT_TRUE(x3.e<float>(i) == buffExpX3[i]);

    NDArray x4 = x(2, {1}, true);
    NDArray x4Exp(buffExpX4, shapeExpX4);
    ASSERT_TRUE(x4Exp.isSameShape(x4));
    ASSERT_TRUE(x4Exp.equalsTo(x4));
//    for(int i = 0; i < shape::shapeInfoLength(x4.rankOf()); ++i)
//        ASSERT_TRUE(x4.getShapeInfo()[i] == shapeExpX4[i]);
//    for(int i = 0; i < x4.lengthOf(); ++i)
//        ASSERT_TRUE(x4.e<float>(i) == buffExpX4[i]);

    NDArray x5 = x(3, {2});
    NDArray x5Exp(buffExpX5, shapeExpX5);
    ASSERT_TRUE(x5Exp.isSameShape(x5));
    ASSERT_TRUE(x5Exp.equalsTo(x5));

//    for(int i = 0; i < shape::shapeInfoLength(x5.rankOf()); ++i)
//        ASSERT_TRUE(x5.getShapeInfo()[i] == shapeExpX5[i]);
//    for(int i = 0; i < x5.lengthOf(); ++i)
//        ASSERT_TRUE(x5.e<float>(i) == buffExpX5[i]);

    // ******************* //
    NDArray y0 = y(0, {1,2});
    NDArray y0Exp(buffExpY0, shapeExpY0);
    ASSERT_TRUE(y0Exp.isSameShape(y0));
    ASSERT_TRUE(y0Exp.equalsTo(y0));
//    for(int i = 0; i < shape::shapeInfoLength(y0.rankOf()); ++i)
//        ASSERT_TRUE(y0.getShapeInfo()[i] == shapeExpY0[i]);
//    for(int i = 0; i < y0.lengthOf(); ++i)
//        ASSERT_TRUE(y0.e<float>(i) == buffExpY0[i]);

    NDArray y1 = y(1, {1,2});
    NDArray y1Exp(buffExpY1, shapeExpY1);
    ASSERT_TRUE(y1Exp.isSameShape(y1));
    ASSERT_TRUE(y1Exp.equalsTo(y1));
//    for(int i = 0; i < shape::shapeInfoLength(y1.rankOf()); ++i)
//        ASSERT_TRUE(y1.getShapeInfo()[i] == shapeExpY1[i]);
//    for(int i = 0; i < y1.lengthOf(); ++i)
//        ASSERT_TRUE(y1.e<float>(i) == buffExpY1[i]);

    NDArray y2 = y(0, {1,2}, true);
    NDArray y2Exp(buffExpY2, shapeExpY2);
    ASSERT_TRUE(y2Exp.isSameShape(y2));
    ASSERT_TRUE(y2Exp.equalsTo(y2));
//    for(int i = 0; i < shape::shapeInfoLength(y2.rankOf()); ++i)
//        ASSERT_TRUE(y2.getShapeInfo()[i] == shapeExpY2[i]);
//    for(int i = 0; i < y2.lengthOf(); ++i)
//        ASSERT_TRUE(y2.e<float>(i) == buffExpY2[i]);

    NDArray y3 = y(2, {1});
    NDArray y3Exp(buffExpY3, shapeExpY3);
    ASSERT_TRUE(y3Exp.isSameShape(y3));
    ASSERT_TRUE(y3Exp.equalsTo(y3));
//    for(int i = 0; i < shape::shapeInfoLength(y3.rankOf()); ++i)
//        ASSERT_TRUE(y3.getShapeInfo()[i] == shapeExpY3[i]);
//    for(int i = 0; i < y3.lengthOf(); ++i)
//        ASSERT_TRUE(y3.e<float>(i) == buffExpY3[i]);

    NDArray y4 = y(2, {1}, true);
    NDArray y4Exp = NDArrayFactory::create<float>('f', {2,1,4}, {5, 6, 11, 12, 17, 18, 23, 24});
    ASSERT_TRUE(y4Exp.isSameShape(y4));
    ASSERT_TRUE(y4Exp.equalsTo(y4));
//    for(int i = 0; i < shape::shapeInfoLength(y4.rankOf()); ++i)
//        ASSERT_TRUE(y4.getShapeInfo()[i] == shapeExpY4[i]);
//    for(int i = 0; i < y4.lengthOf(); ++i)
//        ASSERT_TRUE(y4.e<float>(i) == buffExpY4[i]);

    NDArray y5 = y(3, {2});
    NDArray y5Exp(buffExpY5, shapeExpY5);
    ASSERT_TRUE(y5Exp.isSameShape(y5));
    ASSERT_TRUE(y5Exp.equalsTo(y5));
//    for(int i = 0; i < shape::shapeInfoLength(y5.rankOf()); ++i)
//        ASSERT_TRUE(y5.getShapeInfo()[i] == shapeExpY5[i]);
//    for(int i = 0; i < y5.lengthOf(); ++i)
//        ASSERT_TRUE(y5.e<float>(i) == buffExpY5[i]);

}
//////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, Test_diagonal_1) {

    auto x = NDArrayFactory::create<float>('c', {2, 3}, {1, 2, 3, 4, 5, 6});
    auto exp = NDArrayFactory::create<float>('c', {2, 1}, {1, 5});

    auto diag = x.diagonal('c');
    //diag->syncToDevice();
    for (Nd4jLong e = 0; e < exp.lengthOf(); ++e) {
        printf("VAL[%ld] = %f\n", e, diag->e<float>(e)); //, exp.e<float>(e), 1.e-5);
    }
    diag->printIndexedBuffer("DIAGONAL");
    for (Nd4jLong e = 0; e < exp.lengthOf(); ++e) {
        ASSERT_NEAR(diag->e<float>(e), exp.e<float>(e), 1.e-5);
    }
    double eps(1.e-5);
    NDArray tmp(nd4j::DataType::FLOAT32, x.getContext()); // scalar = 0

    ExtraArguments extras({eps});
    NativeOpExecutioner::execReduce3Scalar(diag->getContext(), reduce3::EqualsWithEps, diag->getBuffer(),
            diag->getShapeInfo(), diag->getSpecialBuffer(), diag->getSpecialShapeInfo(), extras.argumentsAsT(nd4j::DataType::FLOAT32),
            exp.getBuffer(), exp.getShapeInfo(), exp.getSpecialBuffer(), exp.getSpecialShapeInfo(),
            tmp.buffer(), tmp.shapeInfo(), tmp.specialBuffer(), tmp.specialShapeInfo());
    hipStream_t* stream = x.getContext()->getCudaStream();
    auto res = hipStreamSynchronize(*stream);
    // tmp.printBuffer("Compare result is (expected 0)");
    ASSERT_TRUE(exp.isSameShape(diag));
    ASSERT_TRUE(exp.equalsTo(diag));

    delete diag;
}

TEST_F(NDArrayCudaBasicsTests, Test_PermuteEquality_02) {
    auto x = NDArrayFactory::linspace<float>(1.f, 60.f, 60); //('c', {1, 60});
    //x.linspace(1);
    auto exp = NDArrayFactory::create<float>('c', {3, 4, 5}, {1.0, 2.0, 3.0, 4.0, 5.0, 6.0, 7.0, 8.0, 9.0, 10.0, 11.0, 12.0, 13.0, 14.0, 15.0, 16.0, 17.0, 18.0, 19.0, 20.0, 21.0, 22.0, 23.0, 24.0, 25.0, 26.0, 27.0, 28.0, 29.0, 30.0, 31.0, 32.0, 33.0, 34.0, 35.0, 36.0, 37.0, 38.0, 39.0, 40.0, 41.0, 42.0, 43.0, 44.0, 45.0, 46.0, 47.0, 48.0, 49.0, 50.0, 51.0, 52.0, 53.0, 54.0, 55.0, 56.0, 57.0, 58.0, 59.0, 60.0});
    x->reshapei('c', {3, 4, 5});

    x->permutei({0, 1, 2});
    x->streamline();

//    x.printShapeInfo("{0, 1, 2} shape");
//    x.printBuffer("{0, 1, 2} data");

    ASSERT_TRUE(exp.isSameShape(x));
    ASSERT_TRUE(exp.equalsTo(x));
    delete x;
}

TEST_F(NDArrayCudaBasicsTests, Test_PermuteEquality_0) {
    auto x = NDArrayFactory::create<float>('c', {1, 60});
    x.linspace(1);
    auto exp = NDArrayFactory::create<float>('c', {3, 4, 5}, {1.0, 2.0, 3.0, 4.0, 5.0, 6.0, 7.0, 8.0, 9.0, 10.0, 11.0, 12.0, 13.0, 14.0, 15.0, 16.0, 17.0, 18.0, 19.0, 20.0, 21.0, 22.0, 23.0, 24.0, 25.0, 26.0, 27.0, 28.0, 29.0, 30.0, 31.0, 32.0, 33.0, 34.0, 35.0, 36.0, 37.0, 38.0, 39.0, 40.0, 41.0, 42.0, 43.0, 44.0, 45.0, 46.0, 47.0, 48.0, 49.0, 50.0, 51.0, 52.0, 53.0, 54.0, 55.0, 56.0, 57.0, 58.0, 59.0, 60.0});
    x.reshapei('c', {3, 4, 5});

    x.permutei({0, 1, 2});
    x.streamline();

//    x.printShapeInfo("{0, 1, 2} shape");
//    x.printBuffer("{0, 1, 2} data");

    ASSERT_TRUE(exp.isSameShape(&x));
    ASSERT_TRUE(exp.equalsTo(&x));
}
TEST_F(NDArrayCudaBasicsTests, Test_PermuteEquality_1) {
    auto x = NDArrayFactory::create<float>('c', {1, 60});
    x.linspace(1);
    auto exp = NDArrayFactory::create<float>('c', {3, 4, 5}, {1.0, 2.0, 3.0, 4.0, 5.0, 6.0, 7.0, 8.0, 9.0, 10.0, 11.0, 12.0, 13.0, 14.0, 15.0, 16.0, 17.0, 18.0, 19.0, 20.0, 21.0, 22.0, 23.0, 24.0, 25.0, 26.0, 27.0, 28.0, 29.0, 30.0, 31.0, 32.0, 33.0, 34.0, 35.0, 36.0, 37.0, 38.0, 39.0, 40.0, 41.0, 42.0, 43.0, 44.0, 45.0, 46.0, 47.0, 48.0, 49.0, 50.0, 51.0, 52.0, 53.0, 54.0, 55.0, 56.0, 57.0, 58.0, 59.0, 60.0});
    x.reshapei('c', {3, 4, 5});

    x.permutei({0, 1, 2});
    x.streamline();

//    x.printShapeInfo("{0, 1, 2} shape");
//    x.printBuffer("{0, 1, 2} data");

    ASSERT_TRUE(exp.isSameShape(&x));
    ASSERT_TRUE(exp.equalsTo(&x));
}
TEST_F(NDArrayCudaBasicsTests, Test_PermuteEquality_2) {
    //auto x = NDArrayFactory::create<float>('c', {1, 60});
    auto xx = NDArrayFactory::linspace<float>(1.f, 60.f, 60); //('c', {1, 60});
//    auto x = *xx;
    //x.linspace(1);
//    auto exp = NDArrayFactory::create<float>('c', {3, 4, 5}, {1.0, 2.0, 3.0, 4.0, 5.0, 6.0, 7.0, 8.0, 9.0, 10.0, 11.0, 12.0, 13.0, 14.0, 15.0, 16.0, 17.0, 18.0, 19.0, 20.0, 21.0, 22.0, 23.0, 24.0, 25.0, 26.0, 27.0, 28.0, 29.0, 30.0, 31.0, 32.0, 33.0, 34.0, 35.0, 36.0, 37.0, 38.0, 39.0, 40.0, 41.0, 42.0, 43.0, 44.0, 45.0, 46.0, 47.0, 48.0, 49.0, 50.0, 51.0, 52.0, 53.0, 54.0, 55.0, 56.0, 57.0, 58.0, 59.0, 60.0});
//    x.reshapei('c', {3, 4, 5});

//    x.permutei({0, 1, 2});
//    x.streamline();

//    x.printShapeInfo("{0, 1, 2} shape");
//    x.printBuffer("{0, 1, 2} data");

//    ASSERT_TRUE(exp.isSameShape(&x));
//    ASSERT_TRUE(exp.equalsTo(&x));
    delete xx;
}
TEST_F(NDArrayCudaBasicsTests, Test_PermuteEquality_3) {
    auto x = NDArrayFactory::create<float>('c', {1, 60});
    //x.linspace(1);
    for (int l = 0; l < x.lengthOf(); l++)
        x.p(l, float(l + 1.f));
    auto exp = NDArrayFactory::create<float>('c', {3, 4, 5}, {1.0, 2.0, 3.0, 4.0, 5.0, 6.0, 7.0, 8.0, 9.0, 10.0, 11.0, 12.0, 13.0, 14.0, 15.0, 16.0, 17.0, 18.0, 19.0, 20.0, 21.0, 22.0, 23.0, 24.0, 25.0, 26.0, 27.0, 28.0, 29.0, 30.0, 31.0, 32.0, 33.0, 34.0, 35.0, 36.0, 37.0, 38.0, 39.0, 40.0, 41.0, 42.0, 43.0, 44.0, 45.0, 46.0, 47.0, 48.0, 49.0, 50.0, 51.0, 52.0, 53.0, 54.0, 55.0, 56.0, 57.0, 58.0, 59.0, 60.0});
    x.reshapei('c', {3, 4, 5});

    x.permutei({0, 1, 2});
    x.streamline();

//    x.printShapeInfo("{0, 1, 2} shape");
//    x.printBuffer("{0, 1, 2} data");

    ASSERT_TRUE(exp.isSameShape(&x));
    ASSERT_TRUE(exp.equalsTo(&x));
}

TEST_F(NDArrayCudaBasicsTests, Test_Empty_1) {
    auto x = NDArrayFactory::empty<float>();
    ASSERT_TRUE(x.isActualOnHostSide());
    ASSERT_TRUE(x.isEmpty());
}

TEST_F(NDArrayCudaBasicsTests, Test_Empty_2) {
    auto x = NDArrayFactory::empty_<float>();

    ASSERT_TRUE(x->isEmpty());
    delete x;
}

TEST_F(NDArrayCudaBasicsTests, Test_Empty_3) {
    auto x = NDArrayFactory::empty(nd4j::DataType::FLOAT32);

    ASSERT_TRUE(x.isEmpty());
}

TEST_F(NDArrayCudaBasicsTests, Test_Empty_4) {
    auto x = NDArrayFactory::empty_(nd4j::DataType::FLOAT32);

    ASSERT_TRUE(x->isEmpty());
    delete x;
}